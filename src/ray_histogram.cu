#include "hip/hip_runtime.h"
/**
 * Copyright 2013 Erik Zenker, Carlchristian Eckert, Marius Melzer
 *
 * This file is part of HASEonGPU
 *
 * HASEonGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * HASEonGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with HASEonGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */


#include <iostream>
#include <iomanip>
#include <vector>
#include <map>

#include <logging.hpp>

void ray_histogram(const std::vector<unsigned> totalRays, const unsigned max, const double mseThreshold, const std::vector<double> mseValues){
  // length of the maximum number of samples (e.g. max==4210)
  int fillwidth = log10(max)+4;

  // maximum length of the filling bar
  unsigned maxLength=50;

  // necessary size of the histogram
  std::map<unsigned,unsigned> histGreen;
  std::map<unsigned,unsigned> histRed;
  // if the entry doesn't exist, create it
  for(unsigned j=0; j<totalRays.size() ; ++j){
    //std::map<unsigned,unsigned>::iterator it = hist.find(totalRays.at(j));
    if(histGreen.find(totalRays.at(j)) == histGreen.end()){
      histGreen.insert(std::pair<unsigned,unsigned>(totalRays.at(j),0));
      histRed.insert(std::pair<unsigned,unsigned>(totalRays.at(j),0));
    }
    if(mseValues.at(j) <= mseThreshold){
      histGreen.find(totalRays.at(j))->second++;
      //itG->second++;
    } else{
      histRed.find(totalRays.at(j))->second++;
      //itR->second++; 
    }
  }


  std::map<unsigned,unsigned>::iterator itG;
  std::map<unsigned,unsigned>::iterator itR;
  for(itG=histGreen.begin(), itR=histRed.begin(); itG!=histGreen.end(); ++itG, ++itR){
    dout(V_STAT) << std::setw(fillwidth) << std::setfill(' ') <<  itG->first << " (";
    dout(V_STAT | V_NOLABEL) << "\033[0;32m" << std::setw(log10(totalRays.size())+3) << itG->second << "x";
    dout(V_STAT | V_NOLABEL) << "\033[0m" << " / ";
    dout(V_STAT | V_NOLABEL) << "\033[0;31m" << std::setw(log10(totalRays.size())+3) << itR->second << "x";
    dout(V_STAT | V_NOLABEL) << "\033[0m" << "):";

    // set color = green
    dout(V_STAT | V_NOLABEL) << "\033[0;32m";
    for(unsigned j=0;j< ceil(maxLength*(float(itG->second)/totalRays.size())) ; ++j){
      dout(V_STAT | V_NOLABEL) << "#"; 
    }

    // set color = red
    dout(V_STAT | V_NOLABEL) << "\033[0;31m";
    for(unsigned j=0;j< ceil(maxLength*(float(itR->second)/totalRays.size())) ; ++j){
      dout(V_STAT | V_NOLABEL) << "#"; 
    }
    dout(V_STAT | V_NOLABEL) << std::endl;
  }
}
