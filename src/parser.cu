#include "hip/hip_runtime.h"
#include "parser.h"
#include <string> /* string */
#include <vector> /* vector */
#include <stdio.h> /*fprintf*/
#include <logging.h> 

void parseCommandLine(
    const int argc,
    char** argv,
    unsigned *raysPerSample,
    unsigned *maxRaysPerSample,
    std::string *root,
    bool *silent,
    bool *writeVtk,
    std::string *compareLocation,
    RunMode *mode,
    bool *useReflections,
    unsigned *maxgpus,
    int *minSample_i,
    int *maxSample_i
    ) {

  std::vector<std::pair<std::string, std::string> > parameters;

  // Parse Commandline
  for (int i = 1; i < argc; ++i) {

    char* pos = strtok(argv[i], "=");
    std::pair < std::string, std::string > p(std::string(pos), std::string(""));
    pos = strtok(NULL, "=");
    if (pos != NULL) {
      p.second = std::string(pos);
    }
    parameters.push_back(p);
  }
  for (unsigned i = 0; i < parameters.size(); ++i) {
    std::pair < std::string, std::string > p = parameters.at(i);
    dout(V_INFO) << "arg[" << i << "]: (" << p.first << "," << p.second << ")" << std::endl;

    // Parse number of rays
    if (p.first == "--rays") {
      *raysPerSample = atoi(p.second.c_str());
    }

    if (p.first == "--maxrays"){
      *maxRaysPerSample = atoi(p.second.c_str());
    }

    if (p.first == "--experiment") {
      std::string temp_root(p.second);

      // Add slash at the end, if missing
      if ((temp_root)[temp_root.size() - 1] == 'w')
        temp_root.erase(temp_root.size() - 1, 1);
      else if (temp_root[temp_root.size() - 1] != '/')
        temp_root.append("/");

      *root = temp_root;
    }

    // Parse if we want less output
    if (p.first == "--silent") {
      *silent = true;
    }

    if (p.first == "--write-vtk") {
      *writeVtk = true;
    }

    // Parse what vtk file to compare with
    if (p.first == "--compare") {
      *compareLocation = p.second;
    }

    if (p.first == "--mode") {
      if (p.second == "ray_propagation_gpu")
        *mode = RAY_PROPAGATION_GPU;
      if (p.second == "for_loops")
        *mode = FOR_LOOPS;
      if (p.second == "test_environment")
        *mode = TEST;
      if (p.second == "mpi")
        *mode = RAY_PROPAGATION_MPI;

    }

    if (p.first == "--reflection"){
      *useReflections = true;
    }

    if (p.first == "--maxgpus"){
      *maxgpus = atoi(p.second.c_str());
    }

    if (p.first == "--min_sample_i"){
      *minSample_i = atoi(p.second.c_str());
    }
    if (p.first == "--max_sample_i"){
      *maxSample_i = atoi(p.second.c_str());
    }


  }
}

int checkParameterValidity(
    const int argc,
    const unsigned raysPerSample,
    unsigned *maxRaysPerSample,
    const std::string root,
    const unsigned deviceCount,
    const RunMode mode,
    unsigned *maxgpus,
    const int minSample_i,
    const int maxSample_i
    ) {

  if (argc <= 1) {
    dout(V_ERROR) << "No commandline arguments found" << std::endl;
    dout(V_ERROR) << "Usage    : ./octrace --mode=[runmode]" << std::endl;
    dout(V_ERROR) << "                     --rays=[number of rays]" << std::endl;
    dout(V_ERROR) << "                     --experiment=[location to experiment-data]" << std::endl;
    dout(V_ERROR) << "                     --compare=[location of vtk-file to compare with]" << std::endl;
    dout(V_ERROR) << "                     --maxrays=[max number of rays for adaptive sampling]" << std::endl;
    dout(V_ERROR) << "                     --maxgpus=[max number of gpus to use]" << std::endl;
    dout(V_ERROR) << "Runmodes : for_loops" << std::endl;
    dout(V_ERROR) << "           ray_propagation_gpu" << std::endl;
    dout(V_ERROR) << "           mpi" << std::endl;
    dout(V_ERROR) << "           test_environment" << std::endl;
    return 1;
  }
  if (mode == NONE) {
    dout(V_ERROR) << "Please specify the runmode with --mode=MODE" << std::endl;
    return 1;
  }
  if (raysPerSample == 0) {
    dout(V_ERROR) << "Please specify the number of rays per sample Point with --rays=RAYS" << std::endl;
    return 1;
  }
  if (root.size() == 0) {
    dout(V_ERROR) << "Please specify the experiment's location with --experiment=PATH_TO_EXPERIMENT" << std::endl;
    return 1;
  }

  *maxRaysPerSample = max(raysPerSample,*maxRaysPerSample);

  if(*maxgpus > deviceCount){
    dout(V_ERROR) << "You don't have so many devices, use --maxgpus=" << deviceCount << std::endl;
    return 1;
  }

  if(*maxgpus == 0){
    *maxgpus = deviceCount;
  }

  if(minSample_i < 0){
    dout(V_ERROR) << "--min_sample_i < 0!" << std::endl;
    return 1;
  }

  if(maxSample_i < minSample_i){
    dout(V_ERROR) << "maxSample_i < minSample_i!" << std::endl;
    return 1;
  }

  int samplesForNode = maxSample_i-minSample_i+1;
  if(samplesForNode < *maxgpus){
    dout(V_WARN) << "More GPUs requested than there are sample points. Number of used GPUs reduced to " << samplesForNode << std::endl;
     *maxgpus = samplesForNode;
  }
  return 0;
}
