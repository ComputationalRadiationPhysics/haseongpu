
#include <hip/hip_runtime.h>
#include <string> /* string */
#include <vector> /* vector */
#include <stdio.h> /*fprintf*/

void parseCommandLine(
    const int argc,
    char** argv,
    unsigned *raysPerSample,
    std::string *root,
    int *device,
    bool *silent,
    std::string *compareLocation,
    int *mode
    ) {

  std::vector<std::pair<std::string, std::string> > parameters;

  // Parse Commandline
  for (int i = 1; i < argc; ++i) {

    char* pos = strtok(argv[i], "=");
    std::pair < std::string, std::string > p(std::string(pos), std::string(""));
    pos = strtok(NULL, "=");
    if (pos != NULL) {
      p.second = std::string(pos);
    }
    parameters.push_back(p);
  }
  for (unsigned i = 0; i < parameters.size(); ++i) {
    std::pair < std::string, std::string > p = parameters.at(i);
    fprintf(stderr, "arg[%d]: (%s,%s)\n", i, p.first.c_str(), p.second.c_str());

    // Parse number of rays
    if (p.first == "--rays") {
      *raysPerSample = atoi(p.second.c_str());
    }

    if (p.first == "--experiment") {
      std::string temp_root(p.second);

      // Add slash at the end, if missing
      if ((temp_root)[temp_root.size() - 1] == 'w')
        temp_root.erase(temp_root.size() - 1, 1);
      else if (temp_root[temp_root.size() - 1] != '/')
        temp_root.append("/");

      *root = temp_root;
    }

    // Parse which cuda device to choose
    if (p.first == "--device") {
      *device = atoi(p.second.c_str());
    }

    // Parse if we want less output
    if (p.first == "--silent") {
      *silent = true;
    }

    // Parse what vtk file to compare with
    if (p.first == "--compare") {
      *compareLocation = p.second;
    }

    if (p.first == "--mode") {
      if (p.second == "ray_propagation_gpu")
        *mode = 0;
      if (p.second == "for_loops")
        *mode = 1;
    }
  }
}

int checkParameterValidity(
    int argc,
    unsigned raysPerSample,
    std::string root,
    int *device,
    int mode
    ) {

  if (argc <= 1) {
    fprintf(stderr, "C No commandline arguments found\n");
    fprintf(stderr, "C Usage    : ./octrace --mode=[runmode] --rays=[number of rays] --experiment=[location to experiment-data]\n");
    fprintf(stderr, "C Runmodes : for_loops\n");
    fprintf(stderr, "             ray_propagation_gpu\n");
    return 1;
  }
  if (mode == -1) {
    fprintf(stderr, "C Please specify the runmode with --mode=\n");
    return 1;
  }
  if (raysPerSample == 0) {
    fprintf(stderr, "C Please specify the number of rays per sample Point with --rays=\n");
    return 1;
  }
  if (root.size() == 0) {
    fprintf(stderr, "C Please specify the experiment's location with --experiment=\n");
    return 1;
  }
  if (*device == -1) {
    *device = 0;
  }


  return 0;
}
