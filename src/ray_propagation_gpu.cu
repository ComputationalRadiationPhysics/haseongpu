#include "hip/hip_runtime.h"
//#include "ray_propagation_gpu.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_vector_types.h"
#include "assert.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>

#define TEST_VALUES true
#define USE_IMPORTANCE true
#define SMALL 1E-06
#define VERY_SMALL 0.0

#define CUDA_CHECK_RETURN(value) {					\
    hipError_t _m_cudaStat = value;					\
    if (_m_cudaStat != hipSuccess) {					\
      fprintf(stderr, "Error %s at line %d in file %s\n",		\
	      hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);	\
      exit(1);								\
    }									\
  }
#define CUDA_CALL(x) do { if((x) != hipSuccess) {	\
      printf("Error at %s:%d\n",__FILE__,__LINE__);	\
      return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) {	\
      printf("Error at %s:%d\n",__FILE__,__LINE__);		\
      return EXIT_FAILURE;}} while(0)

__device__ double cladAbsorption;
__device__ double nTot;
__device__ double sigmaE;
__device__ double sigmaA;
__device__ double thicknessOfPrism;
__device__ int numberOfLevels;
__device__ int cladNumber;
__device__ int numberOfPoints;
__device__ int numberOfTriangles;

/**
 * @brief Propagate a ray between 2 points and calculate the resulting ASE-Flux at the Destination
 *
 * @params x_pos		the x-coordinate where the ray starts
 *         y_pos		the y-coordinate where the ray starts
 *         z_pos		the z-coordinate where the ray starts
 *         x_dest		the destination of the ray (x-coordinate)
 *         y_dest		the destination of the ray (y-coordinate)
 *         z_dest		the destination of the ray (z-coordinate)
 *         t_start		the index of the triangle, in which the ray starts
 *         mesh_start	the level of the mesh (the slice) in which the ray starts
 *		   p_in			coordinates of the sample-points of one layer (first all x-coordinates, then all y-coordinates)
 *		   n_x			x-coordinates for the normal-vectors for the 3 rectangular sides of each prism
 *		   n_y			y-coordinates for the normal-vectors for the 3 rectangular sides of each prism
 *		   n_p			indices of the points where the normal-vectors start	
 *		   neighbors	indices of the adjacent triangles	
 *		   forbidden	sides of the new triangles which are forbidden, after coming from an adjacent triangle
 *		   cell_type	contains the material-constant for each cell/prism
 *		   beta_v		contains the beta-values for each cell/prism
 *
 */
__device__ double rayPropagationGpu(
				    double xPos,
				    double yPos,
				    double zPos,
				    double xDestination,
				    double yDestination,
				    double zDestination,
				    int firstTriangle,
				    int firstLevel,
				    double *points,
				    double *xOfNormals,
				    double *yOfNormals,
				    int *positionsOfNormalVectors,
				    int *neighbors,
				    int *forbidden,
				    int* cellTypes,
				    double* betaValues){
  //    no reflections
  //
  //    create the vector between both points and calculate which surface would be the shortest to reach.
  //    then get the length, make the integration, get the information about the next cell out of the array
  //    set the point to the surface (this surface is "forbidden" in the calculations)
  //    proceed until you hit a the point or the surface
	
  double xVec, yVec,zVec;
  double distanceRemaining, length, lengthHelp, distanceTotal;
  double nominator, denominator;
  double gain=1.;
  int triangleCurrent, levelCurrent; // the current triangle number and position concerning the z's
  int triangleNext, levelNext, forbiddenCurrent, forbiddenNext;
  int offset;
#if TEST_VALUES==true
  double testDistance = 0;
  int loopbreaker = 0;
#endif


  //    initial positions
  triangleCurrent = firstTriangle;
  levelCurrent = firstLevel;

  // direction-vector (without reflections)
  xVec = (xDestination - xPos);
  yVec = (yDestination - yPos);
  zVec = (zDestination - zPos);

  // total distance to travel
  distanceTotal = sqrt(xVec*xVec+yVec*yVec+zVec*zVec);
  // normalized direction-vector
  xVec = xVec/distanceTotal;
  yVec = yVec/distanceTotal;
  zVec = zVec/distanceTotal;

  // remaining distance to travel
  distanceRemaining = distanceTotal;

  // at the beginning, all surfaces are possible
  forbiddenCurrent = -1;

  for(;;)
    {
      // the length of the ray-part inside the current prism. We try to minimize this value
      length = distanceRemaining;
      lengthHelp=0;
      //        definition for decider
      //        0,1,2: int for the neighbors
      //        3: hor plane up
      //        4: hor plane down
      //        try the triangle faces
      //        remember the correlation between the normals and the points
      //        n1: p1-2, n2: p1-3, n3:p2-3
      //        the third coordinate (z) of the particpating points for the surfaces can be set to be z=0, 
      //        as everything uses triangular "prisms", as well as n_z=0 in this case!
		
      // forb describes the surface, from which the ray enters the prism.
      // this surface is no suitable candidate, since the length would be 0!
      if (forbiddenCurrent != 0){
	denominator = xOfNormals[triangleCurrent]*xVec + yOfNormals[triangleCurrent]*yVec;
	// see if we intersect at all
	if (denominator != 0.0)
	  {
	    nominator = (xOfNormals[triangleCurrent]*points[positionsOfNormalVectors[triangleCurrent]] + yOfNormals[triangleCurrent]*points[positionsOfNormalVectors[triangleCurrent]+ numberOfPoints]) - (xOfNormals[triangleCurrent]*xPos + yOfNormals[triangleCurrent]*yPos);
	    lengthHelp = nominator/denominator;
	    // if we found a new smallest length, use it
	    if (lengthHelp < length && lengthHelp > 0.0)
	      {
		length = lengthHelp;
		forbiddenNext = (forbidden[triangleCurrent]);
		triangleNext = neighbors[triangleCurrent];
		levelNext = levelCurrent;

	      }
	  }
      }

      // see forbiddenCurrent !=0 case
      if (forbiddenCurrent != 1){
	//offset, since the 3 rectangular surfaces are stored at different positions in the array
	offset = triangleCurrent+numberOfTriangles;
	denominator = xOfNormals[offset]*xVec + yOfNormals[offset]*yVec;
	if (denominator != 0.0)
	  {
	    nominator = (xOfNormals[offset]*points[positionsOfNormalVectors[offset]] + yOfNormals[offset]*points[positionsOfNormalVectors[offset]+ numberOfPoints]) - (xOfNormals[offset]*xPos + yOfNormals[offset]*yPos);
	    lengthHelp = nominator/denominator;
	    if (lengthHelp < length && lengthHelp > 0.0)
	      {
		length = lengthHelp;
		forbiddenNext = (forbidden[offset]);
		triangleNext = neighbors[offset];
		levelNext = levelCurrent;
	      }
	  }
      }

      // see forbiddenCurrent !=0 case
      if (forbiddenCurrent !=2){
	offset = triangleCurrent+2*numberOfTriangles;
	denominator = xOfNormals[offset]*xVec + yOfNormals[offset]*yVec;
	if (denominator != 0.0)
	  {
	    nominator = (xOfNormals[offset]*points[positionsOfNormalVectors[offset]] + yOfNormals[offset]*points[positionsOfNormalVectors[offset]+ numberOfPoints]) - (xOfNormals[offset]*xPos + yOfNormals[offset]*yPos);
	    lengthHelp = nominator/denominator;
	    if (lengthHelp < length && lengthHelp > 0.0)
	      {
		length = lengthHelp;
		forbiddenNext = (forbidden[offset]);
		triangleNext = neighbors[offset];
		levelNext = levelCurrent;
	      }
	  }
      }

      // if-structure "optimized"
      denominator = zPos*zVec;
      if (denominator != 0.0){
	if (forbiddenCurrent != 3){
	  {
	    nominator = (levelCurrent+1)* thicknessOfPrism - zPos;
	    lengthHelp = nominator/denominator;
	    if (lengthHelp < length && lengthHelp > 0.0)
	      {
		length = lengthHelp;
		//decider = 3;
		forbiddenNext = 4; // you are not allowed to go down in the next step
		triangleNext = triangleCurrent;
		levelNext = levelCurrent + 1;
	      }
	  }
	}

	// next is the lower plane
	if (forbiddenCurrent != 4){
	  nominator = (levelCurrent)* thicknessOfPrism - zPos;
	  lengthHelp = nominator/denominator;
	  if (lengthHelp < length && lengthHelp > 0.0)
	    {
	      length = lengthHelp;
	      //decider = 4;
	      forbiddenNext = 3; // you are not allowed to go up in the next step
	      triangleNext = triangleCurrent;
	      levelNext = levelCurrent - 1;
	    }
	}
      }

      if (cellTypes[triangleCurrent] == cladNumber){
	gain *= exp((-1)*(cladAbsorption * length));
      }
      else {
	gain *= (double) exp(nTot * (betaValues[triangleCurrent+levelCurrent*numberOfTriangles]*(sigmaE + sigmaA)-sigmaA)*length);
      }


      // the remaining distance is decreased by the length we travelled through the prism
      distanceRemaining -= length;
		

#if TEST_VALUES==true
      testDistance += length;
      if(loopbreaker>500){
	printf("Loopbreaker reached. firstTriangle: %d, level: %d, length: %f, distanceTotal:%f, testDistance%f, distanceRemaining:%f\n",firstTriangle,firstLevel,length,distanceTotal,testDistance,distanceRemaining);
	return 0.;
      }else{
	loopbreaker++;
      }
#endif
      // if the distance between the destination and our current position is small enough, we are done
      if (fabs(distanceRemaining) < SMALL){
	break;
      }

      // now set the next cell and position
      xPos = xPos + length*xVec;
      yPos = yPos + length*yVec;
      zPos = zPos + length*zVec;

      triangleCurrent = triangleNext;
      levelCurrent = levelNext;
      // set the new forbidden surface
      forbiddenCurrent = forbiddenNext;

    }

#if TEST_VALUES==true
  if(fabs(distanceTotal-testDistance) > SMALL)
    printf("Distance too big! firstTriangle: %d, level: %d, length: %f, distanceTotal:%f, testDistance%f, distanceRemaining:%f\n",firstTriangle,firstLevel,length,distanceTotal,testDistance,distanceRemaining);
#endif
	
  return gain /= (distanceTotal*distanceTotal);
}



/**
 * Initializes the global variables of the GPU with the correct values.
 * All those values are from the original propagation-function which we ported.
 */
__global__ void setupGlobalVariablesKernel ( 
					    double hostSigmaE,
					    double hostSigmaA, 
					    int hostCladNum, 
					    double hostCladAbs, 
					    double hostNTot, 
					    int hostNumberOfTriangles, 
					    double hostThicknessOfPrism, 
					    int hostNumberOfLevels, 
					    int hostNumberOfPoints )
{
  sigmaE = hostSigmaE;	
  sigmaA = hostSigmaA;
  cladNumber = hostCladNum;
  cladAbsorption = hostCladAbs;
  nTot = hostNTot;
  numberOfTriangles = hostNumberOfTriangles;
  thicknessOfPrism = hostThicknessOfPrism;
  numberOfLevels = hostNumberOfLevels;
  numberOfPoints = hostNumberOfPoints;
  //printf("Sigma_e in setup=%f\tSigma_eHost=%f\n",sigma_e,host_sigma_e);
} 

// __global__ void importanceKernel(
// 		hiprandState *globalState,
// 		double *points,
// 		double *xOfNormals,
// 		double *yOfNormals,
// 		int *positionsOfNormalVectors,
// 		int *neighbors,
// 		int *forbidden,
// 		int* cell_type,
// 		int hostNumberOfTriangles,
// 		double* betaValues,
// 		double *importance,
// 		int *numberOfImportantRays,
// 		double *xOfTriangleCenter,
// 		double *yOfTriangleCenter,
// 		int *surface,
// 		int totalNumberOfRays) {

// 	int id = threadIdx.x + blockIdx.x * blockDim.x;
// 	for(int i=0; i< hostNumberOfTriangles; ++i){
// 		for(int j=0; j< numberOfLevels; ++j){
// 			importf(globalState[id], i,j, importance, numberOfImportantRays, points, xOfNormals, yOfNormals, positionsOfNormalVectors, neighbors, forbidden, cell_type, betaValues, xOfTriangleCenter, yOfTriangleCenter,surface, totalNumberOfRays);

// 		}

// 	}

// }


/**
 * Prints some of the global device variables.
 * Is only used for testing
 */
__global__ void testKernel (
			    double *points,
			    double *xOfNormals,
			    double *yOfNormals,
			    int *positionsOfNormalVectors,
			    int *neighbors,
			    int *forbidden,
			    int* triangleIndices,
			    int* cellTypes,
			    double* betaValues,
			    int *surfacesNormalized){
  printf("\nSigmaE=%.6e",sigmaE);
  printf("\nSigmaA=%.6e",sigmaA);
  printf("\nNumberOfLevels=%d",numberOfLevels);
  printf("\nNumberOfPoints=%d",numberOfPoints);
  printf("\nthicknessOfPrism_=%.6e",thicknessOfPrism);
  printf("\nnumberOfTriangles=%d",numberOfTriangles);
  printf("\nnTot=%.6e",nTot);
  printf("\ncladAbsorption=%.6e",cladAbsorption);
  printf("\ncladNumber=%d\n\n",cladNumber);

  unsigned limit = 5;
  for(int i=0;i<limit;++i){
    printf("points[%d]: %e\n",i,points[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("xOfNormals[%d]: %e\n",i,xOfNormals[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("yOfNormals[%d]: %e\n",i,yOfNormals[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("positionsOfNormalVectors[%d]: %d\n",i,positionsOfNormalVectors[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("neighbors[%d]: %d\n",i,neighbors[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("forbidden[%d]: %d\n",i,forbidden[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("triangleIndices[%d]: %d\n",i,triangleIndices[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("cellTypes[%d]: %d\n",i,cellTypes[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("betaValues[%d]: %e\n",i,betaValues[i]);
  }
  printf("\n\n");

  for(int i=0;i<limit;++i){
    printf("surfacesNormalized[%d]: %d\n",i,surfacesNormalized[i]);
  }
  printf("\n\n");
} 


/**
 * Does the raytracing for a single Sample point (in a defined level).
 * This Kernel has to be started for each sample point with the same value for iterations
 * and the same number of blocks/threads.
 *
 * \var globalState the state of the mersenneTwister PRNG
 * 		(has a maximum of 200 positions!)
 * \var phi points to a memory region which is initialized with 0
 * 		(can hold one value for each sample point)
 * \var point2D the index of the current sample point (points to p_in)
 * \var level the level of the current sample point (how deep we are through the material)
 * \var raysPerThread the number rays which are computed by this thread
 * 		(always for the same combination of startprism+samplepoint
 */
__global__ void raytraceStep(
			     hiprandStateMtgp32* globalState,
			     float* phiASE,
			     const int point2D,
			     const int level,
			     const int raysPerThread,
			     double *points,
			     double *xOfNormals,
			     double *yOfNormals,
			     int *positionsOfNormalVectors,
			     int *neighbors,
			     int *forbidden,
			     int* triangleIndices,
			     int* cellTypes,
			     double* betaValues,
			     double* importance,
			     int* numberOfImportantRays,
			     int* surfacesNormalized) {

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned numberOfPrisms = (numberOfTriangles * (numberOfLevels-1));
  const unsigned threadsPerPrism = blockDim.x * gridDim.x / numberOfPrisms;
  // break, if we have more threads than we need
  if(id >= threadsPerPrism * numberOfPrisms)
    return;

  double gain = 0.;
  const int endPointX = points[point2D];
  const int endPointY = points[ numberOfPoints + point2D];
  const int endPointZ = level* thicknessOfPrism;


  // this should give the same start values multiple times (so that every thread uses the same prism, which yields
  // big benefits for the memory access (and caching!)
  unsigned startPrism = id % numberOfPrisms;
  int startLevel = (startPrism)/numberOfTriangles;
  int startTriangle = (startPrism-(numberOfTriangles*startLevel));

#if TEST_VALUES==true
  if(startPrism != (startTriangle+(startLevel*numberOfTriangles))){
    printf("StartTriangle/StartLevel incorrect!");
  }
  if(startTriangle >= 600){
	
    printf("StartTriangle/StartLevel incorrect!");
  }
  //if(startPrism == 5399){
  //	printf("startprism: %d, id=%d, threadsPerPrism=%d\n",startPrism,id,threadsPerPrism);
  //}
#endif

  // the indices of the vertices of the starttriangle
  int t1 = triangleIndices[startTriangle];
  int t2 = triangleIndices[startTriangle+ numberOfTriangles];
  int t3 = triangleIndices[startTriangle+2*numberOfTriangles];

  // do all this multiple times (we can't have more than 200 blocks due to restrictions of the Mersenne Twister)
  for (int i=0; i < numberOfImportantRays[startPrism]; ++i){
    //for (int i=0; i<blah; ++i){
    // random startpoint generation
    double u = hiprand_uniform(&globalState[blockIdx.x]);
    double v = hiprand_uniform(&globalState[blockIdx.x]);

    if((u+v)>1)
      {
	u = 1-u;
	v = 1-v;
      }
    double w = 1-u-v;

    // convert the random startpoint into coordinates
    double zRand = (startLevel + hiprand_uniform(&globalState[blockIdx.x]))* thicknessOfPrism;
    double xRand = points[t1]*u + points[t2]*v + points[t3]*w;
    double yRand = points[ numberOfPoints + t1]*u + points[ numberOfPoints + t2]*v + points[ numberOfPoints + t3]*w;

    __syncthreads();
    gain += rayPropagationGpu(xRand, yRand, zRand, endPointX, endPointY, endPointZ, 
			      startTriangle, startLevel ,points, xOfNormals, yOfNormals, 
			      positionsOfNormalVectors, neighbors, forbidden , cellTypes, betaValues);
    //gain += double(propagationOld(xRand, yRand, zRand, endPointX, endPointY, endPointZ, startTriangle, startLevel ,points, xOfNormals, yOfNormals, positionsOfNormalVectors, neighbors, forbidden , cellTypes, betaValues));
  }
	

  // do the multiplication just at the end of all raysPerThread
  // (gives better numeric behaviour)
  gain *= betaValues[startPrism];///surfacesNormalized[startTriangle];
#if USE_IMPORTANCE==true
  atomicAdd(&(phiASE[point2D + level*numberOfPoints]), float(gain * importance[startPrism]));
#else
  atomicAdd(&(phiASE[point2D + level*numberOfPoints]),float(gain));
#endif
  return;
}

/*********************************************************************************************
 * HOST FUNCTIONS
 *********************************************************************************************/
double rayPropagationCpu(double x_pos, 
			 double y_pos, 
			 double z_pos, 
			 double x_dest, 
			 double y_dest, 
			 double z_dest, 
			 int t_start, 
			 int mesh_start, 
			 double *p_in,
			 double *n_x,
			 double *n_y,
			 int *n_p,
			 int *neighbors,
			 int *forbidden,
			 unsigned *cell_type,
			 double *beta_v,
			 unsigned numberOfPoints,
			 unsigned numberOfTriangles,
			 float thicknessOfPrism,
			 float sigmaA,
			 float sigmaE,
			 int cladNumber,
			 float cladAbsorption,
			 float nTot
			 ){
  //    in first try no reflections
  //    calculate the vector and make the the calculation, which surface would be the shortest to reach
  //    then get the length, make the integration, get the information about the next cell out of the array
  //    set the point to the surface (this surface is "forbidden" in the calculations)
  //    proceed until you hit a the point or the surface
  //    if you are closer then "small" stop and return the value
  double vec_x, vec_y,vec_z, norm;
  double distance, length, length_help, distance_total;
  double gain=1;
  double nominator, denominator;
  int tri, cell_z; // the current triangle number and position concerning the z's
  int decider; // which one is the shortest - info
  int tri_next, cell_z_next, forb, forb_dump;
  int ct; // used to read out cell_type 
  unsigned size_p = numberOfPoints;
  unsigned N_cells = numberOfTriangles;
  float z_mesh = thicknessOfPrism;
  float sigma_a = sigmaA;
  float sigma_e = sigmaE;
  int clad_num = cladNumber;
  float clad_abs = cladAbsorption;
  float N_tot = nTot;

	
  //    initial positions
  tri = t_start;
  cell_z = mesh_start;
    
    
  //    definition of the vectors without reflections
  vec_x = (x_dest - x_pos);
  vec_y = (y_dest - y_pos);
  vec_z = (z_dest - z_pos);
    
  norm = sqrt(vec_x*vec_x+vec_y*vec_y+vec_z*vec_z);
    
  vec_x = vec_x/norm;
  vec_y = vec_y/norm;
  vec_z = vec_z/norm;
    
  //    now calculate the length to travel
  distance = sqrt((x_dest - x_pos)*(x_dest - x_pos)+(y_dest - y_pos)*(y_dest - y_pos)+(z_dest - z_pos)*(z_dest - z_pos));
  distance_total = distance;
  // does this make sense?
  length = distance;
    
  forb = -1;
	
  //	mexPrintf("Propagation called");
  //    mexEvalString("drawnow;");
    
  //    the ray has to be set to be ALIVE before!
  //    now do the unlimited for loop - break!!!
  for(;;)
    {
	
      //	  mexPrintf("Propagation for part called\n\n");
      //    mexEvalString("drawnow;");
      //        definition for decider
      //        0,1,2: int for the neighbors
      //        3: hor plane up
      //        4: hor plane down
        
      //        at first set the decider = -1;
      decider = -1;
      length = distance;
		
		
      //		  read, which type of cell it is you are propagation in
      ct = cell_type[tri];
        
      //        mexPrintf("forb: %i\n",forb);
      //        mexEvalString("drawnow;");
        
      //        try the triangle faces
      //        remember the correlation between the normals and the points
      //        n1: p1-2, n2: p1-3, n3:p2-3
      //        the third coordinate (z) of the particpating points for the surfaces can be set to be z=0, 
      //        as everything uses triangular "tubes/prisms", as well as n_z=0 in this case!
      if (forb != 0){
	nominator = (n_x[tri]*p_in[n_p[tri]] + n_y[tri]*p_in[n_p[tri]+size_p]) - (n_x[tri]*x_pos + n_y[tri]*y_pos);
	denominator = n_x[tri]*vec_x + n_y[tri]*vec_y;
	if (denominator != 0.0)
	  {
	    length_help = nominator/denominator;
	    if (length_help < length && length_help > VERY_SMALL)
	      {
		length = length_help;
		decider = 0;
		forb_dump = (forbidden[tri]);
	      }
	  }
      }
        
      if (forb != 1){
	nominator = (n_x[tri+N_cells]*p_in[n_p[tri+N_cells]] + n_y[tri+N_cells]*p_in[n_p[tri+N_cells]+size_p]) - (n_x[tri+N_cells]*x_pos + n_y[tri+N_cells]*y_pos);
	denominator = n_x[tri+N_cells]*vec_x + n_y[tri+N_cells]*vec_y;
	if (denominator != 0.0)
	  {
	    length_help = nominator/denominator;
	    if (length_help < length && length_help > VERY_SMALL)
	      {
		length = length_help;
		decider = 1;
		forb_dump = (forbidden[tri+N_cells]);
	      }
	  }
      }
        
      if (forb !=2){
	nominator = (n_x[tri+2*N_cells]*p_in[n_p[tri+2*N_cells]] + n_y[tri+2*N_cells]*p_in[n_p[tri+2*N_cells]+size_p]) - (n_x[tri+2*N_cells]*x_pos + n_y[tri+2*N_cells]*y_pos);
	denominator = n_x[tri+2*N_cells]*vec_x + n_y[tri+2*N_cells]*vec_y;
	if (denominator != 0.0)
	  {
	    length_help = nominator/denominator;
	    if (length_help < length && length_help > VERY_SMALL)
	      {
		length = length_help;
		decider = 2;
		forb_dump = (forbidden[tri+2*N_cells]);
	      }
	  }
      }
        
      //        try the horizontal planes, which one is the shortest, n_x and n_y are zero!, n_z =1!
      //        at first the upper plane
      if (forb != 3){
	nominator = (cell_z+1)*z_mesh - z_pos;
	denominator = z_pos*vec_z;
	if (denominator != 0.0)
	  {
	    length_help = nominator/denominator;
	    if (length_help < length && length_help > VERY_SMALL)
	      {
		length = length_help;
		decider = 3;
		forb_dump = 4; // you are not allowed to go down in the next step
	      }
	  }
      }
        
      //        next is the lower plane
      if (forb != 4){
	nominator = (cell_z)*z_mesh - z_pos;
	denominator = z_pos*vec_z;
            
	if (denominator != 0.0)
	  {
	    length_help = nominator/denominator;
	    if (length_help < length && length_help > VERY_SMALL)
	      {
		length = length_help;
		decider = 4;
		forb_dump = 3; // you are not allowed to go up in the next step
	      }
	  }
      }
        
      forb = forb_dump;
		
        
      //        now make a switch to differ the different cases
      switch(decider){
                
      case 0:
	//                this is the case for the intersection with the first choice triangle-surface
	tri_next = neighbors[tri];
	cell_z_next = cell_z;
	break;
                
      case 1:
	//                second triangle surface
	tri_next = neighbors[tri+N_cells];
	cell_z_next = cell_z;
	break;
                
      case 2:
	//                third triangle surface
	tri_next = neighbors[tri+2*N_cells];
	cell_z_next = cell_z;
	break;
                
      case 3:
	//                go one plane up
	tri_next = tri;
	cell_z_next = cell_z + 1;
	break;
                
      case 4:
	//                go one plane down
	tri_next = tri;
	cell_z_next = cell_z - 1;
	break;
                
      default:
	//                make an error statement
	break;
      }
        
      //        now we know where to go, let's make the integration
      //        take the beta_v[tri+cell_z*N_cells] 

      //		  at this position do the decision whether it is a gain part or cladding
      //		  it might be absorbing or amplifying, for the cladding only absorbing
      //		  a simple "if then"

      if (ct == clad_num){
	gain = gain * exp(-clad_abs * length);
      }
      else {
	gain = gain * exp(N_tot*(beta_v[tri+cell_z*N_cells]*(sigma_e + sigma_a)-sigma_a)*length);
      }
      //        gain = LineIntegralMCRK4_S(3, tri, cell_z, gain, length);
        
      //        after integration make the propagation
        
      //        mexPrintf("Distance: %f, Length: %f\n",distance, length);
      //        mexPrintf("decider: %i, forbidden: %i\n",decider, forb);
      //        mexPrintf("vec_x: %f, vec_y: %f, vec_z: %f\n", vec_x, vec_y, vec_z);
      //        mexPrintf("current_x: %f current_y: %f current_z: %f\n", x_pos, y_pos, z_pos);
      //        mexPrintf("tri: %i, tri_next: %i, cell_z: %i, cell_next: %i\n", tri, tri_next, cell_z, cell_z_next);
      //        mexEvalString("drawnow;");
      //        str=mxCreateString("Press a key");
      //        mexCallMATLAB(1,&dump,1,&str,"input"); 
      //        str and dump should be defined to be a *mxArray and don't forget to kill them at the end
        
      distance -= length;
        
      //        return 1;
      //        
        
      x_pos = x_pos + length*vec_x;
      y_pos = y_pos + length*vec_y;
      z_pos = z_pos + length*vec_z;
        
      if (abs(distance)< SMALL)
        {
	  break;
        }
        
        
      //        now set the next cell
      tri = tri_next;
      cell_z = cell_z_next;      
        
      //        break;
      //        now we should make the integration routine
    }
    
  gain /= (distance_total*distance_total);

  return gain;
}



/**
 * calculate the gain from the centers of each of the boxes to the observed point
 * calculate the gain and make a "mapping"
 * receipt: pick the point in the center of one cell, 
 * calculate the gain from this point to the observed point,
 * estimate the inner part of the Phi_ASE - Integral,
 * scale the amount of rays proportionally with it
 * sum the amount of rays and scale it to Int=1, which gives the inverse weights
 * the number of rays is determined via floor(), with ceil(), zero-redions could be added
 * use the routine "propagation"!, test: no reflections, just exponential
 *
 **/
void importf(int point,
	     int startLevel,
	     double *importance,
	     int *numberOfImportantRays,
	     double *points,
	     double *xOfNormals,
	     double *yOfNormals,
	     int *positionsOfNormalVectors,
	     int *neighbors,
	     int *forbidden,
	     unsigned *cellTypes,
	     double *betaValues,
	     double *xOfTriangleCenter,
	     double *yOfTriangleCenter,
	     float *surface,
	     unsigned raysPerSample,
	     unsigned numberOfPoints,
	     unsigned numberOfLevels,
	     unsigned numberOfTriangles,
	     float thicknessOfPrism,
	     float sigmaA,
	     float sigmaE,
	     int cladNumber,
	     float cladAbsorption,
	     float nTot
	     )
{
  int raysLeft;
  int raysDump;
  double sumPhi;
  double surfaceTotal;
  double xPos, yPos, zPos;
  double prop;

  raysDump = 0;
  sumPhi = 0.0;
  surfaceTotal = 0.0;
  xPos = points[point];
  yPos = points[point + numberOfPoints];
  zPos = startLevel * thicknessOfPrism;

  // Calculate importance by propagation from trianglecenter to every other center
  for (int i_t=0; i_t < numberOfTriangles; ++i_t){
    for (int i_z=0; i_z < (numberOfLevels-1); ++i_z){
      prop = rayPropagationCpu(xOfTriangleCenter[i_t], yOfTriangleCenter[i_t], 
      			       thicknessOfPrism * (i_z+0.5),  xPos, yPos, zPos, i_t, i_z, 
      			       points, xOfNormals, yOfNormals, positionsOfNormalVectors, 
      			       neighbors, forbidden , cellTypes, betaValues,
      			       numberOfPoints, numberOfTriangles, thicknessOfPrism,
      			       sigmaA, sigmaE, cladNumber, cladAbsorption, nTot
      			       );

      importance[i_t + i_z * numberOfTriangles] = betaValues[i_t + i_z * numberOfTriangles]*(prop);
      sumPhi += importance[i_t + i_z * numberOfTriangles];

    }
    surfaceTotal += surface[i_t];

  }

  // Calculate number of rays/prism
  for (int i_t=0; i_t < numberOfTriangles; ++i_t){
    for (int i_z=0; i_z < (numberOfLevels-1); ++i_z){
      numberOfImportantRays[i_t + i_z*numberOfTriangles] = (int)(floor(importance[i_t + i_z * numberOfTriangles] / sumPhi * raysPerSample));
      raysDump +=  numberOfImportantRays[i_t + i_z*numberOfTriangles];
      //fprintf(stderr, "[%d][%d] i: %.20f n: %d\n", i_z, i_t, importance[i_t + i_z*numberOfTriangles], numberOfImportantRays[i_t + i_z*numberOfTriangles]);
    }

  }
  raysLeft = raysPerSample - raysDump;

  // Distribute the remaining rays randomly
  for (int i_r=0; i_r < raysLeft; i_r++){
    int rand_t = (int )(rand() % numberOfTriangles);
    int rand_z = (int )(rand() % (numberOfLevels-1));
    numberOfImportantRays[rand_t + rand_z * numberOfTriangles]++;

  }

  //  Now think about the mount of rays which would come out of this volume(surface)
  //  dividing this number with the new amount of rays gives the final importance weight for this area!
  for (int i_t=0; i_t<numberOfTriangles; ++i_t){
    for (int i_z=0; i_z<(numberOfLevels-1); ++i_z){
      if (numberOfImportantRays[i_t + i_z*numberOfTriangles] > 0){
  	importance[i_t + i_z*numberOfTriangles] = raysPerSample * surface[i_t] / surfaceTotal / numberOfImportantRays[i_t + i_z*numberOfTriangles];

      }
      else{
  	importance[i_t + i_z*numberOfTriangles] = 0; 

      }


    }

  }


}


double* doubleVectorToArray(std::vector<double> *input){
  double* output;
  output = (double*) malloc(sizeof(double) * input->size());
  for(int i=0; i< input->size(); ++i){
    output[i] = input->at(i);	
  }
  return output;
}
int* intVectorToArray(std::vector<int> *input){
  int* output;
  output = (int*) malloc(sizeof(int) * input->size());
  for(int i=0; i< input->size(); ++i){
    output[i] = input->at(i);	
  }
  return output;
}
unsigned* unsignedVectorToArray(std::vector<unsigned> *input){
  unsigned* output;
  output = (unsigned*) malloc(sizeof(unsigned) * input->size());
  for(int i=0; i< input->size(); ++i){
    output[i] = input->at(i);	
  }
  return output;
}

//----------------------------------------------------
// Host Code
//----------------------------------------------------
/** GPU Kernel Variables
 * The idea is, that the number of threads is fixed (to maximize GPU occupancy)
 * and the number of blocks as well (200 is the maximum for the standard
 * Mersenne Twister implementaion). Therefore, the number of rays per sample
 * are fixed to be k*200*256.
 * That means, sometimes we have to increase the number of rays a little.
 *
 * \var raysPerThread is used to give every thread k iterations (to simulate k rays)
 *
 * note that every samplepoint receives the exact same number of rays.
 */
	
//

/** Variables for the device
 * These are on-GPU representations of the input parameters
 * of variable size.
 *
 * \var p_in: coordinates of the sample-points of one layer (first all x-coordinates, then all y-coordinates)
 * \var n_*: values of the normal-vectors for the 3 rectangular sides of each prism (described in 2D)
 * \var beta_v: the beta values of the prisms
 * \var phi: the accumulated ASE-Flux for each sample point
 * \var forbidden: the side of the prism through which the ray "entered" the prism
 * \var n_p: the points where the normals (n_x,n_y) start
 * \var neighbors: indices to the adjacent triangles in t_in
 * \var t_in: indices of the points which are considered to be a triangle (A points start from 0, B points from size_t, C points from size_t*2)
 * \var cell_type: determines which cell type we are looking at.
 * other input parameters are put to the GPU by the setupGlobalVariablesKernel
 */

float runRayPropagationGpu(
			   std::vector<double> *dndtAse, 
			   unsigned &threads, 
			   unsigned &blocks, 
			   unsigned &hostRaysPerSample,
			   std::vector<double> *betaValuesVector,
			   std::vector<double> *xOfNormalsVector,
			   std::vector<double> *yOfNormalsVector,
			   std::vector<unsigned> *cellTypesVector,
			   std::vector<unsigned> *triangleIndicesVector,
			   std::vector<int> *forbiddenVector,
			   std::vector<int> *neighborsVector,
			   std::vector<int> *positionsOfNormalVectorsVector,
			   std::vector<double> *pointsVector,
			   std::vector<double> *betaCellsVector,
			   std::vector<float> *surfacesVector,
			   std::vector<double> *xOfTriangleCenterVector,
			   std::vector<double> *yOfTriangleCenterVector,
			   float hostCladAbsorption,
			   int hostCladNumber,
			   float hostNTot,
			   float hostSigmaA,
			   float hostSigmaE,
			   unsigned hostNumberOfPoints,
			   unsigned hostNumberOfTriangles,
			   unsigned hostNumberOfLevels,
			   float hostThicknessOfPrism,
			   float hostCrystalFluorescence)
{
  fprintf(stderr, "\nConverting Vectors to Arrays\n");

  // Variable declarations
  // CPU
  // INPUT
  double* hostBetaValues;
  double* hostXOfNormals;
  double* hostYOfNormals;
  double* hostXOfTriangleCenter;
  double* hostYOfTriangleCenter;
  unsigned* hostCellTypes;
  unsigned* hostTriangleIndices;
  int* hostForbidden;
  int* hostNeighbors;
  int* hostPositionsOfNormalVectors;
  double* hostPoints;
  float* hostSurfaces;
  // TMP CALC
  unsigned hostNumberOfPrisms;
  unsigned hostRaysPerThread;
  hipEvent_t start, stop;
  float runtimeGpu;
  float hostPhiASE[hostNumberOfPoints * (hostNumberOfLevels)];
  unsigned kernelcount;
  float surfaceTotal;
  int hostSurfacesNormalized[hostNumberOfTriangles];
  float minSurface;
  int surfacesNormalizedSum;
  // GPU
  double  *points, *xOfNormals, *yOfNormals, *betaValues;
  float *phiASE;
  int *forbidden, *positionsOfNormalVectors, *neighbors, *triangleIndices, *cellTypes, *surfacesNormalized;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  int *numberOfImportantRays;
  double *importance;
	
  // Variables defintions
  threads = 120;
  blocks = 90;  // TODO: change number of blocks/threads dynamically to allow more flexible number of rays (increase up to 200)
  hostBetaValues = doubleVectorToArray(betaValuesVector);
  hostXOfNormals = doubleVectorToArray(xOfNormalsVector);
  hostYOfNormals = doubleVectorToArray(yOfNormalsVector);
  hostXOfTriangleCenter = doubleVectorToArray(xOfTriangleCenterVector);
  hostYOfTriangleCenter = doubleVectorToArray(yOfTriangleCenterVector);
  hostCellTypes = unsignedVectorToArray(cellTypesVector);
  hostTriangleIndices = unsignedVectorToArray(triangleIndicesVector);
  hostForbidden = intVectorToArray(forbiddenVector);
  hostNeighbors = intVectorToArray(neighborsVector);
  hostSurfaces =  (float*) &(surfacesVector[0]);
  hostPositionsOfNormalVectors = intVectorToArray(positionsOfNormalVectorsVector);
  hostPoints = doubleVectorToArray(pointsVector);
  hostNumberOfPrisms = (hostNumberOfTriangles * (hostNumberOfLevels-1));

  hostRaysPerThread = ceil(double(hostRaysPerSample) /  (blocks * threads));
  hostRaysPerSample = threads * blocks * hostRaysPerThread;
  
  runtimeGpu = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  kernelcount = 0;
  surfaceTotal=0;
  minSurface=9999999;
  surfacesNormalizedSum=0;
  double hostImportance[hostNumberOfPrisms];
  int hostNumberOfImportantRays[hostNumberOfPrisms];

  for(int i=0; i < hostNumberOfPoints * hostNumberOfLevels; ++i){
    hostPhiASE[i] = 0.0;

  }

  for(int i=0; i < hostNumberOfPrisms; ++i){
    hostNumberOfImportantRays[i] = 1;
    hostImportance[i] = 1.0;
  }

  //TODO: remove
  surfacesVector->pop_back();
  for(int i=0;i<surfacesVector->size();++i){
    surfaceTotal += surfacesVector->at(i);
    minSurface = min(minSurface,surfacesVector->at(i));
  }

  for(int i=0;i<surfacesVector->size();++i){
    hostSurfacesNormalized[i] = (surfacesVector->at(i)) / minSurface;
    surfacesNormalizedSum += hostSurfacesNormalized[i];
  }

  // Init mersenne twister PRNG
  {
    /**Allocate space for PRNG states on device */
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, blocks * sizeof(hiprandStateMtgp32)));

    /** Allocate space for MTGP kernel parameters */
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));

    /**Reformat from predefined parameter sets to kernel format,
     * and copy kernel parameters to device memory */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));

    /** Initialize one state per thread block */
    /** \TODO initialize with time */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, blocks, 1234));
  }



  // Allocation of memory on the GPU and setting of global GPU-variables
  {
    fprintf(stderr, "\nFilling the device Variables\n");
    //Create constant values on GPU
    setupGlobalVariablesKernel<<<1,1>>>(
					double(hostSigmaE), 
					double(hostSigmaA),
					hostCladNumber,
					double(hostCladAbsorption),
					double(hostNTot), 
					hostNumberOfTriangles, 
					double(hostThicknessOfPrism),
					hostNumberOfLevels, 
					hostNumberOfPoints); //@OPTIMIZE: initialize the constants as constants...

    hipDeviceSynchronize();

    // Memory allocation on device
    CUDA_CHECK_RETURN(hipMalloc(&points, 2 * hostNumberOfPoints * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&xOfNormals, 3 * hostNumberOfTriangles * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&yOfNormals, 3 * hostNumberOfTriangles * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&neighbors, 3* hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&forbidden, 3* hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&positionsOfNormalVectors, 3* hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&triangleIndices, 3* hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&cellTypes,hostNumberOfTriangles * hostNumberOfLevels * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&betaValues,hostNumberOfTriangles * (hostNumberOfLevels-1) * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&phiASE,hostNumberOfPoints * hostNumberOfLevels * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc(&surfacesNormalized,hostNumberOfTriangles * sizeof(int)));
    // Memory importance sampling
    CUDA_CHECK_RETURN(hipMalloc(&importance, hostNumberOfPrisms * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&numberOfImportantRays, hostNumberOfPrisms * sizeof(int)));

    /// Copy data from host to device
    CUDA_CHECK_RETURN(hipMemcpy(points, hostPoints, 2 * hostNumberOfPoints * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(xOfNormals, hostXOfNormals, 3 * hostNumberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(yOfNormals, hostYOfNormals, 3 * hostNumberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(neighbors, hostNeighbors, 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(forbidden,hostForbidden, 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(positionsOfNormalVectors ,hostPositionsOfNormalVectors, 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(triangleIndices ,hostTriangleIndices, 3* hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(cellTypes,hostCellTypes, hostNumberOfTriangles * hostNumberOfLevels * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(betaValues, hostBetaValues, hostNumberOfTriangles * (hostNumberOfLevels-1) * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(phiASE, hostPhiASE, hostNumberOfPoints * hostNumberOfLevels * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(surfacesNormalized, hostSurfacesNormalized, hostNumberOfTriangles * sizeof(int),hipMemcpyHostToDevice));
    // Copy importance sampling data
    CUDA_CHECK_RETURN(hipMemcpy(importance, hostImportance, hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(numberOfImportantRays, hostNumberOfImportantRays, hostNumberOfPrisms * sizeof(int), hipMemcpyHostToDevice));
  }

  fprintf(stderr, "hostCrystalFluorescence: %e\n",hostCrystalFluorescence);
  //testKernel<<<1,1>>>(points, xOfNormals, yOfNormals, positionsOfNormalVectors, neighbors, forbidden, triangleIndices, cellTypes, betaValues,surfacesNormalized);

  // Start Kernels
  {
    fprintf(stderr, "\nStarting the propagation\n");
    hipEventRecord(start, 0);
		
    // Every Kernel calculates one sample point
    for(int point2D = 0; point2D < hostNumberOfPoints ; ++point2D){
      //for(int level = 0; level < hostNumberOfLevels; ++level){
       for(int level = 0; level < 1; ++level){
	hipDeviceSynchronize();
	// Importance for one sample
	importf(point2D, level, hostImportance, hostNumberOfImportantRays, 
		hostPoints, hostXOfNormals, hostYOfNormals, hostPositionsOfNormalVectors, 
		hostNeighbors, hostForbidden, hostCellTypes, hostBetaValues, 
		hostXOfTriangleCenter, hostYOfTriangleCenter, hostSurfaces, hostRaysPerSample,
		hostNumberOfPoints, hostNumberOfLevels, hostNumberOfTriangles, hostThicknessOfPrism,
		hostSigmaA, hostSigmaE, hostCladNumber, hostCladAbsorption,hostNTot
		);
	CUDA_CHECK_RETURN(hipMemcpy(importance, hostImportance, hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(numberOfImportantRays, hostNumberOfImportantRays, hostNumberOfPrisms * sizeof(int), hipMemcpyHostToDevice));

	// Calculate for one sample
	raytraceStep<<< blocks, threads >>> ( devMTGPStates, phiASE, point2D, level, hostRaysPerThread, 
					      points, xOfNormals, yOfNormals, positionsOfNormalVectors, 
					      neighbors, forbidden, triangleIndices, cellTypes, betaValues, importance, 
					      numberOfImportantRays, surfacesNormalized );

	if(kernelcount % 200 == 0)
	  fprintf(stderr, "Sampling point %d done\n",kernelcount);
	kernelcount++;
      }
    }

    hipDeviceSynchronize();
  }


  // Final calculations
  {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runtimeGpu, start, stop);
    CUDA_CHECK_RETURN(hipMemcpy(hostPhiASE, phiASE, hostNumberOfPoints * hostNumberOfLevels * sizeof(float), hipMemcpyDeviceToHost));
    //int raysPerSampleNormalized = hostThreadsPerPrism * surfacesNormalizedSum * (hostNumberOfLevels-1) * hostRaysPerThread;
    for(int i=0; i< hostNumberOfPoints;++i){
      for(int j=0 ; j<hostNumberOfLevels ; ++j)
	{
	  int pos = i*hostNumberOfLevels+j;
	  hostPhiASE[pos] = float( (double(hostPhiASE[pos]) / (hostRaysPerSample * 4.0f * 3.14159)));
	  double gain_local = double(hostNTot)*(betaCellsVector->at(pos))*double(hostSigmaE+hostSigmaA)-double(hostNTot*hostSigmaA);
	  dndtAse->at(pos) = gain_local*hostPhiASE[pos]/hostCrystalFluorescence;
	}
    }
  }

  // Free Memory
  {
    hipFree(points);
    hipFree(xOfNormals);
    hipFree(yOfNormals);
    hipFree(neighbors);
    hipFree(forbidden);
    hipFree(positionsOfNormalVectors);
    hipFree(betaValues);
    hipHostFree(hostBetaValues);
    hipHostFree(hostXOfNormals);
    hipHostFree(hostYOfNormals);
    hipHostFree(hostCellTypes);
    hipHostFree(hostTriangleIndices);
    hipHostFree(hostForbidden);
    hipHostFree(hostNeighbors);
    hipHostFree(hostPositionsOfNormalVectors);
    hipHostFree(hostPoints);
    hipFree(importance);
    hipFree(numberOfImportantRays);
  }

  hipDeviceReset();
  return runtimeGpu;
}

