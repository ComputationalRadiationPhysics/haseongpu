#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>

#include "filtergrid.h"
#include "geometry_gpu.h"

__device__ int cellIdx(const Grid *grid, int i, int j, int k){
	return k*grid->dim.x*grid->dim.y+j*grid->dim.x+i;
}

//__global__ void filter(const Grid *grid, const RayCu *ray, char *results) {
//  int gx = threadIdx.x;
//  int gy = threadIdx.y;
//  int gz = threadIdx.z;
//
//  PointCu a, b, c, d;
//  PrismCu p1, p2;
//
//  VectorCu gridAabbDim = createVector(grid->aabb.min, grid->aabb.max);
//  VectorCu cellDim;
//  cellDim.x = gridAabbDim.x / grid->dim.x;
//  cellDim.y = gridAabbDim.y / grid->dim.y;
//  cellDim.z = gridAabbDim.z / grid->dim.z;
//
//  a = createPoint(gx*cellDim.x, gy*cellDim.y, gz*cellDim.z, cellDim.z);
//  b = createPoint(gx*cellDim.x, gy*cellDim.y, gz*cellDim.z, cellDim.z);
//  c = createPoint(gx*cellDim.x, gy*cellDim.y, gz*cellDim.z, cellDim.z);
//  d = createPoint(gx*cellDim.x, gy*cellDim.y, gz*cellDim.z, cellDim.z);
//
//  p1.t1 = createTriangle(a, b, c);
//  p2.t1 = createTriangle(b, c, d);
//
//  results[cellIdx(grid, gx, gy, gz)] = (collide_prism_gpu(p1, *ray) != 0) || (collide_prism_gpu(p2, *ray) != 0);
//}

__device__ int iIdx(const Grid *grid, float x){
	//assert(x >= grid->aabb.min.x && x <= grid->aabb.max.x);
	
	return (x - grid->aabb.min.x)/(grid->aabb.max.x - grid->aabb.min.x) * grid->dim.x;
}

__device__ int jIdx(const Grid *grid, float y){
	//assert(y >= grid->aabb.min.y && y <= grid->aabb.max.y);
	
	return (y - grid->aabb.min.y)/(grid->aabb.max.y - grid->aabb.min.y) * grid->dim.y;
}

__device__ int kIdx(const Grid *grid, float z){
	//assert(z >= grid->aabb.min.z && z <= grid->aabb.max.z);
	
	return (z - grid->aabb.min.z)/(grid->aabb.max.z - grid->aabb.min.z) * grid->dim.z;
}

//void cellIdx2ijk(const Grid *grid, int cellIdx, int *i, int *j, int *k){
//	
//	int dimXdimY=grid->dim.x*grid->dim.y;
//	int tmp;
//	
//	*k = cellIdx / dimXdimY;
//	
//	tmp=cellIdx - *k * dimXdimY;
//	//*j = (cellIdx - *k * dimXdimY) / grid->dim.x;
//	*j = tmp / grid->dim.x;
//	
//	//*i = cellIdx - *k * dimXdimY - *j* grid->dim.x;
//	*i = tmp - *j * grid->dim.x;
//}

__device__ PointCu calcPointOnRay(const RayCu *ray, float t) {
  PointCu p;

  p.x = ray->P.x + t*ray->direction.x;
  p.y = ray->P.y + t*ray->direction.y;
  p.z = ray->P.z + t*ray->direction.z;
  
  return p;
}

__device__ bool nextIntersection(const Grid *grid, const RayCu *ray, float *t, PointCu p) {
  float4 nextTs;
  nextTs.x = 0; nextTs.y = 0; nextTs.z = 0; // mark as invalid

  int step, next;
  float nextCoord, nextT;
  if(ray->direction.x != 0 && grid->dim.x != 0) {
    if(ray->direction.x > 0) step = 1;
    else step = -1;
    next = iIdx(grid, p.x) + step;
    nextCoord = ((float)next)/grid->dim.x * (grid->aabb.max.x - grid->aabb.min.x) + grid->aabb.min.x;
    nextT = (nextCoord - ray->P.x) / ray->direction.x;
    nextTs.x = nextT;
  }
  if(ray->direction.y != 0 && grid->dim.y != 0) {
    if(ray->direction.y > 0) step = 1;
    else step = -1;
    next = jIdx(grid, p.y) + step;
    nextCoord = ((float)next)/grid->dim.y * (grid->aabb.max.y - grid->aabb.min.y) + grid->aabb.min.y;
    nextT = (nextCoord - ray->P.y) / ray->direction.y;
    nextTs.y = nextT;
  }
  if(ray->direction.z != 0 && grid->dim.z != 0) {
    if(ray->direction.z > 0) step = 1;
    else step = -1;
    next = kIdx(grid, p.z) + step;
    nextCoord = ((float)next)/grid->dim.z * (grid->aabb.max.z - grid->aabb.min.z) + grid->aabb.min.z;
    nextT = (nextCoord - ray->P.z) / ray->direction.z;
    nextTs.z = nextT;
  }

  if(nextTs.x <= *t) nextTs.x = 1.1;
  if(nextTs.y <= *t) nextTs.y = 1.1;
  if(nextTs.z <= *t) nextTs.z = 1.1;
  *t = fmin(nextTs.x, fmin(nextTs.y, nextTs.z));
  if(*t > 1) return false;
  
  return true;
}

__device__ int4 calcCellpos(const Grid *grid, PointCu p) {
  int4 result;

  result.x = ((int) p.x) % grid->dim.x;
  result.y = ((int) p.y) % grid->dim.y;
  result.z = ((int) p.z) % grid->dim.z;

  return result;
}

__device__ int* cuRealloc(int *old, int *size) {
  int oldSize = *size;
  int* res = (int*) malloc(oldSize * 2);
  *size *= 2;

  for(int i=0; i<oldSize; ++i)
    res[i] = old[i];

  free(old);
  return res;
}

__device__ bool isDuplicate(int *a, int length, int o) {
  for(int i=0; i<length; ++i) {
    if(a[i] == o) return true;
  }
  return false;
}

__device__ int* addPrismsToResult(const GridCell *cell, int* results, int *results_count, int *results_size) {
  int limit = *results_count + cell->length;
  if(limit >= *results_size) {
    while(limit >= *results_size) *results_size += 256;
    results = cuRealloc(results, results_size);
  }

  int added = 0;
  for(int i=0; i<cell->length; ++i) {
    int current = cell->prismIdxList[i];
    if(!isDuplicate(results, *results_count, current)) {
      results[*results_count + added] = current;
      added++;
    }
  }
  *results_count += added;
  return results;
}

__device__ int* filter(const Grid *grid, const RayCu *ray, int* result_size) {
  int *results = (int*) malloc(256 * sizeof(int));
  int results_count = 0;
  int results_size = 256;

  float t = 0;
  PointCu p;
  do {
    p = calcPointOnRay(ray, t);
    int4 cellpos = calcCellpos(grid, p);
    GridCell cell = grid->cellList[cellIdx(grid, cellpos.x, cellpos.y, cellpos.z)];
    results = addPrismsToResult(&cell, results, &results_count, &results_size); // uses sometimes realloc and changes results, results_count and results_size
  } while(nextIntersection(grid, ray, &t, p)); // changes t, but not p (is being done in next loop)

  *result_size = results_count;
  return results;
}
