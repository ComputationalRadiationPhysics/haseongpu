#include "hip/hip_runtime.h"
#include "map_rays_to_prisms.h"
#include <stdio.h>
#include <assert.h>
#include <iterator>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <logging.h>


using thrust::device_vector;
using thrust::host_vector;
using thrust::raw_pointer_cast;

__global__ void mapPrefixSumToPrisms(
    const unsigned numberOfPrisms,
    const unsigned raysPerSample,
    const unsigned reflectionSlices,
    const unsigned* raysPerPrism,
    const unsigned* prefixSum,
    unsigned *indicesOfPrisms,
    unsigned *numberOfReflections
    ){

  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  if(id >= numberOfPrisms*reflectionSlices) return;

  const unsigned count            = raysPerPrism[id];
  const unsigned startingPosition = prefixSum[id];
  const unsigned reflection_i     = id / numberOfPrisms;
  const unsigned prism_i          = id % numberOfPrisms;

  for(unsigned i=0; i < count ; ++i){
    indicesOfPrisms[startingPosition + i] = prism_i;     
    numberOfReflections[startingPosition + i] = reflection_i; 
  }
}


void GPU_algorithm(
    const unsigned numberOfPrisms, 
    const unsigned raysPerSample,
    const unsigned reflectionSlices,
    const device_vector<unsigned>& raysPerPrism, 
    device_vector<unsigned> &prefixSum, 
    device_vector<unsigned> &indicesOfPrisms, 
    device_vector<unsigned> &numberOfReflections
    )
{
  const unsigned blocksize = 256;
  const unsigned gridsize  = (raysPerPrism.size()+blocksize-1)/blocksize;

  thrust::exclusive_scan(raysPerPrism.begin(), raysPerPrism.end(),prefixSum.begin());

  mapPrefixSumToPrisms <<<gridsize,blocksize>>> (
      numberOfPrisms, 
      raysPerSample, 
      reflectionSlices,
      raw_pointer_cast( &raysPerPrism[0] ),
      raw_pointer_cast( &prefixSum[0] ), 
      raw_pointer_cast( &indicesOfPrisms[0] ),
      raw_pointer_cast( &numberOfReflections[0] )
      );
}


void CPU_algorithm(
    const unsigned numberOfPrisms, 
    const unsigned raysPerSample,
    const unsigned reflectionSlices,
    const host_vector<unsigned>& raysPerPrism, 
    host_vector<unsigned>& indicesOfPrisms, 
    host_vector<unsigned>& numberOfReflections
    ){

  unsigned absoluteRay = 0;
  for(unsigned reflection_i=0; reflection_i < reflectionSlices ; ++reflection_i){
    unsigned reflectionOffset = reflection_i * numberOfPrisms;

    for(unsigned prism_i=0 ; prism_i < numberOfPrisms; ++prism_i){
      for(unsigned ray_i=0; ray_i < raysPerPrism[prism_i + reflectionOffset]; ++ray_i){
        indicesOfPrisms[absoluteRay]     = prism_i;
        numberOfReflections[absoluteRay] = reflection_i;
        ++absoluteRay;
        assert(absoluteRay <= raysPerSample);
      }
    }
  }
}


void mapRaysToPrisms(
    device_vector<unsigned> &indicesOfPrisms,
    device_vector<unsigned> &numberOfReflections,
    const device_vector<unsigned> &raysPerPrism,
    device_vector<unsigned> &prefixSum,
    const unsigned reflectionSlices,
    const unsigned raysPerSample,
    const unsigned numberOfPrisms
    ){

  //time_t before_GPU = clock();
  GPU_algorithm(
      numberOfPrisms,
      raysPerSample,
      reflectionSlices,
      raysPerPrism,
      prefixSum,
      indicesOfPrisms,
      numberOfReflections
      );
  //time_t after_GPU = clock();

  // only for error-checking!
  //time_t before_CPU = clock();
  //host_vector<unsigned> indicesOfPrisms2(indicesOfPrisms);
  //host_vector<unsigned> numberOfReflections2(numberOfReflections);
  //CPU_algorithm(
  //   numberOfPrisms,
  //   raysPerSample,
  //   reflectionSlices,
  //   host_vector<unsigned>(raysPerPrism),
  //   indicesOfPrisms2, 
  //   numberOfReflections2
  //   );
  //indicesOfPrisms = indicesOfPrisms2;
  //numberOfReflections = numberOfReflections2; 
  //time_t after_CPU = clock();

  //some timing
  //int timeGPU = after_GPU - before_GPU;
  //int timeCPU = after_CPU - before_CPU;
  //dout(V_STAT) << "time GPU including malloc: " << timeGPU/1000 << "k Cycles" << std::endl;
  //dout(V_STAT) << "time CPU: " << timeCPU/1000 << "k Cycles" << std::endl;

  // some errorchecking
  //for(unsigned i=0; i<indicesOfPrisms2.size(); ++i){
  //  assert(indicesOfPrisms2[i] == indicesOfPrisms[i]);
  //}
}
