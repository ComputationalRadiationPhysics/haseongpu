#include "hip/hip_runtime.h"
#include "map_rays_to_prisms.h"
#include "cudachecks.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <iterator>
#include <thrust/scan.h>
#include <thrust/device_vector.h>


using thrust::device_vector;
using thrust::host_vector;
using thrust::raw_pointer_cast;

__global__ void mapPrefixSumToPrisms(
    const unsigned numberOfPrisms,
    const unsigned raysPerSample,
    const unsigned reflectionSlices,
    const unsigned* raysPerPrism,
    const unsigned* prefixSum,
    unsigned *indicesOfPrisms,
    unsigned *numberOfReflections
    ){

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id >= numberOfPrisms*reflectionSlices) return;

  const unsigned count = raysPerPrism[id];
  const unsigned startingPosition = prefixSum[id];
  const unsigned reflection_i = id/numberOfPrisms;
  const unsigned prism_i      = id%numberOfPrisms;

  for(unsigned i=0; i<count ; ++i){
    indicesOfPrisms[startingPosition + i] = prism_i;     
    numberOfReflections[startingPosition + i] = reflection_i; 
  }
}



void GPU_algorithm(
    const unsigned numberOfPrisms, 
    const unsigned raysPerSample,
    const unsigned reflectionSlices,
    const thrust::device_vector<unsigned>& raysPerPrism, 
    thrust::device_vector<unsigned> &prefixSum, 
    thrust::device_vector<unsigned> &indicesOfPrisms, 
    thrust::device_vector<unsigned> &numberOfReflections
    )
{
  const unsigned blocksize = 256;
  const unsigned gridsize  = (raysPerPrism.size()+blocksize-1)/blocksize;

  thrust::exclusive_scan(raysPerPrism.begin(), raysPerPrism.end(),prefixSum.begin());

  CUDA_CHECK_KERNEL_SYNC(mapPrefixSumToPrisms <<<gridsize,blocksize>>> (
      numberOfPrisms, 
      raysPerSample, 
      reflectionSlices,
      raw_pointer_cast( &raysPerPrism[0] ),
      raw_pointer_cast( &prefixSum[0] ), 
      raw_pointer_cast( &indicesOfPrisms[0] ),
      raw_pointer_cast( &numberOfReflections[0] )
      ));

}


void CPU_algorithm(
    thrust::host_vector<unsigned>& indicesOfPrisms, 
    const thrust::host_vector<unsigned>& raysPerPrism, 
    const unsigned numberOfPrisms, 
    const unsigned raysPerSample,
    thrust::host_vector<unsigned>& numberOfReflections,
    const unsigned reflectionSlices
    ){
  unsigned absoluteRay = 0;
  for(unsigned reflection_i=0; reflection_i < reflectionSlices ; ++reflection_i){
    unsigned reflectionOffset = reflection_i * numberOfPrisms;

    for(unsigned prism_i=0 ; prism_i < numberOfPrisms; ++prism_i){
      for(unsigned ray_i=0; ray_i < raysPerPrism[prism_i + reflectionOffset]; ++ray_i){
        indicesOfPrisms[absoluteRay] = prism_i;
        numberOfReflections[absoluteRay] = reflection_i;
        absoluteRay++;
        assert(absoluteRay <= raysPerSample);
      }
    }
  }
}

void mapRaysToPrisms(
    thrust::device_vector<unsigned> &indicesOfPrisms,
    thrust::device_vector<unsigned> &numberOfReflections,
    const thrust::device_vector<unsigned> &raysPerPrism,
    thrust::device_vector<unsigned> &prefixSum,
    const unsigned reflectionSlices,
    const unsigned raysPerSample,
    const unsigned numberOfPrisms
    ){


  //fill(indicesOfPrisms.begin(),indicesOfPrisms.end(),0);
  //fill(numberOfReflections.begin(),numberOfReflections.end(),0);

  //time_t before_GPU = clock();
  GPU_algorithm(
      numberOfPrisms,
      raysPerSample,
      reflectionSlices,
      raysPerPrism,
      prefixSum,
      indicesOfPrisms,
      numberOfReflections
      );
  //time_t after_GPU = clock();

  // only for error-checking!
  //time_t before_CPU = clock();
  //host_vector<unsigned> indicesOfPrisms2(indicesOfPrisms);
  //host_vector<unsigned> numberOfReflections2(numberOfReflections);
  //CPU_algorithm(indicesOfPrisms2, host_vector<unsigned>(raysPerPrism), numberOfPrisms, raysPerSample,numberOfReflections2,reflectionSlices);
  //indicesOfPrisms = indicesOfPrisms2;
  //numberOfReflections = numberOfReflections2; 
  //time_t after_CPU = clock();

  //some timing
  //int timeGPU = after_GPU - before_GPU;
  //int timeCPU = after_CPU - before_CPU;
  //std::cout << "time GPU including malloc: " << timeGPU/1000 << "k Cycles" << std::endl;
  //std::cout << "time CPU: " << timeCPU/1000 << "k Cycles" << std::endl;

  // some errorchecking
  //for(unsigned i=0; i<indicesOfPrisms2.size(); ++i){
  //  assert(indicesOfPrisms2[i] == indicesOfPrisms[i]);
  //}
}
