#include "hip/hip_runtime.h"
/**
 * Copyright 2013 Erik Zenker, Carlchristian Eckert, Marius Melzer
 *
 * This file is part of HASEonGPU
 *
 * HASEonGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * HASEonGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with HASEonGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */


#include <thrust/scan.h>

#include <thrust_device_vector_nowarn.hpp> 
#include <map_rays_to_prisms.hpp>


using thrust::device_vector;
using thrust::host_vector;
using thrust::raw_pointer_cast;

/**
 * @brief takes a prefix sum (obtained by an exclusive scan over raysPerPrism) 
 *        and writes it into a unary representation. The value from the 
 *        prefixSum at index i describes the offset where to start writing,
 *        whereas i itself is the new value to be stored in the output array.
 *
 *        example:
 *        raysPerPrism [3,0,2,1] 
 *
 *        -> exclusive prefixSum [0,3,3,5] 
 *
 *        beginning from place 0 in the output should be 0 (length 3 according to raysPerPrism[0])
 *        beginning from place 3 in the output should be 1 (EMPTY range at raysPerPrism[1])
 *        beginning from place 3 in the output should be 2 (length 2 according to raysPerPrism[2])
 *        beginning from place 5 in the output should be 3 (length 1 according to raysPerPrism[3])
 *        
 *        resulting output arrays:
 *        [0 0 0 2 2 3] (indicesOfPrisms)
 *
 *        output numberOfReflections is handled in a similar way
 *
 *
 * @param numberOfPrisms the number of prisms. numberOfPrisms * reflectionSlices
 *                       must be equal to the the length of the prefixSum.
 * @param raysPerSample the size of indicesOfPrisms/numberOfReflections. Actually 
 *                      identical to the sum of all values in raysPerPrism
 * @param reflectionSlices the number of reflectionSlices. see numberOfPrisms
 * @param raysPerPrism the input array from which prefixSum was generated
 * @param prefixSum the prefixSum generated from raysPerPrism
 * @param indicesOfPrisms a pointer to the OUTPUT generated like described in the example
 * @param numberOfReflections a pointer to the OUTPUT similar to indicesOfPrisms
 */
__global__ void mapPrefixSumToPrisms(
    const unsigned numberOfPrisms,
    const unsigned raysPerSample,
    const unsigned reflectionSlices,
    const unsigned* raysPerPrism,
    const unsigned* prefixSum,
    unsigned *indicesOfPrisms,
    unsigned *numberOfReflections
    ){

  int id = threadIdx.x + (blockIdx.x * blockDim.x);
  // break if we have too many threads (this is likely)
  if(id >= numberOfPrisms*reflectionSlices) return;

  const unsigned count            = raysPerPrism[id];
  const unsigned startingPosition = prefixSum[id];
  const unsigned reflection_i     = id / numberOfPrisms;
  const unsigned prism_i          = id % numberOfPrisms;

  for(unsigned i=0; i < count ; ++i){
    indicesOfPrisms[startingPosition + i] = prism_i;     
    numberOfReflections[startingPosition + i] = reflection_i; 
  }
}

void mapRaysToPrisms(
    device_vector<unsigned> &indicesOfPrisms, 
    device_vector<unsigned> &numberOfReflections,
    const device_vector<unsigned> &raysPerPrism, 
    device_vector<unsigned> &prefixSum, 
    const unsigned reflectionSlices,
    const unsigned raysPerSample,
    const unsigned numberOfPrisms
    ){

  // blocksize chosen by occupancyCalculator
  const unsigned blocksize = 256;
  const unsigned gridsize  = (raysPerPrism.size()+blocksize-1)/blocksize;

  thrust::exclusive_scan(raysPerPrism.begin(), raysPerPrism.end(),prefixSum.begin());

  mapPrefixSumToPrisms <<<gridsize,blocksize>>> (
      numberOfPrisms, 
      raysPerSample, 
      reflectionSlices,
      raw_pointer_cast( &raysPerPrism[0] ),
      raw_pointer_cast( &prefixSum[0] ), 
      raw_pointer_cast( &indicesOfPrisms[0] ),
      raw_pointer_cast( &numberOfReflections[0] )
      );
}
