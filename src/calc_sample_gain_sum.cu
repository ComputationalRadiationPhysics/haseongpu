#include "hip/hip_runtime.h"
/**
 * Copyright 2013 Erik Zenker, Carlchristian Eckert, Marius Melzer
 *
 * This file is part of HASEonGPU
 *
 * HASEonGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * HASEonGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with HASEonGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */


#include <cassert> /* assert */

#include <mesh.hpp>
#include <geometry.hpp> /* generateRay */
#include <propagate_ray.hpp> /* propagateRay */
#include <reflection.hpp> /* ReflectionPlane */

#include <hiprand/hiprand_kernel.h> /*hiprand_uniform*/

/**
 * @brief get the offset for accessing indicesOfPrisms and numberOfReflectionSlices (slow!).
 *
 * @param blockOffset shared memory location that holds the offset for the whole block (4 warps)
 * @param raysPerSample number of raysPerSample (can be any number higher than raysPerSample/warpsize)
 * @param globalOffsetMultiplicator is incremented by 1 each time a warp asks for a new workload
 * @return an unused offset in the global arrays indicesOfPrisms/numberOfReflectionSlices
 *
 */
__device__ unsigned getRayNumberWarpbased(unsigned* blockOffset,unsigned raysPerSample, unsigned *globalOffsetMultiplicator){
	// if this is warpID 0
	if((threadIdx.x &31) == 0){
		//get a new offset for the warp (threadId % 32)
		blockOffset[(threadIdx.x>>5)] = atomicInc(globalOffsetMultiplicator,raysPerSample);
	}
	__syncthreads();

	// multiply blockoffset by 32 (size of warp)
	return (threadIdx.x &31) + (blockOffset[(threadIdx.x>>5)] <<5) ;

}

/**
 * @brief get the offset for accessing indicesOfPrisms and numberOfReflectionSlices.
 *        Warning: works only for a blocksize of 128 threads!
 *
 * @param blockOffset shared memory location that holds the offset for the whole block
 * @param raysPerSample number of raysPerSample (can be any number higher than raysPerSample/blocksize)
 * @param globalOffsetMultiplicator is incremented by 1 each time a block asks for a new workload
 * @return an unused offset in the global arrays indicesOfPrisms/numberOfReflectionSlices
 *
 */
__device__ unsigned getRayNumberBlockbased(unsigned* blockOffset,unsigned raysPerSample,unsigned *globalOffsetMultiplicator){
	// The first thread in the threadblock increases the globalOffsetMultiplicator (without real limit) 
	if(threadIdx.x == 0){
		//blockOffset is the new value of the globalOffsetMultiplicator
		blockOffset[0] = atomicInc(globalOffsetMultiplicator,raysPerSample);
	}
	__syncthreads();

	//multiply blockOffset by 128 (size of the threadblock) 
	return threadIdx.x + (blockOffset[0] <<7) ;
}

/**
 * @brief get a random number from [0..length)
 *
 * @param length the maximum number to return (exclusive)
 * @param globalState State for random number generation (mersenne twister).
 *                    The state need to be initialized before. See
 *                    http://www.math.sci.hiroshima-u.ac.jp/~m-mat/MT/MTGP/
 *                    for more information.
 *
 * @return a random number
 *
 */
__device__ __inline__ unsigned genRndSigmas(unsigned length, hiprandStateMtgp32* globalState) {
  return unsigned(hiprand_uniform(&globalState[blockIdx.x])*(length-1));
}

__global__ void calcSampleGainSumWithReflection(hiprandStateMtgp32* globalState,
						const Mesh mesh, 
						const unsigned* indicesOfPrisms, 
						const unsigned reflection_i,
						const double* importance,
						const unsigned raysPerSample,
						float *gainSum, 
						float *gainSumSquare,
						const unsigned sample_i,
						const double *sigmaA, 
						const double *sigmaE,
						const unsigned maxInterpolation,
						unsigned *globalOffsetMultiplicator
						) {

  int rayNumber = 0;
  double gainSumTemp = 0;
  double gainSumSquareTemp = 0;
  Point samplePoint = mesh.getSamplePoint(sample_i);
  __shared__ unsigned blockOffset[4]; // 4 in case of warp-based raynumber

  // One thread can compute multiple rays
  while (true) {
	// the whole block gets a new offset (==workload)
	rayNumber = getRayNumberBlockbased(blockOffset,raysPerSample,globalOffsetMultiplicator);
	if(rayNumber >= raysPerSample) break;

    // Get triangle/prism to start ray from
    unsigned startPrism             = indicesOfPrisms[rayNumber];
    unsigned reflections            = (reflection_i + 1) / 2;
    ReflectionPlane reflectionPlane = (reflection_i % 2 == 0) ? BOTTOM_REFLECTION : TOP_REFLECTION;
    unsigned startLevel             = startPrism / mesh.numberOfTriangles;
    unsigned startTriangle          = startPrism - (mesh.numberOfTriangles * startLevel);
    Point startPoint                = mesh.genRndPoint(startTriangle, startLevel, globalState);
	
	//get a random index in the wavelength array
    unsigned sigma_i                = genRndSigmas(maxInterpolation, globalState);

    // Calculate reflections as different ray propagations
    double gain    = propagateRayWithReflection(startPoint, samplePoint, reflections, reflectionPlane, startLevel, startTriangle, mesh, sigmaA[sigma_i], sigmaE[sigma_i]);

	// include the stimulus from the starting prism and the importance of that ray
    gain          *= mesh.getBetaVolume(startPrism) * importance[startPrism];
    
    assert(!isnan(mesh.getBetaVolume(startPrism)));
    assert(!isnan(importance[startPrism]));
    assert(!isnan(gain));

    gainSumTemp       += gain;
    gainSumSquareTemp += gain * gain;


  }
  atomicAdd(&(gainSum[0]), float(gainSumTemp));
  atomicAdd(&(gainSumSquare[0]), float(gainSumSquareTemp));

}

__global__ void calcSampleGainSum(hiprandStateMtgp32* globalState,
				  const Mesh mesh, 
				  const unsigned* indicesOfPrisms, 
				  const double* importance,
				  const unsigned raysPerSample,
				  float *gainSum, 
				  float *gainSumSquare,
				  const unsigned sample_i,
				  const double* sigmaA, 
				  const double* sigmaE,
				  const unsigned lambdaResolution,
				  unsigned *globalOffsetMultiplicator
				  ) {

  int rayNumber = 0; 
  double gainSumTemp = 0;
  double gainSumSquareTemp = 0;
  Point samplePoint = mesh.getSamplePoint(sample_i);
  __shared__ unsigned blockOffset[4]; // 4 in case of warp-based raynumber
  
  // One thread can compute multiple rays
  while(true){
	// the whole block gets a new offset (==workload)
    rayNumber = getRayNumberBlockbased(blockOffset,raysPerSample,globalOffsetMultiplicator);
    if(rayNumber>=raysPerSample) break;

    // Get triangle/prism to start ray from
    unsigned startPrism             = indicesOfPrisms[rayNumber];
    unsigned startLevel             = startPrism/mesh.numberOfTriangles;
    unsigned startTriangle          = startPrism - (mesh.numberOfTriangles * startLevel);
    Point startPoint                = mesh.genRndPoint(startTriangle, startLevel, globalState);
    Ray ray                         = generateRay(startPoint, samplePoint);

	// get a random index in the wavelength array
    unsigned sigma_i                = genRndSigmas(lambdaResolution, globalState);
    assert(sigma_i < lambdaResolution);

	// calculate the gain for the whole ray at once
    double gain    = propagateRay(ray, &startLevel, &startTriangle, mesh, sigmaA[sigma_i], sigmaE[sigma_i]);
    gain          /= ray.length * ray.length; // important, since usually done in the reflection device function

	// include the stimulus from the starting prism and the importance of that ray
    gain          *= mesh.getBetaVolume(startPrism) * importance[startPrism];

    gainSumTemp       += gain;
    gainSumSquareTemp += gain * gain;

  }
  atomicAdd(&(gainSum[0]), float(gainSumTemp));
  atomicAdd(&(gainSumSquare[0]), float(gainSumSquareTemp));

}
