#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <mesh.h>
#include <stdio.h>
#include <propagate_ray.h>
#include <geometry.h>
#include <assert.h>

// ##############################################################
// # Reconstruction                                             #
// ##############################################################
void importanceSamplingNew(Point samplePoint, Mesh mesh, unsigned raysPerSample, double sigmaA, double sigmaE, double nTot,  double *importance, unsigned *raysPerPrism){
  Triangle *triangles = mesh.triangles;
  unsigned numberOfLevels = mesh.numberOfLevels;
  unsigned numberOfTriangles = mesh.numberOfTriangles;
  float thickness = mesh.thickness;

  int raysLeft = 0;
  int raysDump = 0;
  double sumPhi = 0;
  double surfaceTotal = 0;
  double gain = 0;
  Ray ray;
  Point startPoint;
  Triangle startTriangle;

  // Calculate importance by propagation from trianglecenter to every other center
  for(unsigned triangle_i = 0; triangle_i < numberOfTriangles; ++triangle_i){
    for(unsigned level_i = 0; level_i < numberOfLevels - 1; ++level_i){
      startTriangle = triangles[triangle_i];
      startPoint.x = startTriangle.center.x;
      startPoint.y = startTriangle.center.y;
      startPoint.z = (level_i + 0.5) * thickness;
      // DEBUG
      // printf("\nstartpoint x %f\n", startPoint.x );
      // printf("startpoint y %f\n",  startPoint.y);
      // printf("startpoint z %f\n",  startPoint.z);
      // printf("endpoint x %f\n", samplePoint.x);
      // printf("endpoint y %f\n",  samplePoint.y);
      // printf("endpoint z %f\n", samplePoint.z);
      ray = generateRay(startPoint, samplePoint);

      gain = propagateRay(ray, level_i, startTriangle, triangles, sigmaA, sigmaE, nTot, thickness);

      importance[triangle_i + level_i * numberOfTriangles] = startTriangle.betaValues[level_i] * gain;
      sumPhi += importance[triangle_i + level_i * numberOfTriangles];

    }
    surfaceTotal += triangles[triangle_i].surface;
  }

  // Calculate number of rays/prism
  for(unsigned triangle_i = 0; triangle_i < numberOfTriangles; ++triangle_i){
    for(unsigned level_i = 0; level_i < numberOfLevels - 1; ++level_i){
      raysPerPrism[triangle_i + level_i * numberOfTriangles] =  (unsigned)(floor(importance[triangle_i + level_i * numberOfTriangles] / sumPhi * raysPerSample));
      raysDump +=  raysPerPrism[triangle_i + level_i * numberOfTriangles];
      raysDump +=  raysPerPrism[0];
    }

  }

  raysLeft = raysPerSample - raysDump;

  // TODO What happens with random failure ?
  // TODO Distribute the remaining rays randomly
  // for (int i_r=0; i_r < raysLeft; i_r++){
  //   int rand_t = (int )(rand() % numberOfTriangles);
  //   int rand_z = (int )(rand() % (numberOfLevels-1));
  //   raysPerPrism[rand_t + rand_z * numberOfTriangles]++;

  // }

  //  Now think about the mount of rays which would come out of this volume(surface)
  //  dividing this number with the new amount of rays gives the final importance weight for this area!
  // for (int triangle_i=0; triangle_i < numberOfTriangles; ++triangle_i){
  //   for (int level_i=0; level_i < numberOfLevels; ++level_i){
  //     if (raysPerPrism[triangle_i + (level_i * numberOfTriangles)] > 0){
  // 	importance[triangle_i + (level_i * numberOfTriangles)] = raysPerSample * triangles[triangle_i].surface / surfaceTotal / raysPerPrism[triangle_i + (level_i * numberOfTriangles)];

  //     }
  //     else{
  // 	importance[triangle_i + (level_i * numberOfTriangles)] = 0; 

  //     }

  //   }

  // }

}


// #################################################
// # Old Code                                      #
// #################################################
=======
#include <stdio.h>
#include "propagate_ray.h"
>>>>>>> c872b097b14330c8dd939cf52fada8582d7015d6


/**
 * calculate the gain from the centers of each of the boxes to the observed point
 * calculate the gain and make a "mapping"
 * receipt: pick the point in the center of one cell, 
 * calculate the gain from this point to the observed point,
 * estimate the inner part of the Phi_ASE - Integral,
 * scale the amount of rays proportionally with it
 * sum the amount of rays and scale it to Int=1, which gives the inverse weights
 * the number of rays is determined via floor(), with ceil(), zero-redions could be added
 * use the routine "propagation"!, test: no reflections, just exponential
 *
 **/
unsigned importanceSampling(int point,
    int startLevel,
    double *importance,
    unsigned *numberOfImportantRays,
    double *points,
    double *xOfNormals,
    double *yOfNormals,
    int *positionsOfNormalVectors,
    int *neighbors,
    int *forbidden,
    double *betaValues,
    double *xOfTriangleCenter,
    double *yOfTriangleCenter,
    float *surface,
    unsigned raysPerSample,
    unsigned numberOfPoints,
    unsigned numberOfLevels,
    unsigned numberOfTriangles,
    float thicknessOfPrism,
    float sigmaA,
    float sigmaE,
    float nTot
    )
{
  int raysLeft;
  unsigned raysDump;
  double sumPhi;
  double surfaceTotal;
  double xPos, yPos, zPos;
  double prop;

  raysDump = 0;
  sumPhi = 0.0;
  surfaceTotal = 0.0;
  xPos = points[point];
  yPos = points[point + numberOfPoints];
  zPos = startLevel * thicknessOfPrism;

  // Calculate importance by propagation from trianglecenter to every other center
  for (int i_t=0; i_t < numberOfTriangles; ++i_t){
    for (int i_z=0; i_z < (numberOfLevels-1); ++i_z){
      prop = propagateRay(xOfTriangleCenter[i_t], yOfTriangleCenter[i_t], 
          thicknessOfPrism * (i_z+0.5),  xPos, yPos, zPos, i_t, i_z, 
          points, xOfNormals, yOfNormals, positionsOfNormalVectors, 
          neighbors, forbidden, betaValues,
          nTot, sigmaE, sigmaA, thicknessOfPrism, numberOfLevels, numberOfPoints, numberOfTriangles);

      importance[i_t + i_z * numberOfTriangles] = betaValues[i_t + i_z * numberOfTriangles]*(prop);
      sumPhi += importance[i_t + i_z * numberOfTriangles];

    }
    surfaceTotal += surface[i_t];

  }

  // Calculate number of rays/prism
  for (int i_t=0; i_t < numberOfTriangles; ++i_t){
    for (int i_z=0; i_z < (numberOfLevels-1); ++i_z){
      numberOfImportantRays[i_t + i_z * numberOfTriangles] = (unsigned)(floor(importance[i_t + i_z * numberOfTriangles] / sumPhi * raysPerSample));
      raysDump +=  numberOfImportantRays[i_t + i_z * numberOfTriangles];
    }

  }
  raysLeft = raysPerSample - raysDump;

  // TODO What happens with random failure ?
  // Distribute the remaining rays randomly
<<<<<<< HEAD
  // for (int i_r=0; i_r < raysLeft; i_r++){
  //   int rand_t = (int )(rand() % numberOfTriangles);
  //   int rand_z = (int )(rand() % (numberOfLevels-1));
  //   numberOfImportantRays[rand_t + rand_z * numberOfTriangles]++;

  // }
=======
    for (int i_r=0; i_r < raysLeft; i_r++){
      int rand_t = (int )(rand() % numberOfTriangles);
      int rand_z = (int )(rand() % (numberOfLevels-1));
      numberOfImportantRays[rand_t + rand_z * numberOfTriangles]++;
  
    }
>>>>>>> c872b097b14330c8dd939cf52fada8582d7015d6

  //  Now think about the mount of rays which would come out of this volume(surface)
  //  dividing this number with the new amount of rays gives the final importance weight for this area!
  for (int i_t=0; i_t<numberOfTriangles; ++i_t){
    for (int i_z=0; i_z<(numberOfLevels-1); ++i_z){
      if (numberOfImportantRays[i_t + i_z*numberOfTriangles] > 0){
        importance[i_t + i_z*numberOfTriangles] = raysPerSample * surface[i_t] / surfaceTotal / numberOfImportantRays[i_t + i_z*numberOfTriangles];
      }
      else{
        importance[i_t + i_z*numberOfTriangles] = 0; 
      }
    }
  }
  //return raysDump;
  return raysPerSample;
}
