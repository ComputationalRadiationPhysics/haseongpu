#include "hip/hip_runtime.h"
#include <mesh.h>
#include <stdio.h>
#include <propagate_ray.h>
#include <geometry.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>

/**
 * @brief calculates a first estimate on the importance of each prism, based on a single ray started in the center of each prism
 *
 * @param *importance will contain the initial importance for each prism
 *
 * @param *sumPhi will contain the cumulative sum of the importance values
 *
 * For other parameters, see documentation of importanceSampling()
 *
 */
__global__ void propagateFromTriangleCenter(
					    Mesh *mesh,
					    double *importance,
					    float *sumPhi,
					    unsigned sample_i,
					    double sigmaA, 
					    double sigmaE, 
					    double nTot){

  //Triangle *triangles = mesh.triangles;
  __shared__ double threadPhi[256];
  double gain = 0;
  Ray ray;
  

  threadPhi[threadIdx.x] = 0;

  int startPrism = threadIdx.x + blockIdx.x * blockDim.x;
  if(startPrism >= mesh->numberOfPrisms){
    return;
  }
  int level_i = startPrism/(mesh->numberOfTriangles);
  unsigned triangle_i = startPrism - (mesh->numberOfTriangles * level_i);
  Point startPoint = mesh->getCenterPoint(triangle_i, level_i);
  Point samplePoint = mesh->getSamplePoint(sample_i);

  ray = generateRay(startPoint, samplePoint);
  gain = propagateRay(ray, level_i, triangle_i, mesh, sigmaA, sigmaE, nTot, mesh->thickness);
  importance[startPrism] = mesh->getBetaValue(triangle_i, level_i) * gain;

  threadPhi[threadIdx.x] = importance[triangle_i + level_i * mesh->numberOfTriangles];
  __syncthreads();

  unsigned i = blockDim.x/2;
  while(i != 0){
    if(threadIdx.x < i){
      threadPhi[threadIdx.x] += threadPhi[threadIdx.x + i];
    }
    __syncthreads();
    i /= 2;
  }
  if(threadIdx.x == 0){
    atomicAdd(sumPhi, float(threadPhi[threadIdx.x]));
  }
}

/**
 * @brief uses a given importance distribution to decide how many rays will be launched from each prism
 *
 * @param *raysDump will contain the number of rays which were mapped to a specific prism
 * 
 * for other parameters, see documentation of importanceSampling()
 */
__global__ void distributeRaysByImportance(
					   Mesh *mesh,
					   unsigned *raysPerPrism,
					   double *importance,
					   float *sumPhi,
					   unsigned raysPerSample,
					   unsigned *raysDump){
  __shared__ unsigned raySum[256];
  raySum[threadIdx.x] = 0;
  int startPrism = threadIdx.x + blockIdx.x * blockDim.x;
  if(startPrism >= mesh->numberOfPrisms) return;
  raysPerPrism[startPrism] = (unsigned) floor(importance[startPrism] / (*sumPhi) * raysPerSample);
  raySum[threadIdx.x] = raysPerPrism[startPrism];
  __syncthreads();

  unsigned i = blockDim.x/2;
  while(i != 0){
    if(threadIdx.x < i){
      raySum[threadIdx.x] += raySum[threadIdx.x + i];
    }
    __syncthreads();
    i /= 2;
  }
  if(threadIdx.x == 0){
    atomicAdd(raysDump, raySum[threadIdx.x]);
  }
}

/**
 * @brief takes a number of rays and distributes them randomly over the available prisms
 *
 * @param *raysPerPrism the number of rays for each prism (will be changed)
 *
 * @param *raysDump the number of rays which were already distributed
 *
 * for other parameters, see documentation of importanceSampling()
 *
 */
__global__ void distributeRemainingRaysRandomly(
						Mesh *mesh,
						unsigned *raysPerPrism,
						unsigned raysPerSample,
						unsigned *raysDump){

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int raysLeft = raysPerSample-(*raysDump);

  if(id < raysLeft){
    hiprandState randomState;
    hiprand_init(id,0,0,&randomState);
    int rand_t = (int ) ceil(hiprand_uniform(&randomState) * mesh->numberOfTriangles) - 1;
    int rand_z = (int ) ceil(hiprand_uniform(&randomState) * (mesh->numberOfLevels-1)) - 1;
    atomicAdd(&raysPerPrism[rand_t + rand_z * mesh->numberOfTriangles],1);
  }
}


/**
 * @brief corrects the importance to match with the randomly distributed rays
 *
 * @param *raysPerPrism the number of rays to be launced for each prism
 *
 * @param *importance the importance for each prism (will be changed)
 *
 * for other parameters, see documentation of importanceSampling()
 */
__global__ void recalculateImportance(
				      Mesh *mesh,
				      unsigned *raysPerPrism,
				      unsigned raysPerSample,
				      double *importance){ 
  int startPrism = threadIdx.x + blockIdx.x * blockDim.x;
  if(startPrism >= mesh->numberOfPrisms){
    return;
  }
  int startLevel = startPrism/(mesh->numberOfTriangles);
  int startTriangle = startPrism - (mesh->numberOfTriangles * startLevel);
  if(raysPerPrism[startPrism] > 0){
    importance[startPrism] = raysPerSample * mesh->surfaces[startTriangle] / (mesh->surfaceTotal * raysPerPrism[startPrism]);
  }else{
    importance[startPrism] = 0;
  }
}


// unused, because we didn't find a good way to parallelize it...
// OPTIMIZE
// TODO
/**
 * @brief maps every ray to a specific prism
 *
 * @param *raysPerPrism the number of rays to launch in each prism
 *
 * @param raysPerSample the total number of rays to launch 
 *
 * @param *indicesOfPrisms a mapping for each ray to a specific prism
 *
 */
__global__ void mapRaysToPrism(
			       Mesh mesh,
			       unsigned *raysPerPrism,
			       unsigned raysPerSample,
			       unsigned *indicesOfPrisms){

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id==0){
    // Prism scheduling for gpu threads
    unsigned absoluteRay = 0;
    for(unsigned prism_i=0; prism_i < mesh->numberOfPrisms; ++prism_i){
      for(unsigned ray_i=0; ray_i < raysPerPrism[prism_i]; ++ray_i){
        indicesOfPrisms[absoluteRay++] = prism_i;
#if TEST_VALUES==true
        assert(absoluteRay <= raysPerSample);
#endif
      }
    }
  }
}

unsigned importanceSampling(
			    unsigned sample_i,
			    Mesh deviceMesh,
			    unsigned raysPerSample, 
			    double sigmaA, 
			    double sigmaE, 
			    double nTot,  
			    double *importance, 
			    float *sumPhi,
			    unsigned *raysPerPrism,
			    unsigned *indicesOfPrisms,
			    unsigned *raysDump,
			    int threads,
			    int blocks){

  float *sumPhiHost = (float*) malloc(sizeof(float));
  unsigned *raysDumpHost = (unsigned*) malloc(sizeof(unsigned));

  *sumPhiHost = 0.f;
  *raysDumpHost = 0;

  CUDA_CHECK_RETURN(hipMemcpy(sumPhi,sumPhiHost,sizeof(float),hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(raysDump,raysDumpHost,sizeof(unsigned),hipMemcpyHostToDevice));

  propagateFromTriangleCenter<<< blocks,threads >>>(&deviceMesh,importance,sumPhi,sample_i,sigmaA,sigmaE,nTot);
  distributeRaysByImportance<<< blocks,threads >>>(&deviceMesh,raysPerPrism,importance,sumPhi,raysPerSample,raysDump);
  distributeRemainingRaysRandomly<<< blocks,threads >>>(&deviceMesh,raysPerPrism,raysPerSample,raysDump);
  recalculateImportance<<< blocks,threads >>>(&deviceMesh,raysPerPrism,raysPerSample,importance);

  //  CUDA_CHECK_RETURN(hipMemcpy(hostRaysPerPrism,raysPerPrism, hostMesh.numberOfPrisms*sizeof(unsigned),hipMemcpyDeviceToHost));
  //
  //    // Prism scheduling for gpu threads
  //  for(unsigned prism_i=0, absoluteRay = 0; prism_i < hostMesh.numberOfPrisms; ++prism_i){
  //    for(unsigned ray_i=0; ray_i < hostRaysPerPrism[prism_i]; ++ray_i){
  //      hostIndicesOfPrisms[absoluteRay++] = prism_i;
  //      assert(absoluteRay <= hostRaysPerSample);
  //    }
  //  }
  //  // Copy dynamic sample data to device
  //  CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, hostRaysPerSample * sizeof(unsigned), hipMemcpyHostToDevice));

  return raysPerSample;
}
