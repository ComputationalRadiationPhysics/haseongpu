#include "mesh.h"
#include <stdio.h>
#include <cudachecks.h>

/**
 * @brief converts a vector of points into a vector of TwoDimPoint
 *
 * @param *points an array of points, containing numPoints x-values, followed by numPoints y-values
 *
 * @param numPoints the number of points which are stored
 *
 * @return an array of TwoDimPoint with the length numPoints 
 *
 */
TwoDimPoint* parsePoints(std::vector<double> *points, unsigned numPoints) {
  TwoDimPoint *p = new TwoDimPoint[numPoints];

  for(unsigned i=0; i < numPoints; ++i) {
    p[i].x = points->at(i);
    p[i].y = points->at(numPoints + i);
  }

  return p;
}

Mesh::~Mesh() {
  if(!triangles) delete triangles;
}

/**
 * @brief creates the Mesh datastructures on device and host for the propagation
 *
 * @param *hMesh the host mesh
 *
 * @param *dMesh the mesh on the device
 *
 * @param *triangleIndices indices of the points which form a triangle
 *
 * @param numberOfTriangles the number of triangles
 *
 * @param numberOfLeves the number of layers of the mesh
 *
 * @param numberOfPoints the number of vertices in one layer of the mesh
 *
 * @param thicknessOfPrism  the thickness of one layer of the mesh
 *
 * @param *pointXY coordinates of the vertices in one layer of the mesh
 * 
 * @param *betaValues constant values for each meshed prism
 *
 * @param *xOfTriangleCenter the x coordinates of each triangle's center
 *
 * @param *yOfTriangleCenter the y coordinates of each triangle's center
 *
 * @param *positionsOfNormalVectors indices to the points (pointXY), where the normals xOfNormals,yOfNormals start
 *
 * @param *xOfNormals the x components of a normal vector for each of the 3 sides of a triangle
 *
 * @param *yOfNormals the y components of a normal vector for each of the 3 sides of a triangle
 *
 * @param *forbidden the sides of the triangle from which a ray "entered" the triangle
 *
 * @param *neighbors indices to the adjacent triangles in triangleIndices
 *
 * @param *surfaces the sizes of the surface of each prism
 *
 */

/**
 * @brief fills the host mesh with the correct datastructures
 *
 * See parseMultiGPU for details on the parameters
 */
void fillHMesh(
    Mesh *hMesh,
    unsigned numberOfTriangles, 
    unsigned numberOfLevels,
    unsigned numberOfPoints, 
    float thicknessOfPrism
    ) {

  hMesh->numberOfTriangles = numberOfTriangles;
  hMesh->numberOfLevels = numberOfLevels;
  hMesh->numberOfPrisms = numberOfTriangles*(numberOfLevels-1);
  hMesh->numberOfPoints = numberOfPoints;
  hMesh->numberOfSamples = numberOfPoints * numberOfLevels;
  hMesh->thickness = thicknessOfPrism;
}

/**
 * @brief fills a device mesh with the correct datastructures
 *
 * See parseMultiGPU for details on the parameters
 */
void fillDMesh(
    Mesh *hMesh,
    Mesh *dMesh, 
    std::vector<unsigned> *triangleIndices, 
    unsigned numberOfTriangles, 
    unsigned numberOfLevels,
    unsigned numberOfPoints, 
    float thicknessOfPrism,
    TwoDimPoint *pointsVector, 
    std::vector<double> *xOfTriangleCenter, 
    std::vector<double> *yOfTriangleCenter, 
    std::vector<int> *positionsOfNormalVectors,
    std::vector<double> *xOfNormals, 
    std::vector<double> *yOfNormals,
    std::vector<int> *forbiddenVector, 
    std::vector<int> *neighborsVector, 
    std::vector<float> *surfacesVector,
	std:vector<double> *betaValuesVector
    ) {


  // GPU variables
  double totalSurface = 0.;

  // constants
  dMesh->numberOfTriangles = numberOfTriangles;
  dMesh->numberOfLevels = numberOfLevels;
  dMesh->numberOfPrisms = numberOfTriangles*(numberOfLevels-1);
  dMesh->numberOfPoints = numberOfPoints;
  dMesh->numberOfSamples = numberOfPoints*numberOfLevels;
  dMesh->thickness = thicknessOfPrism;

  for(unsigned i=0;i<numberOfTriangles;++i){
    totalSurface+=double(surfaces->at(i));	
  }
  dMesh->surfaceTotal = float(totalSurface);


  // values
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->points), 2 * numberOfPoints * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->normalVec), 2 * 3 * numberOfTriangles * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->betaValues), numberOfPrisms * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->centers), 2 * numberOfTriangles * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->surfaces), numberOfTriangles * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->forbidden), 3 * numberOfTriangles * sizeof(int)));

  // indexStructs
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->triangles), 3 * numberOfTriangles * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->neighbors), 3 * numberOfTriangles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMalloc(&(dMesh->normalPoint), 3 * numberOfTriangles * sizeof(unsigned)));


    /// fill values
  CUDA_CHECK_RETURN(hipMemcpy(dMesh->points, (double*) &(pointsVector->at(0)), 2 * numberOfPoints * sizeof(double), hipMemcpyHostToDevice));

  std::vector<double> *hostNormalVec = new std::vector<double>(xOfNormals);
  hostNormalVec->insert(hostNormalVec->end(),yOfNormals->begin(),yOfNormals->end());
  CUDA_CHECK_RETURN(hipMemcpy(dMesh->normalVec, (double*) &(hostNormalVec->at(0)), 2 * 3 * numberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
  free(hostNormalVec);

  CUDA_CHECK_RETURN(hipMemcpy(betaValues, (double*) &(betaValuesVector->at(0)), numberOfPrisms * sizeof(double), hipMemcpyHostToDevice));

  std::vector<double> hostCenters = new std::vector<double>(xOfTriangleCenter);
  hostCenters.insert(hostCenters->end(),yOfTriangleCenter->begin(),yOfTriangleCenter->end());
  CUDA_CHECK_RETURN(hipMemcpy(dMesh->centers, (double*) &(hostCenters->at(0)), 2 * numberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
  free(hostCenters);

  CUDA_CHECK_RETURN(hipMemcpy(dMesh->surfaces, (float*) &(surfacesVector->at(0)), numberOfTriangles * sizeof(double), hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMemcpy(dMesh->forbidden, (int*) &(forbiddenVector->at(0)), 3 * numberOfTriangles * sizeof(int), hipMemcpyHostToDevice));



  // fill indexStructs
  CUDA_CHECK_RETURN(hipMemcpy(dMesh->triangles, (unsigned*) &(triangleIndices->at(0)), 3 * numberOfTriangles * sizeof(int), hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMemcpy(dMesh->neighbors,(int*) &(neighborsVector->at(0)), 3 * numberOfTriangles * sizeof(int), hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMemcpy(dMesh->normalPoint, (unsigned*) &(positionsOfNormalVectors->at(0)), 3 * numberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
  
}

__device__ int Mesh::getNeighbor(unsigned triangle, unsigned edge){
	return neighbors[triangle + edge*numberOfTriangles];
}

__device__ Point Mesh::genRndPoint(unsigned triangle, unsigned level, hiprandStateMtgp32 *globalState){
	Point startPoint = {0,0,0};
	double u = hiprand_uniform(&globalState[blockIdx.x]);
	double v = hiprand_uniform(&globalState[blockIdx.x]);

	if((u+v)>1)
	{
		u = 1-u;
		v = 1-v;
	}
	double w = 1-u-v;
	int t1 = triangleIndices[triangle];
	int t2 = triangleIndices[triangle + numberOfTriangles];
	int t3 = triangleIndices[triangle + 2 * numberOfTriangles];

	// convert the random startpoint into coordinates
	startPoint.z = (level + hiprand_uniform(&globalState[blockIdx.x])) * thickness;
	startPoint.x = (points[t1] * u) + (points[t2] * v) + (points[t3] * w);
	startPoint.y = (points[t1+numberOfPoints] * u) + (points[t2+numberOfPoints] * v) + (points[t3+numberOfPoints] * w);

	return startPoint;
}
  
double Mesh::getBetaValue(unsigned triangle, unsigned level){
	return betaValues[triangle + level*numberOfTriangles];
}

double Mesh::getBetaValue(unsigned prism){
	return betaValues[prism];
}

NormalRay Mesh::getNormal(unsigned triangle, int edge){
	NormalRay ray = { {0,0},{0,0}};
	int offset =  edge*numberOfTriangles + triangle;
	ray.p.x = points[ normalPoint [offset] ];
	ray.p.y = points[ normalPoint [offset] + numberOfPoints ];

	ray.d.x = normalVec[offset];
	ray.d.y = normalVec[offset + 3*numberOfTriangles];

	return ray;
}	

Point Mesh::getSamplePoint(unsigned sample){
	Point p = {0,0,0};
	unsigned level = sample/numberOfPoints;
	p.z = level*thickness;
	unsigned pos = sample - (numberOfPoints*level);
	p.x = points[pos];
	p.y = points[pos + numberOfPoints]
	return p;
}

Point Mesh::getCenterPoint(unsigned triangle,unsigned level){
	Point p = {0,0,(level+0.5)*thickness};
	p.x = centers[triangle];
	p.y = centers[triangle + numberOfTriangles];
	return p;
}





/**
 * @brief creates the Mesh datastructures on the host and on all possible devices for the propagation
 *
 * @param *hMesh the host mesh
 *
 * @param **dMesh an array of device meshes (one for each device) 
 *
 * @param *triangleIndices indices of the points which form a triangle
 *
 * @param numberOfTriangles the number of triangles
 *
 * @param numberOfLeves the number of layers of the mesh
 *
 * @param numberOfPoints the number of vertices in one layer of the mesh
 *
 * @param thicknessOfPrism  the thickness of one layer of the mesh
 *
 * @param *pointXY coordinates of the vertices in one layer of the mesh
 * 
 * @param *betaValues constant values for each meshed prism
 *
 * @param *xOfTriangleCenter the x coordinates of each triangle's center
 *
 * @param *yOfTriangleCenter the y coordinates of each triangle's center
 *
 * @param *positionsOfNormalVectors indices to the points (pointXY), where the normals xOfNormals,yOfNormals start
 *
 * @param *xOfNormals the x components of a normal vector for each of the 3 sides of a triangle
 *
 * @param *yOfNormals the y components of a normal vector for each of the 3 sides of a triangle
 *
 * @param *forbidden the sides of the triangle from which a ray "entered" the triangle
 *
 * @param *neighbors indices to the adjacent triangles in triangleIndices
 *
 * @param *surfaces the sizes of the surface of each prism
 *
 * @param numberOfDevices number of devices in *devices
 *
 * @param *devices array of device indices for all possible devices 
 *
 */
void Mesh::parseMultiGPU(
    Mesh *hMesh,
    Mesh **dMesh, 
    std::vector<unsigned> *triangleIndices, 
    unsigned numberOfTriangles, 
    unsigned numberOfLevels,
    unsigned numberOfPoints, 
    float thicknessOfPrism,
    std::vector<double> *pointXY, 
    std::vector<double> *betaValues, 
    std::vector<double> *xOfTriangleCenter, 
    std::vector<double> *yOfTriangleCenter, 
    std::vector<int> *positionsOfNormalVectors,
    std::vector<double> *xOfNormals, 
    std::vector<double> *yOfNormals,
    std::vector<int> *forbidden, 
    std::vector<int> *neighbors, 
    std::vector<float> *surfaces,
    unsigned numberOfDevices,
    unsigned *devices) {

  TwoDimPoint *points = parsePoints(pointXY, numberOfPoints);

  fillHMesh(
      hMesh,
      numberOfTriangles, 
      numberOfLevels,
      numberOfPoints, 
      thicknessOfPrism
      );

 for( unsigned i=0;i<numberOfDevices;i++){
  CUDA_CHECK_RETURN( hipSetDevice(devices[i]) );
  fillDMesh(
      hMesh,
      &((*dMesh)[i]),
      triangleIndices, 
      numberOfTriangles, 
      numberOfLevels,
      numberOfPoints, 
      thicknessOfPrism,
      points, 
      xOfTriangleCenter, 
      yOfTriangleCenter, 
      positionsOfNormalVectors,
      xOfNormals, 
      yOfNormals,
      forbidden, 
      neighbors, 
      surfaces,
	  betaValues
      );
  hipDeviceSynchronize();
 }

/**
 * @brief fills a device mesh with the correct datastructures
 *
 * See parseMultiGPU for details on the parameters
 */
void fillDMeshOLD(
    Mesh *hMesh,
    Mesh *dMesh, 
    std::vector<unsigned> *triangleIndices, 
    unsigned numberOfTriangles, 
    unsigned numberOfLevels,
    unsigned numberOfPoints, 
    float thicknessOfPrism,
    TwoDimPoint *points, 
    std::vector<double> *xOfTriangleCenter, 
    std::vector<double> *yOfTriangleCenter, 
    std::vector<int> *positionsOfNormalVectors,
    std::vector<double> *xOfNormals, 
    std::vector<double> *yOfNormals,
    std::vector<int> *forbidden, 
    std::vector<int> *neighbors, 
    std::vector<float> *surfaces
    ) {

  double totalSurface = 0.;

  // constants
  dMesh->numberOfTriangles = numberOfTriangles;
  dMesh->numberOfLevels = numberOfLevels;
  dMesh->numberOfPrisms = numberOfTriangles*(numberOfLevels-1);
  dMesh->numberOfPoints = numberOfPoints;
  dMesh->numberOfSamples = numberOfPoints*numberOfLevels;
  dMesh->thickness = thicknessOfPrism;

  

  CUDA_CHECK_RETURN( hipMalloc((void**) &dMesh->samples, numberOfPoints*numberOfLevels*sizeof(Point)) );
  CUDA_CHECK_RETURN( hipMemcpy(dMesh->samples, hMesh->samples, numberOfPoints*numberOfLevels*sizeof(Point), hipMemcpyHostToDevice) );

  Triangle *trianglesForDevice = new Triangle[numberOfTriangles];
  CUDA_CHECK_RETURN( hipMalloc((void**) &dMesh->triangles, numberOfTriangles*sizeof(Triangle)) );

  double totalSurface = 0;
  for(unsigned i=0; i<numberOfTriangles; ++i) {
    Triangle triangle;
    triangle.A = points[triangleIndices->at(i)];
    triangle.B = points[triangleIndices->at(numberOfTriangles + i)];
    triangle.C = points[triangleIndices->at(2*numberOfTriangles + i)];

    TwoDimPoint center = {xOfTriangleCenter->at(i), yOfTriangleCenter->at(i)};
    triangle.center = center;
    triangle.surface = surfaces->at(i);
    totalSurface += triangle.surface;

    trianglesForDevice[i] = triangle;

    for(unsigned e=0; e<3; ++e) {
      NormalRay normal;
      normal.p = points[positionsOfNormalVectors->at(e*numberOfTriangles + i)];
      normal.dir.x = xOfNormals->at( e*numberOfTriangles + i );
      normal.dir.y = yOfNormals->at( e*numberOfTriangles + i );

      Edge edge;
      edge.normal = normal;
      edge.forbidden = forbidden->at( e*numberOfTriangles + i);

      edge.neighbor = &(dMesh->triangles[neighbors->at( e*numberOfTriangles + i)]);
      trianglesForDevice[i].edges[e] = edge;
    }

    CUDA_CHECK_RETURN( hipMalloc((void**) &trianglesForDevice[i].betaValues, (numberOfLevels-1)*sizeof(double)) );
    CUDA_CHECK_RETURN( hipMemcpy(trianglesForDevice[i].betaValues, hMesh->triangles[i].betaValues, (numberOfLevels-1)*sizeof(double), hipMemcpyHostToDevice) );
  }
  dMesh->surface = totalSurface;

  CUDA_CHECK_RETURN( hipMemcpy(dMesh->triangles, trianglesForDevice, numberOfTriangles*sizeof(Triangle), hipMemcpyHostToDevice) );

}
}
