#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <write_to_vtk.h>
#include <calc_phi_ase.h>
#include <map_rays_to_prisms.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <calc_sample_phi_ase.h>
#include <mesh.h>
#include <progressbar.h> /*progressBar */
#include <logging.h>
#include <types.h>

#define SEED 4321
#define RAY_STEPS 5

double calcMSE(const double phiAse, const double phiAseSquare, const unsigned raysPerSample){
  double a = phiAseSquare / raysPerSample;
  double b = (phiAse / raysPerSample) * (phiAse / raysPerSample);

  return sqrt(abs((a - b) / raysPerSample));
}

std::vector<int> generateRaysPerSampleLinList(int minRaysPerSample, int maxRaysPerSample, int steps){
  std::vector<int> raysPerSample;

  raysPerSample.push_back(minRaysPerSample);
  if(minRaysPerSample == maxRaysPerSample)
    return raysPerSample;

  int step_wide = (maxRaysPerSample - minRaysPerSample) / steps;

  for(int i = 0; i < steps - 1; ++i){
    minRaysPerSample += step_wide;
    raysPerSample.push_back(minRaysPerSample);

  }
  raysPerSample.push_back(maxRaysPerSample);
  
  return raysPerSample;

}

std::vector<int> generateRaysPerSampleExpList(int minRaysPerSample, int maxRaysPerSample, int steps){
  std::vector<int> raysPerSample;

  if((minRaysPerSample == maxRaysPerSample) || steps < 2){
    raysPerSample.push_back(minRaysPerSample);
    return raysPerSample;
  }

  for(int i = 0; i < steps; ++i){
    int step_val = minRaysPerSample * pow((maxRaysPerSample / minRaysPerSample), (i / (float)(steps - 1)));
    raysPerSample.push_back(step_val);

  }
  
  return raysPerSample;

}

float calcPhiAse (const unsigned hMinRaysPerSample,
		  const unsigned maxRaysPerSample,
		  const unsigned maxRepetitions,
		  const Mesh& dMesh,
		  const Mesh& hMesh,
		  const std::vector<double>& hSigmaA,
		  const std::vector<double>& hSigmaE,
		  const std::vector<float>& mseThreshold,
		  const bool useReflections,
		  std::vector<float> &phiAse,
		  std::vector<double> &mse,
		  std::vector<unsigned> &totalRays,
		  const unsigned gpu_i,
		  const unsigned minSample_i,
		  const unsigned maxSample_i,
		  float &runtime){

  // Optimization to use more L1 cache
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  hipSetDevice(gpu_i);

  using thrust::device_vector;
  using thrust::raw_pointer_cast;

  // variable Definitions CPU
  time_t starttime                = time(0);
  unsigned maxReflections         = useReflections ? hMesh.getMaxReflections() : 0;
  unsigned reflectionSlices       = 1 + (2 * maxReflections);
  unsigned numberOfWavelengths    = hSigmaE.size();
  // In some cases distributeRandomly has to be true !
  // Otherwise bad or no ray distribution possible.
  bool distributeRandomly         = true;
  dim3 blockDim(128);             //can't be more than 256 due to restrictions from the Mersenne Twister
  dim3 gridDim(200);              //can't be more than 200 due to restrictions from the Mersenne Twister

  // Divide RaysPerSample range into steps
  std::vector<int>  raysPerSampleList = generateRaysPerSampleExpList(hMinRaysPerSample, maxRaysPerSample, RAY_STEPS);
  std::vector<int>::iterator raysPerSampleIter = raysPerSampleList.begin();

  // DEBUG
  // for(;raysPerSampleIter != raysPerSampleList.end(); raysPerSampleIter++){
  //   dout(V_DEBUG) << "RayStep " << *raysPerSampleIter << std::endl;
  // }

  raysPerSampleIter = raysPerSampleList.begin();

  // Memory allocation/init and copy for device memory
  device_vector<unsigned> dNumberOfReflections(maxRaysPerSample, 0);
  device_vector<float>    dGainSum            (1, 0);
  device_vector<float>    dGainSumSquare      (1, 0);
  device_vector<unsigned> dLostRays           (1, 0); // OPTIMIZE: remove unnecessary parameter
  device_vector<unsigned> dRaysPerPrism       (hMesh.numberOfPrisms * reflectionSlices, 1);
  device_vector<unsigned> dPrefixSum          (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<double>   dImportance         (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<double>   dImportanceSave     (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<unsigned> dIndicesOfPrisms    (maxRaysPerSample,  0);

  // DEBUG
  // OUTPUT DATA
  // thrust::host_vector<unsigned> hNumberOfReflections(maxRaysPerSample,0);
  // thrust::host_vector<unsigned> hIndicesOfPrisms(maxRaysPerSample,0);
  //thrust::host_vector<unsigned> hRaysPerPrism(hMesh.numberOfPrisms * reflectionSlices, 0);
  //unsigned midRaysPerSample=0;

  // CUDA Mersenne twister (can not have more than 200 blocks!)
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.x  * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, gridDim.x, SEED + minSample_i));

  // Calculate Phi Ase for each wavelength
  for(unsigned wave_i = 0; wave_i < numberOfWavelengths; ++wave_i){

    // Calculation for each sample point
    for(unsigned sample_i = minSample_i; sample_i < maxSample_i; ++sample_i){
      // DEBUG
    //unsigned sample_i = 4;{
      unsigned sampleOffset  = sample_i + hMesh.numberOfSamples * wave_i;
      unsigned hRaysPerSampleDump = 0; 
      raysPerSampleIter = raysPerSampleList.begin();
      bool mseTooHigh=true;

      float hSumPhi = importanceSamplingPropagation(
          sample_i,
          reflectionSlices,
          dMesh,
          hMesh.numberOfPrisms,
          hSigmaA[wave_i],
          hSigmaE[wave_i],
          dImportanceSave);


      while(mseTooHigh){
        CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, gridDim.x, SEED + sample_i));
        unsigned run = 0;
        while(run < maxRepetitions && mseTooHigh){
          run++;
          dLostRays[0] = 0;

          thrust::copy(dImportanceSave.begin(),dImportanceSave.end(),dImportance.begin());
          hRaysPerSampleDump = importanceSamplingDistribution(
							      reflectionSlices,
							      dMesh,
							      hMesh.numberOfPrisms,
							      *raysPerSampleIter,
							      dImportance, 
							      dRaysPerPrism,
							      hSumPhi,
							      distributeRandomly);
	  
	  // DEBUG
          // if(dRaysPerPrism[6495] > 10000){
          //   dout(V_DEBUG) << "Too high raysPerprism " << dRaysPerPrism[6495] << " sample_i: " << sample_i <<std::endl;
          //   exit(0);
          // }

          // Prism scheduling for gpu threads
          mapRaysToPrisms(dIndicesOfPrisms, dNumberOfReflections, dRaysPerPrism, dPrefixSum, reflectionSlices, hRaysPerSampleDump, hMesh.numberOfPrisms);

          // DEBUG
          // if(sample_i == 0){
          //   thrust::copy(dNumberOfReflections.begin(),dNumberOfReflections.end(),hNumberOfReflections.begin());
          //   thrust::copy(dIndicesOfPrisms.begin(),dIndicesOfPrisms.end(),hIndicesOfPrisms.begin());
          // thrust::copy(dRaysPerPrism.begin(), dRaysPerPrism.end(), hRaysPerPrism.begin());
          //   midRaysPerSample=hMinRaysPerSample;
          // }

          // Start Kernel
          dGainSum[0]       = 0;
          dGainSumSquare[0] = 0;

          if(useReflections){
            calcSampleGainSum<<< gridDim, blockDim >>>( devMTGPStates,
                dMesh, 
                raw_pointer_cast(&dIndicesOfPrisms[0]), 
                wave_i, 
                raw_pointer_cast(&dNumberOfReflections[0]), 
                raw_pointer_cast(&dImportance[0]),
                hRaysPerSampleDump, 
                raw_pointer_cast(&dGainSum[0]), 
                raw_pointer_cast(&dGainSumSquare[0]),
                raw_pointer_cast(&dLostRays[0]),
                sample_i, 
                hSigmaA[wave_i], 
                hSigmaE[wave_i],
                raw_pointer_cast(&(device_vector<unsigned> (1,0))[0]));
          }
          else{
            calcSampleGainSumWithoutReflections<<< gridDim, blockDim >>>( devMTGPStates,
                dMesh, 
                raw_pointer_cast(&dIndicesOfPrisms[0]), 
                wave_i, 
                raw_pointer_cast(&dImportance[0]),
                hRaysPerSampleDump, 
                raw_pointer_cast(&dGainSum[0]), 
                raw_pointer_cast(&dGainSumSquare[0]),
                sample_i, 
                hSigmaA[wave_i], 
                hSigmaE[wave_i],
                raw_pointer_cast(&(device_vector<unsigned> (1,0))[0]));
          }

	  // Remove lost rays (reflections) from ray counter
    // Don't do this, if you want MonteCarlo to work properly!!
	  //hRaysPerSampleDump -= dLostRays[0];

          float mseTmp = calcMSE(dGainSum[0], dGainSumSquare[0], hRaysPerSampleDump);

          // DEBUG
          if(isnan(mseTmp)){
            dout(V_ERROR) << "mseTmp: " << mseTmp << " gainSum:" << dGainSum[0] << " gainSum²:" << dGainSumSquare[0] << " RaysPerSample:" << hRaysPerSampleDump <<std::endl;
          }

          assert(!isnan(dGainSum[0]));
          assert(!isnan(dGainSumSquare[0]));
          assert(!isnan(mseTmp));

	  // DEBUG
          //MSE TESTs
          // if(mseTmp > mse.at(sampleOffset)){
          //   // this happens in calcMSE
          //   double ca = dGainSumSquare[sampleOffset] / hRaysPerSampleDump;
          //   double cb = (dGainSum[sampleOffset] / hRaysPerSampleDump) * (dGainSum[sampleOffset] / hRaysPerSampleDump);

          //   dout(V_WARNING) << "MSE_BUG for sample " << sample_i << ": " << mseTmp << " > " << mse.at(sampleOffset) << std::endl;
          //   dout(V_DEBUG) << "Run: " << run << std::endl;
          //   dout(V_DEBUG) << "RaysPerSample: " << hRaysPerSample << std::endl;
          //   dout(V_DEBUG) << "RaysPerSampleDump: "<< hRaysPerSampleDump << std::endl;
          //   dout(V_DEBUG) << "phiAseSquare / raysPerSample = " << ca << std::endl; 
          //   dout(V_DEBUG) << "(phiAse / raysPerSample) * (phiAse / raysPerSample) = " << cb << std::endl; 
          //   dout(V_DEBUG) << "sqrt(abs((a - b) / raysPerSample)) = " << sqrt(abs((ca - cb) / hRaysPerSampleDump)) << std::endl; 
          //   dout(V_DEBUG) << std::endl;
          // }

          //mse.at(sampleOffset) = mseTmp;
          if(mse.at(sampleOffset) > mseTmp){
	    mse.at(sampleOffset) = mseTmp;
	    phiAse.at(sampleOffset) = dGainSum[0]; 
	    phiAse.at(sampleOffset)   /= *raysPerSampleIter * 4.0f * M_PI;
            totalRays.at(sampleOffset) = *raysPerSampleIter;
          }
          if(mse.at(sampleOffset) < mseThreshold.at(wave_i)) mseTooHigh = false;
        }

	// Increase rays per sample or break, when mseThreshold was not met
	++raysPerSampleIter;
	if(raysPerSampleIter == raysPerSampleList.end())
	  break;
	  
      }
      // Update progressbar
      fancyProgressBar(maxSample_i);
    }
    
  }
  
  // DEBUG
  // JUST OUTPUT
  // std::vector<unsigned> reflectionsPerPrism(hMesh.numberOfPrisms, 0);
  // std::vector<unsigned> raysPerPrism(hMesh.numberOfPrisms, 0);

  // for(unsigned i=0; i < midRaysPerSample; ++i){
  //   unsigned index = hIndicesOfPrisms[i];
  //   reflectionsPerPrism[index] = max(reflectionsPerPrism[index], (hNumberOfReflections[i] + 1) / 2);
  // }

  // for(unsigned i=0; i < hMesh.numberOfPrisms; ++i){
  //   for(unsigned j=0; j < reflectionSlices; ++j){
  //     unsigned index = i + hMesh.numberOfPrisms * j;
  //     raysPerPrism[i] += hRaysPerPrism[index];
  //   }
  // }

  //writePrismToVtk(hMesh, reflectionsPerPrism, "octrace_0_reflections", hMinRaysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, 0);
  //writePrismToVtk(hMesh, raysPerPrism, "octrace_0_rays", hMinRaysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, 0);

  //dout(V_INFO | V_NOLABEL) << "\n" << std::endl;
  // Free Memory
  hipFree(devMTGPStates);
  hipFree(devKernelParams);


  runtime = difftime(time(0),starttime);
  return runtime;
}
