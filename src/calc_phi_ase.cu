#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <write_to_vtk.h>
#include <calc_phi_ase.h>
#include <map_rays_to_prisms.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <calc_sample_phi_ase.h>
#include <mesh.h>
#include <progressbar.h> /*progressBar */
#include <logging.h>
#include <types.h>

#define SEED 4321
#define RAY_STEPS 5

double calcMSE(const double phiAse, const double phiAseSquare, const unsigned raysPerSample){
  double a = phiAseSquare / raysPerSample;
  double b = (phiAse / raysPerSample) * (phiAse / raysPerSample);

  return sqrt(abs((a - b) / raysPerSample));
}

std::vector<int> generateRaysPerSampleExpList(int minRaysPerSample, int maxRaysPerSample, int steps){
  std::vector<int> raysPerSample;

  if((minRaysPerSample == maxRaysPerSample) || steps < 2){
    raysPerSample.push_back(minRaysPerSample);
    return raysPerSample;
  }

  for(int i = 0; i < steps; ++i){
    int step_val = minRaysPerSample * pow((maxRaysPerSample / minRaysPerSample), (i / (float)(steps - 1)));
    raysPerSample.push_back(step_val);

  }
  
  return raysPerSample;

}

float calcPhiAse (const unsigned hMinRaysPerSample,
		  const unsigned maxRaysPerSample,
		  const unsigned maxRepetitions,
		  const Mesh& dMesh,
		  const Mesh& hMesh,
		  const std::vector<double>& hSigmaA,
		  const std::vector<double>& hSigmaE,
		  const std::vector<float>& mseThreshold,
		  const bool useReflections,
		  std::vector<float> &phiAse,
		  std::vector<double> &mse,
		  std::vector<unsigned> &totalRays,
		  const unsigned gpu_i,
		  const unsigned minSample_i,
		  const unsigned maxSample_i,
		  float &runtime){

  // Optimization to use more L1 cache
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  hipSetDevice(gpu_i);

  using thrust::device_vector;
  using thrust::raw_pointer_cast;

  // variable Definitions CPU
  time_t starttime                = time(0);
  unsigned maxReflections         = useReflections ? hMesh.getMaxReflections() : 0;
  unsigned reflectionSlices       = 1 + (2 * maxReflections);
  unsigned numberOfWavelengths    = hSigmaE.size();
  // In some cases distributeRandomly has to be true !
  // Otherwise bad or no ray distribution possible.
  bool distributeRandomly         = true;
  dim3 blockDim(128);             //can't be more than 256 due to restrictions from the Mersenne Twister
  dim3 gridDim(200);              //can't be more than 200 due to restrictions from the Mersenne Twister

  // Divide RaysPerSample range into steps
  std::vector<int>  raysPerSampleList = generateRaysPerSampleExpList(hMinRaysPerSample, maxRaysPerSample, RAY_STEPS);
  std::vector<int>::iterator raysPerSampleIter = raysPerSampleList.begin();

  // Memory allocation/init and copy for device memory
  device_vector<unsigned> dNumberOfReflectionSlices(maxRaysPerSample, 0);
  device_vector<float>    dGainSum            (1, 0);
  device_vector<float>    dGainSumSquare      (1, 0);
  device_vector<unsigned> dRaysPerPrism       (hMesh.numberOfPrisms * reflectionSlices, 1);
  device_vector<unsigned> dPrefixSum          (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<double>   dImportance         (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<double>   dPreImportance      (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<unsigned> dIndicesOfPrisms    (maxRaysPerSample,  0);
  device_vector<double>   dSigmaA             (hSigmaA.begin(),hSigmaA.end());
  device_vector<double>   dSigmaE             (hSigmaE.begin(),hSigmaE.end());

  // CUDA Mersenne twister (can not have more than 200 blocks!)
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.x  * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, gridDim.x, SEED + minSample_i));

  // Calculate Phi Ase for each wavelength
  for(unsigned wave_i = 0; wave_i < numberOfWavelengths; ++wave_i){

    // Calculation for each sample point
    for(unsigned sample_i = minSample_i; sample_i < maxSample_i; ++sample_i){
      unsigned sampleOffset  = sample_i + hMesh.numberOfSamples * wave_i;
      unsigned hRaysPerSampleDump = 0; 
      raysPerSampleIter = raysPerSampleList.begin();
      bool mseTooHigh=true;

      importanceSamplingPropagation(sample_i,
				    reflectionSlices,
				    dMesh,
				    hSigmaA[wave_i],
				    hSigmaE[wave_i],
				    raw_pointer_cast(&dPreImportance[0]), 
				    blockDim,
				    gridDim);

      float hSumPhi = thrust::reduce(dPreImportance.begin(), dPreImportance.end(),0.);

      while(mseTooHigh){
        CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, gridDim.x, SEED + sample_i));
        unsigned run = 0;
        while(run < maxRepetitions && mseTooHigh){
          run++;

	  thrust::copy(dPreImportance.begin(),dPreImportance.end(),dImportance.begin());
          hRaysPerSampleDump = importanceSamplingDistribution(reflectionSlices,
							      dMesh,
							      *raysPerSampleIter,
							      raw_pointer_cast(&dImportance[0]), 
							      raw_pointer_cast(&dRaysPerPrism[0]),
							      hSumPhi,
							      distributeRandomly,
							      blockDim,
							      gridDim);
          
          // Prism scheduling for gpu threads
          mapRaysToPrisms(dIndicesOfPrisms, dNumberOfReflectionSlices, dRaysPerPrism, dPrefixSum, reflectionSlices, hRaysPerSampleDump, hMesh.numberOfPrisms);

          // Start Kernel
          dGainSum[0]       = 0;
          dGainSumSquare[0] = 0;

          if(useReflections){
            calcSampleGainSumWithReflection<<< gridDim, blockDim >>>(devMTGPStates,
								     dMesh, 
								     raw_pointer_cast(&dIndicesOfPrisms[0]), 
								     wave_i, 
								     raw_pointer_cast(&dNumberOfReflectionSlices[0]), 
								     raw_pointer_cast(&dImportance[0]),
								     hRaysPerSampleDump, 
								     raw_pointer_cast(&dGainSum[0]), 
								     raw_pointer_cast(&dGainSumSquare[0]),
								     sample_i, 
								     //hSigmaA[wave_i], 
								     //hSigmaE[wave_i],
                     raw_pointer_cast(&dSigmaA[0]),
                     raw_pointer_cast(&dSigmaE[0]),
                     hSigmaA.size(),
								     raw_pointer_cast(&(device_vector<unsigned> (1,0))[0]));
          }
          else{
            calcSampleGainSum<<< gridDim, blockDim >>>(devMTGPStates,
						       dMesh, 
						       raw_pointer_cast(&dIndicesOfPrisms[0]), 
						       wave_i, 
						       raw_pointer_cast(&dImportance[0]),
						       hRaysPerSampleDump, 
						       raw_pointer_cast(&dGainSum[0]), 
						       raw_pointer_cast(&dGainSumSquare[0]),
						       sample_i, 
						       //hSigmaA[wave_i], 
						       //hSigmaE[wave_i],
                   raw_pointer_cast(&dSigmaA[0]),
                   raw_pointer_cast(&dSigmaE[0]),
                   hSigmaA.size(),
						       raw_pointer_cast(&(device_vector<unsigned> (1,0))[0]));
          }

          float mseTmp = calcMSE(dGainSum[0], dGainSumSquare[0], hRaysPerSampleDump);

          assert(!isnan(dGainSum[0]));
          assert(!isnan(dGainSumSquare[0]));
          assert(!isnan(mseTmp));

          if(mse.at(sampleOffset) > mseTmp){
	    mse.at(sampleOffset) = mseTmp;
	    phiAse.at(sampleOffset) = dGainSum[0]; 
	    phiAse.at(sampleOffset)   /= *raysPerSampleIter * 4.0f * M_PI;
            totalRays.at(sampleOffset) = *raysPerSampleIter;
          }
          if(mse.at(sampleOffset) < mseThreshold.at(wave_i)) mseTooHigh = false;
        }

	// Increase rays per sample or break, when mseThreshold was not met
	raysPerSampleIter++;
	if(raysPerSampleIter == raysPerSampleList.end())
	  break;
	  
      }
      // Update progressbar
      if(verbosity & V_PROGRESS){
        fancyProgressBar(hMesh.numberOfSamples);
      }
    }

  }
  
  // Free Memory
  hipFree(devMTGPStates);
  hipFree(devKernelParams);


  runtime = difftime(time(0),starttime);
  return runtime;
}
