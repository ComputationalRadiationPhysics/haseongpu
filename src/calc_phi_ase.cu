#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <hip/hip_runtime_api.h>
#include <ctime> /* progressBar */
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <write_to_vtk.h>
#include <calc_phi_ase.h>
#include <map_rays_to_prisms.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <calc_sample_phi_ase.h>
#include <mesh.h>
#include <progressbar.h> /*progressBar */
#include <logging.h>

#define SEED 1234

double calcExpectation(const double phiAse, const double phiAseSquare, const unsigned raysPerSample){
  double a = phiAseSquare / raysPerSample;
  double b = (phiAse / raysPerSample) * (phiAse / raysPerSample);

  return sqrt(abs((a - b) / raysPerSample));
}


float calcPhiAse ( unsigned &hRaysPerSample,
		   const unsigned maxRaysPerSample,
		   const Mesh& dMesh,
		   const Mesh& hMesh,
		   const std::vector<double>& hSigmaA,
		   const std::vector<double>& hSigmaE,
		   const std::vector<float>& mseThreshold,
		   const bool useReflections,
		   std::vector<float> &hPhiAse,
		   std::vector<double> &mse,
		   std::vector<unsigned> &totalRays,
		   unsigned gpu_i,
		   unsigned minSample_i,
		   unsigned maxSample_i,
		   float &runtime){

  // Optimization to use more L1 cache
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  hipSetDevice(gpu_i);

  using thrust::device_vector;
  using thrust::raw_pointer_cast;

  // variable Definitions CPU
  time_t starttime                = time(0);
  unsigned hRaysPerSampleSave     = hRaysPerSample;
  unsigned maxReflections         = useReflections ? hMesh.getMaxReflections() : 0;
  unsigned reflectionSlices       = 1 + (2 * maxReflections);
  unsigned numberOfWavelengths    = hSigmaE.size();
  bool distributeRandomly         = true;
  dim3 blockDim(128);             
  dim3 gridDim(200);              //can't be more than 200 due to restrictions from the Mersenne Twister

  // Memory allocation/init and copy for device memory
  device_vector<unsigned> dNumberOfReflections(maxRaysPerSample,  0);
  device_vector<unsigned> dIndicesOfPrisms    (maxRaysPerSample,  0);
  device_vector<float>    dPhiAse             (hPhiAse.size(), 0);
  device_vector<unsigned> dRaysPerPrism       (hMesh.numberOfPrisms * reflectionSlices, 1);
  device_vector<unsigned> dPrefixSum          (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<double>   dImportance         (hMesh.numberOfPrisms * reflectionSlices, 0);
  device_vector<float>    dPhiAseSquare       (hMesh.numberOfSamples * numberOfWavelengths, 0); //OPTIMIZE: use only 1 value

  // OUTPUT DATA
  // thrust::host_vector<unsigned> hNumberOfReflections(maxRaysPerSample,0);
  // thrust::host_vector<unsigned> hIndicesOfPrisms(maxRaysPerSample,0);
  //thrust::host_vector<unsigned> hRaysPerPrism(hMesh.numberOfPrisms * reflectionSlices, 0);
  //unsigned midRaysPerSample=0;
 
  // CUDA Mersenne twister (can not have more than 200 blocks!)
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.x  * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, gridDim.x, SEED));

  // Calculate Phi Ase for each wavelength
  for(unsigned wave_i = 0; wave_i < numberOfWavelengths; ++wave_i){
    //time_t progressStartTime = time(0);
    //calculation for each sample point
    for(unsigned sample_i = minSample_i; sample_i < maxSample_i; ++sample_i){
      float mseRunZero = 0.0;
      // MSE BUG TEST
      //for(unsigned sample_i = 71; sample_i < 72; ++sample_i){
      //dout(V_DEBUG) << "SAMPLE " << sample_i << std::endl;
      unsigned sampleOffset  = sample_i + hMesh.numberOfSamples * wave_i;
      hRaysPerSample = hRaysPerSampleSave;

      unsigned hRaysPerSampleDump = 0; while(true){
	unsigned run = 0;
	hRaysPerSampleDump = importanceSampling(
						sample_i, reflectionSlices, dMesh, hRaysPerSample, hSigmaA[wave_i], hSigmaE[wave_i],
						raw_pointer_cast(&dImportance[0]), 
						raw_pointer_cast(&dRaysPerPrism[0]),
						distributeRandomly, blockDim, gridDim
						);

        // Prism scheduling for gpu threads
        mapRaysToPrisms(dIndicesOfPrisms, dNumberOfReflections, dRaysPerPrism, dPrefixSum, reflectionSlices, hRaysPerSampleDump, hMesh.numberOfPrisms);

	// OUTPUT DATA
        // if(sample_i == 0){
        //   thrust::copy(dNumberOfReflections.begin(),dNumberOfReflections.end(),hNumberOfReflections.begin());
        //   thrust::copy(dIndicesOfPrisms.begin(),dIndicesOfPrisms.end(),hIndicesOfPrisms.begin());
	// thrust::copy(dRaysPerPrism.begin(), dRaysPerPrism.end(), hRaysPerPrism.begin());
        //   midRaysPerSample=hRaysPerSample;
        // }

        // Start Kernel
        calcSamplePhiAse<<< gridDim, blockDim >>>( devMTGPStates,
						   dMesh, 
						   raw_pointer_cast(&dIndicesOfPrisms[0]), 
						   wave_i, 
						   raw_pointer_cast(&dNumberOfReflections[0]), 
						   raw_pointer_cast(&dImportance[0]),
						   hRaysPerSampleDump, 
						   raw_pointer_cast(&dPhiAse[0]), 
						   raw_pointer_cast(&dPhiAseSquare[0]),
						   sample_i, 
						   hSigmaA[wave_i], 
						   hSigmaE[wave_i] );


	float mseTmp = calcExpectation(dPhiAse[sampleOffset], dPhiAseSquare[sampleOffset], hRaysPerSampleDump);
	if(run == 0){
	  mseRunZero = mseTmp;
	  run++;
	}

	// MSE TESTs
	//dout(V_DEBUG) << "MSE: " << mseTmp << " with " << hRaysPerSampleDump << " rays,[" << dPhiAse[sampleOffset] << " || " << dPhiAseSquare[sampleOffset] << "]"<< std::endl;
	 //if(mseTmp > mse.at(sampleOffset)){
	 //  double a = dPhiAse[sampleOffset];
	 //  double b = dPhiAseSquare[sampleOffset];
	 //  dout(V_DEBUG) << "\nRaysPerSampleDump: "<< hRaysPerSampleDump << std::endl;
	 //  dout(V_DEBUG) << "RaysPerSample: " << hRaysPerSample << std::endl;
	 //  dout(V_DEBUG) << mseTmp << " > " << mse.at(sampleOffset) << "(" << sample_i << ")\n" << std::endl;
	 //}
        mse.at(sampleOffset) = mseTmp;

        if(mse.at(sampleOffset) < mseThreshold.at(wave_i))     break;
        if(hRaysPerSample * 10 > (unsigned long)maxRaysPerSample)break;

        // If the threshold is still too high, increase the number of rays and reset the previously calculated value
        hRaysPerSample             *= 10;
        dPhiAse[sampleOffset]       = 0;
        dPhiAseSquare[sampleOffset] = 0;

      }
      // Update progressbar
      //if((sample_i+1) % 10 == 0) fancyProgressBar(sample_i-minSample_i, maxSample_i / (gpu_i + 1), 60, progressStartTime);
      fancyProgressBar(maxSample_i / (gpu_i + 1));

      // get phiASE
      hPhiAse.at(sampleOffset) = dPhiAse[sampleOffset];
      hPhiAse.at(sampleOffset)   /= hRaysPerSampleDump * 4.0f * M_PI;
      totalRays.at(sampleOffset)  = hRaysPerSampleDump;
      mse.at(sampleOffset) = mseRunZero;

    }
    

  }

  // JUST OUTPUT
  // std::vector<unsigned> reflectionsPerPrism(hMesh.numberOfPrisms, 0);
  // std::vector<unsigned> raysPerPrism(hMesh.numberOfPrisms, 0);
  
  // for(unsigned i=0; i < midRaysPerSample; ++i){
  //   unsigned index = hIndicesOfPrisms[i];
  //   reflectionsPerPrism[index] = max(reflectionsPerPrism[index], (hNumberOfReflections[i] + 1) / 2);
  // }

  // for(unsigned i=0; i < hMesh.numberOfPrisms; ++i){
  //   for(unsigned j=0; j < reflectionSlices; ++j){
  //     unsigned index = i + hMesh.numberOfPrisms * j;
  //     raysPerPrism[i] += hRaysPerPrism[index];
  //   }
  // }

  //writePrismToVtk(hMesh, reflectionsPerPrism, "octrace_0_reflections", hRaysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, 0);
  //writePrismToVtk(hMesh, raysPerPrism, "octrace_0_rays", hRaysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, 0);

  dout(V_INFO | V_NOLABEL) << "\n" << std::endl;
  // Free Memory
  hipFree(devMTGPStates);
  hipFree(devKernelParams);

  runtime = difftime(time(0),starttime);
  return runtime;
}
