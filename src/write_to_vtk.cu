#include "hip/hip_runtime.h"
#include <iostream> /* cerr */
#include <fstream> /* ofstream */
#include <vector> /* vector */
#include <iomanip> /* std::setprecision() */
#include <mesh.h>
#include <cstdlib> /* atof */
#include <string>

int writeToVtk(Mesh *mesh,
	       std::vector<double>* ase,
	       std::string filename){

  std::cerr << "C Write experiment data to vtk-file" << std::endl;
  std::ofstream vtkFile;
  vtkFile.open(filename.c_str());

  // Write header of vtk file
  vtkFile << "# vtk DataFile Version 2.0" << std::endl;
  vtkFile << "octrace vtk file" << std::endl;
  vtkFile << "ASCII" << std::endl;

  // Write point data
  vtkFile << "DATASET UNSTRUCTURED_GRID" << std::endl;
  vtkFile << "POINTS " << mesh->numberOfSamples <<  " float" << std::endl;
  for(unsigned level_i=0; level_i < mesh->numberOfLevels; ++level_i){
    for(unsigned point_i=0; point_i < mesh->numberOfPoints; ++point_i){
      vtkFile << std::fixed << std::setprecision(6) << mesh->points[point_i] << " " << mesh->points[point_i + mesh->numberOfPoints] << " " << level_i * mesh->thickness << std::endl;

    }

  }

  // Write prism data
  vtkFile << "CELLS" << " " << mesh->numberOfPrisms << " " << mesh->numberOfPrisms * 7 << std::endl;
  for(unsigned level_i=0; level_i < (mesh->numberOfLevels - 1); ++level_i){
    for(unsigned triangle_i=0; triangle_i < mesh->numberOfTriangles; ++triangle_i){
      vtkFile << "6 " 
		  << level_i * mesh->numberOfPoints + mesh->triangles[triangle_i] << " "
	      << level_i * mesh->numberOfPoints + mesh->triangles[mesh->numberOfTriangles + triangle_i] << " "
	      << level_i * mesh->numberOfPoints + mesh->triangles[2 * mesh->numberOfTriangles + triangle_i] << " "
	      << (level_i+1) * mesh->numberOfPoints + mesh->triangles[triangle_i] << " "
	      << (level_i+1) * mesh->numberOfPoints + mesh->triangles[mesh->numberOfTriangles + triangle_i] << " "
	      << (level_i+1) * mesh->numberOfPoints + mesh->triangles[2 * mesh->numberOfTriangles + triangle_i] << std::endl;
	
    }

  }

  // Write cell type
  vtkFile << "CELL_TYPES " << mesh->numberOfPrisms << std::endl;
  for(unsigned prism_i=0; prism_i < mesh->numberOfPrisms; ++prism_i){
    vtkFile << "13" << std::endl;
  }

  // Write ase phi
  vtkFile << "POINT_DATA " << mesh->numberOfSamples << std::endl;
  vtkFile << "SCALARS scalars float 1" << std::endl;
  vtkFile << "LOOKUP_TABLE default" << std::endl;

  for(unsigned ase_i=0; ase_i < mesh->numberOfSamples; ++ase_i){
    vtkFile << std::fixed << std::setprecision(6) << ase->at(ase_i) << std::endl;
  }
  
  vtkFile.close();

  return 0;
}

int compareVtk(std::vector<double> *ase, std::string filename){
  std::ifstream filestream;
  std::string line;
  bool foundLine = false;
  double value = 0;
  double diff = 0;
  unsigned i = 0;
  double minDiff = 10000; // should be enough
  double maxDiff = 0;
  double totalDiff = 0;
  double aseTotal = 0;
  double smallDiff = 0.1;
  

  // No compare vtk was given
  if(!filename.compare("")){
    return 0;
  }

  // Sum up ase values
  for(unsigned i = 0; i < ase->size(); ++i){
    aseTotal += ase->at(i);
  }

  filestream.open(filename.c_str(), std::ifstream::in);

  if(filestream.is_open()){
    while(filestream.good()){
      std::getline(filestream, line);
      std::size_t found = line.find("LOOKUP_TABLE default");
      if(found != std::string::npos){ 
	foundLine = true;
	std::getline(filestream, line);
      }
      if(foundLine){
	if(i == ase->size())
	  break;
	value = (double) atof(line.c_str());
	totalDiff += abs(ase->at(i) - value);
	diff = abs(ase->at(i) / value - 1);
	ase->at(i) = diff;

	if(diff >= maxDiff)
	  maxDiff = diff;

	if(diff <= minDiff)
	  minDiff = diff;

	if(diff >= smallDiff){
	  std::cerr << "C ASE relative difference[" << i << "]: " << diff << " > " << smallDiff << std::endl;
	}
	i++;

      }

    }

  }
  else{
    std::cerr << "C Can't open file " << filename << " for comparison" << std::endl;
    return 1;
  }

  std::cerr << "C ASE max. relative difference: " << maxDiff << std::endl;
  std::cerr << "C ASE min. relative difference: " << minDiff << std::endl;
  std::cerr << "C ASE tot. relative difference: " << totalDiff / aseTotal << std::endl;
  filestream.close();
  return 0;
}
