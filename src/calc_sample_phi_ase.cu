#include "hip/hip_runtime.h"
#include <mesh.h>
#include <geometry.h> /* generateRay */
#include <propagate_ray.h> /* propagateRay */


__global__ void calcSamplePhiAse(hiprandStateMtgp32* globalState,
				 Mesh mesh, 
				 const unsigned* indicesOfPrisms, 
				 const int* indicesOfWavelengths, 
				 const double* importance,
				 const unsigned raysPerSample, 
				 float *phiAse, 
				 float *phiAseSquare,
				 const unsigned sample_i,
				 double *sigmaA, 
				 double *sigmaE
				 ) {

  int wave_i = indicesOfWavelengths[blockIdx.y];
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  int rayNumber = 0;
  unsigned stride = 0;
  double gainSum = 0;
  double gainSumSquare = 0;
  Point samplePoint = mesh.getSamplePoint(sample_i);

  if(wave_i == -1) return;

  // One thread can compute multiple rays
  // The current ray which we compute is based on the gid and an offset (number of threads*blocks)
  while ((rayNumber = gid + stride) < raysPerSample) {
          stride += blockDim.x * gridDim.x;
  	  // Get triangle prism to start from
  	  int startPrism = indicesOfPrisms[rayNumber + wave_i * raysPerSample];
  	  int startLevel = startPrism/mesh.numberOfTriangles;
  	  int startTriangle = startPrism - (mesh.numberOfTriangles * startLevel);

	  Point startPoint = mesh.genRndPoint(startTriangle, startLevel, &(globalState[wave_i * gridDim.x]));
	  Ray ray          = generateRay(startPoint, samplePoint);
	  double gain      = propagateRay(ray, startLevel, startTriangle, &mesh, sigmaA[wave_i], sigmaE[wave_i]);

	  gain *= mesh.getBetaValue(startPrism);
	  gain *= importance[startPrism + wave_i * mesh.numberOfPrisms];

	  gainSum += gain;
	  gainSumSquare += gain * gain;


  }
  atomicAdd(&(phiAse[sample_i  + wave_i * mesh.numberOfSamples]), float(gainSum));
  atomicAdd(&(phiAseSquare[sample_i  + wave_i * mesh.numberOfSamples]), float(gainSumSquare));


}
