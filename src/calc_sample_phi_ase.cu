#include "hip/hip_runtime.h"
#include <stdio.h> /* printf */
#include <mesh.h>
#include <geometry.h> /* generateRay */
#include <propagate_ray.h> /* propagateRay */
#include <assert.h> /* assert */



__device__ double calcIntersectionAngle(const Ray ray, float *reflectionAngle){
  // Calc intesection angle with z-plane
  float nominator = abs(ray.dir.z);
  float denominator = sqrt((ray.dir.x * ray.dir.x) + (ray.dir.y * ray.dir.y) + (ray.dir.z * ray.dir.z));
  if(denominator != 0.0){
    double radian = asin(nominator / denominator);
    *reflectionAngle = (180 / 3.1415926) * radian;
    return 0;
  }
  return 1;
}

__device__ int calcPlaneIntersectionPoint(const Ray reflectionRay, const int reflectionPlane, const Mesh *mesh, Point *intersectionPoint){
  // Assume that mesh is on x/y axis and parallel to x/y axis
  double planeZ = 0.0;
  if(reflectionPlane == 1){
    // Reflection on TOP plane
    planeZ = mesh->thickness * mesh->numberOfLevels;
  }
  double denominator = reflectionRay.dir.z;  
  if (denominator != 0.0){
    double nominator = planeZ - reflectionRay.p.z;
    double length = nominator / denominator;
    if(length > 0){
      intersectionPoint->x = reflectionRay.p.x + length * reflectionRay.dir.x;
      intersectionPoint->y = reflectionRay.p.y + length * reflectionRay.dir.y;
      intersectionPoint->z = reflectionRay.p.z + length * reflectionRay.dir.z;
      return 0;
    }
    else{
      printf("length < 0 ");
    }
  }
  return 1;
}


__device__ Ray generateReflectionRay(const Point startPoint, Point endPoint,  int reflectionsLeft, const int reflectionPlane, const Mesh *mesh){
  float mirrorPlaneZ = 0;
  if(reflectionsLeft % 2 == 0){
    // Even reflectionCount is postponement
    endPoint.z = endPoint.z + reflectionPlane * (reflectionsLeft * mesh->thickness * mesh->numberOfLevels); 
  }
  else {
    // Odd reflectionsCount is reflection

    if(reflectionPlane == 1){
      // TOP reflection
      mirrorPlaneZ = ceil(reflectionsLeft/(double)2) * mesh->thickness * mesh->numberOfLevels;
    }
    else{
      // BOTTOM reflection
      mirrorPlaneZ = floor(reflectionsLeft/(double)2) * mesh->thickness * mesh->numberOfLevels;
    }

    endPoint.z = reflectionPlane * abs(( mirrorPlaneZ + mirrorPlaneZ - endPoint.z));
    
  }
  return generateRay(startPoint, endPoint);
}

__device__ int calcNextReflection(Point startPoint, Point endPoint, unsigned reflectionsLeft, int reflectionPlane, Point *reflectionPoint, float *reflectionAngle, Mesh *mesh){
  Ray reflectionRay = generateReflectionRay(startPoint, endPoint, reflectionsLeft, reflectionPlane, mesh);
  if(calcPlaneIntersectionPoint(reflectionRay, reflectionPlane, mesh, reflectionPoint)) return 1;
  if(calcIntersectionAngle(reflectionRay, reflectionAngle)) return 1;
  return 0;
}

__global__ void calcSamplePhiAse(hiprandStateMtgp32* globalState,
				 Mesh mesh, 
				 const unsigned* indicesOfPrisms, 
				 const int* indicesOfWavelengths, 
				 const double* importance,
				 const unsigned raysPerSample, 
				 float *phiAse, 
				 float *phiAseSquare,
				 const unsigned sample_i,
				 double *sigmaA, 
				 double *sigmaE
				 ) {

  int wave_i = indicesOfWavelengths[blockIdx.y];
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  int rayNumber = 0;
  unsigned stride = 0;
  double gainSum = 0;
  double gainSumSquare = 0;
  Point samplePoint = mesh.getSamplePoint(sample_i);

  // Const data for reflection test
  // Should be random generated (0 to X)
  const unsigned reflections = 0; 
  const float reflectivity = 0.5;
  const float totalReflectionAngle = 45;
  // Should be random generated (-1 or 1)
  int reflectionPlane = 1; // -1 = BOTTOM, 1 = TOP

  if(wave_i == -1) return;

  // One thread can compute multiple rays
  // The current ray which we compute is based on the gid and an offset (number of threads*blocks)
  while ((rayNumber = gid + stride) < raysPerSample) {
    stride += blockDim.x * gridDim.x;
    // Get triangle/prism to start ray from
    unsigned startPrism = indicesOfPrisms[rayNumber + wave_i * raysPerSample];
    unsigned startLevel = startPrism/mesh.numberOfTriangles;
    unsigned startTriangle = startPrism - (mesh.numberOfTriangles * startLevel);
    Point startPoint = mesh.genRndPoint(startTriangle, startLevel, &(globalState[wave_i * gridDim.x]));

    // Calculate reflections as different ray propagations
    double gain = 1.0;
    for(unsigned reflection_i = 0; reflection_i < reflections; ++reflection_i){
      Point reflectionPoint = {0,0,0};
      float reflectionAngle = 0;
      Ray   reflectionRay   = {{0,0,0},{0,0,0}};

      // Calc reflectionPoint and reflectionAngle
      calcNextReflection(startPoint, samplePoint, (reflections-reflection_i), reflectionPlane, &reflectionPoint, &reflectionAngle, &mesh);

      // Debug output
      // if(gid == 0) printf("[%d][%d] angle: %f\n", sample_i, reflection_i, reflectionAngle);

      // Propagate this part of the ray
      reflectionRay       = generateRay(startPoint, reflectionPoint);
      gain *= propagateRay(reflectionRay, &startLevel, &startTriangle, &mesh, sigmaA[wave_i], sigmaE[wave_i]);
      if(reflectionAngle < totalReflectionAngle) 
	gain *= reflectivity;
      startPoint          = reflectionPoint;
      reflectionPlane     = (reflectionPlane * -1);

    }
    // Calculate last part of ray without reflection
    Ray ray = generateRay(startPoint, samplePoint);
    gain   *= propagateRay(ray, &startLevel, &startTriangle, &mesh, sigmaA[wave_i], sigmaE[wave_i]);

    gain *= mesh.getBetaValue(startPrism) * importance[startPrism + wave_i * mesh.numberOfPrisms];
    
    gainSum += gain;
    gainSumSquare += gain * gain;

  }
  atomicAdd(&(phiAse[sample_i  + wave_i * mesh.numberOfSamples]), float(gainSum));
  atomicAdd(&(phiAseSquare[sample_i  + wave_i * mesh.numberOfSamples]), float(gainSumSquare));

}
