#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h> /* hiprand_uniform */
#include <mesh.h>
#include <geometry.h> /* generateRay */
#include <propagate_ray.h> /* propagateRay */


// ##############################################################
// # Reconstruction                                             #
// ##############################################################
__device__ Point calcRndStartPoint(Triangle triangle, unsigned level, double thickness, hiprandStateMtgp32* globalState){
  Point startPoint = {0,0,0};
  double u = hiprand_uniform(&globalState[blockIdx.x]);
  double v = hiprand_uniform(&globalState[blockIdx.x]);

  if((u+v)>1)
    {
      u = 1-u;
      v = 1-v;
    }
  double w = 1-u-v;

  // convert the random startpoint into coordinates
  startPoint.x = (triangle.A.x * u) + (triangle.B.x * v) + (triangle.C.x * w);
  startPoint.y = (triangle.A.y * u) + (triangle.B.y * v) + (triangle.C.y * w);
  startPoint.z = (level + hiprand_uniform(&globalState[blockIdx.x])) * thickness;

  return startPoint;
}


__global__ void calcSamplePhiAse(
		hiprandStateMtgp32* globalState,
		Point samplePoint,
		Mesh mesh, 
		unsigned* indicesOfPrisms, 
		double* importance,
		unsigned raysPerSample, 
		float *phiAse, 
		const unsigned sample_i,
		const double sigmaA, 
		const double sigmaE, 
		const double nTot) {

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  Triangle *triangles = mesh.triangles;
  unsigned numberOfTriangles = mesh.numberOfTriangles;

  __shared__ double threadGain[256]; //MUST be the same as number of threads
  threadGain[threadIdx.x] = 0.;

  // One thread can compute multiple rays
  int rayNumber;
  unsigned i=0;
  // the current ray which we compute is based on the id and an offset (number of threads*blocks)
  while ((rayNumber = id + (blockDim.x*gridDim.x * i++)) < raysPerSample) {

  	  // Get triangle prism to start from
  	  int startPrism = indicesOfPrisms[rayNumber];
  	  int startLevel = startPrism/numberOfTriangles;
  	  int startTriangle_i = startPrism - (numberOfTriangles * startLevel);
  	  Triangle startTriangle = triangles[startTriangle_i];

  	  // Random startpoint generation
	  Point startPoint = calcRndStartPoint(startTriangle, startLevel, mesh.thickness, globalState);

	  // Ray generation
	  Ray ray = generateRay(startPoint, samplePoint);

  	  // // propagate the ray
	  double gain = propagateRayNew(ray, startLevel, startTriangle, sigmaA, sigmaE, nTot, mesh.thickness );

	  gain *= startTriangle.betaValues[startLevel];
	  gain *= importance[startPrism];

	  threadGain[threadIdx.x] += gain;
  }

  // reduce the shared memory to one element (CUDA by Example, Chapter 5.3)
  __syncthreads();
  
  i = blockDim.x/2;
  while(i != 0){
	  if(threadIdx.x < i){
		  threadGain[threadIdx.x] += threadGain[threadIdx.x + i];
	  }
	  __syncthreads();
	  i /= 2;
  }
  // thread 0 writes it to the global memory
  if(threadIdx.x == 0){
	  atomicAdd(&(phiAse[sample_i]), float(threadGain[threadIdx.x]));
  }
}
