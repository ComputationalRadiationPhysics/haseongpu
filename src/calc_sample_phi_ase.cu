#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h> /* hiprand_uniform */
#include <stdio.h> /* printf */
#include "propagate_ray.h"

/**
 * Does the raytracing for a single Sample point (in a defined level).
 * This Kernel has to be started for each sample point with the same value for iterations
 * and the same number of blocks/threads.
 *
 * \var globalState the state of the mersenneTwister PRNG
 * 		(has a maximum of 200 positions!)
 * \var phi points to a memory region which is initialized with 0
 * 		(can hold one value for each sample point)
 * \var point2D the index of the current sample point (points to p_in)
 * \var level the level of the current sample point (how deep we are through the material)
 * 		(always for the same combination of startprism+samplepoint
 */
__global__ void calcSamplePhiAse(
		hiprandStateMtgp32* globalState,
		float* phiASE,
		int point2D,
		int level,
		double *points,
		double *xOfNormals,
		double *yOfNormals,
		int *positionsOfNormalVectors,
		int *neighbors,
		int *forbidden,
		int* triangleIndices,
		double* betaValues,
		double* importance,
		unsigned* indicesOfPrisms,
		unsigned raysPerSample,
		double nTot,
		double sigmaE,
		double sigmaA,
		double thicknessOfPrism,
		int numberOfLevels,
		int numberOfPoints,
		int numberOfTriangles
		){

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  double endPointX = points[point2D];
  double endPointY = points[numberOfPoints + point2D];
  double endPointZ = level * thicknessOfPrism;
  __shared__ double threadGain[256]; //MUST be the same as number of threads
  threadGain[threadIdx.x] = 0.;

  // on thread can compute multiple rays
  for (int i=0; ; ++i){

	  // the current ray which we compute is based on the id and an offset (number of threads*blocks)
	  int rayNumber = id + (blockDim.x*gridDim.x * i);
	  if(rayNumber >= raysPerSample){
		  break;
	  }

	  // get a new prism to start from
	  int startPrism = indicesOfPrisms[rayNumber];
	  int startLevel = startPrism/numberOfTriangles;
	  int startTriangle = startPrism - (numberOfTriangles * startLevel);

	  // Get triangle vertex indicies
	  int t1 = triangleIndices[startTriangle];
	  int t2 = triangleIndices[startTriangle + numberOfTriangles];
	  int t3 = triangleIndices[startTriangle + 2 * numberOfTriangles];

	  // random startpoint generation
	  double u = hiprand_uniform(&globalState[blockIdx.x]);
	  double v = hiprand_uniform(&globalState[blockIdx.x]);

	  if((u+v)>1)
	  {
		  u = 1-u;
		  v = 1-v;
	  }
	  double w = 1-u-v;

	  // convert the random startpoint into coordinates
	  double xRand = (points[t1] * u) + (points[t2] * v) + (points[t3] * w);
	  double yRand = (points[numberOfPoints + t1] * u) + (points[numberOfPoints + t2] * v) + (points[numberOfPoints + t3] * w);
	  double zRand = (startLevel + hiprand_uniform(&globalState[blockIdx.x])) * thicknessOfPrism;

	  // propagate the ray
	  double gain = propagateRay(xRand, yRand, zRand, endPointX, endPointY, endPointZ, 
				   startTriangle, startLevel, points, xOfNormals, yOfNormals, 
				   positionsOfNormalVectors, neighbors, forbidden,  betaValues,
				   nTot, sigmaE, sigmaA, thicknessOfPrism, numberOfLevels, numberOfPoints, numberOfTriangles);

	  threadGain[threadIdx.x] += gain * betaValues[startPrism] * importance[startPrism];
  }

  // reduce the shared memory to one element (CUDA by Example, Chapter 5.3)
  __syncthreads();
  unsigned i = blockDim.x/2;
  while(i != 0){
	  if(threadIdx.x < i){
		  threadGain[threadIdx.x] += threadGain[threadIdx.x + i];
	  }
	  __syncthreads();
	  i /= 2;
  }

  // thread 0 writes it to the global memory
  if(threadIdx.x == 0){
	  atomicAdd(&(phiASE[point2D + (level * numberOfPoints)]), float(threadGain[threadIdx.x]));
  }
}
