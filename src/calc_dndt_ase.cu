#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <calc_sample_phi_ase.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>
#include <mesh.h>
#include <ctime> /* progressBar */
#include <progressbar.h> /*progressBar */

#define SEED 4321

float calcDndtAse (unsigned &threads, 
		   unsigned &blocks,
		   unsigned &hostRaysPerSample,
		   Mesh mesh,
		   Mesh hostMesh,
		   std::vector<double> *betaCellsVector,
		   float nTot,
		   std::vector<double> *hostSigmaA,
		   std::vector<double> *hostSigmaE,
		   float crystalFluorescence,
		   std::vector<double> *dndtAse
		   ){

  // Variable declaration
  // CPU
  double *hostImportance;
  unsigned *hostRaysPerPrism;
  float runtime;
  unsigned *hostIndicesOfPrisms;
  float *hostPhiAse;
  time_t starttime,progressStartTime;

  // GPU
  float *phiAse;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  double *importance;
  unsigned *indicesOfPrisms;
  float *sumPhi;
  unsigned *raysDump;
  unsigned *raysPerPrism;
  unsigned *cumulativeSums;
  double * sigmaA;
  double * sigmaE;

  //OPTIMIZE: find perfect number of threads - MUST be the same as the size of shared memory in kernel
  dim3 blockDim(256);
  dim3 gridDim(200, hostSigmaE->size());
  threads = blockDim.x;
  blocks = gridDim.x * gridDim.y;
    
  starttime = time(0);

  hostPhiAse          = (float*)    malloc (hostMesh.numberOfSamples * gridDim.y * sizeof(float));
  hostImportance      = (double*)   malloc (hostMesh.numberOfPrisms  * gridDim.y * sizeof(double));
  hostRaysPerPrism    = (unsigned*) malloc (hostMesh.numberOfPrisms  * gridDim.y * sizeof(unsigned));
  hostIndicesOfPrisms = (unsigned*) malloc (hostRaysPerSample        * gridDim.y * sizeof(unsigned));

  for(unsigned i=0; i < hostRaysPerSample * gridDim.y; ++i) hostIndicesOfPrisms[i] = 0;
  for(unsigned i=0; i < hostMesh.numberOfSamples * gridDim.y; ++i) hostPhiAse[i] = 0.f;
  for(unsigned i=0; i < hostMesh.numberOfPrisms * gridDim.y; ++i) hostRaysPerPrism[i] = 1;
  for(unsigned i=0; i < hostMesh.numberOfPrisms * gridDim.y; ++i) hostImportance[i] = 1.0;

  // CUDA Mersenne twister
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.y * gridDim.x  * sizeof(hiprandStateMtgp32)));

  // TODO maybe change seed for different mersenne twister
  CUDA_CALL(hipMalloc((void**)&devKernelParams, gridDim.y * sizeof(mtgp32_kernel_params)));
  for(unsigned mersenne_i = 0; mersenne_i < gridDim.y; ++mersenne_i){
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, &(devKernelParams[mersenne_i])));
    CURAND_CALL(hiprandMakeMTGP32KernelState(&(devMTGPStates[gridDim.x * mersenne_i]), mtgp32dc_params_fast_11213, &(devKernelParams[mersenne_i]), gridDim.x, SEED + mersenne_i));
  }

  // Memory allocation on device
  CUDA_CHECK_RETURN(hipMalloc(&phiAse, hostMesh.numberOfSamples * gridDim.y * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&importance, hostMesh.numberOfPrisms * gridDim.y * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&indicesOfPrisms, hostRaysPerSample * gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&raysPerPrism, hostMesh.numberOfPrisms * gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&sumPhi, gridDim.y * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&raysDump, gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&cumulativeSums,  hostMesh.numberOfPrisms * gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&sigmaA, gridDim.y * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&sigmaE, gridDim.y * sizeof(double)));

  // Copy host to device
  CUDA_CHECK_RETURN(hipMemcpy(phiAse, hostPhiAse, hostMesh.numberOfSamples * gridDim.y * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(sigmaA, &(hostSigmaA->at(0)), hostSigmaA->size() * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(sigmaE, &(hostSigmaE->at(0)), gridDim.y * sizeof(double), hipMemcpyHostToDevice));
  
  // Calculate Phi Ase foreach sample
  fprintf(stderr, "\nC Start Phi Ase calculation\n");
  progressStartTime = time(0);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){

    hostRaysPerSample = importanceSampling(sample_i, mesh, hostRaysPerSample, sigmaA, sigmaE, nTot, importance, sumPhi, raysPerPrism, indicesOfPrisms, raysDump, cumulativeSums, blockDim, gridDim);

    CUDA_CHECK_RETURN(hipMemcpy(hostRaysPerPrism, raysPerPrism, hostMesh.numberOfPrisms * gridDim.y * sizeof(unsigned),hipMemcpyDeviceToHost));

    // Prism scheduling for gpu threads
    for(unsigned wave_i=0; wave_i < gridDim.y; ++wave_i){
      for(unsigned prism_i=0, absoluteRay = 0; prism_i < hostMesh.numberOfPrisms; ++prism_i){
    	for(unsigned ray_i=0; ray_i < hostRaysPerPrism[prism_i + hostMesh.numberOfPrisms * wave_i]; ++ray_i){
    	  hostIndicesOfPrisms[absoluteRay + hostRaysPerSample * wave_i] = prism_i;
    	  absoluteRay++;
    	  assert(absoluteRay <= hostRaysPerSample);
    	}
      }
    }

    // Copy dynamic sample data to device
    CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, hostRaysPerSample * gridDim.y * sizeof(unsigned), hipMemcpyHostToDevice));

    // Start Kernel
    calcSamplePhiAse<<< gridDim, blockDim >>>(devMTGPStates, mesh, indicesOfPrisms, importance, hostRaysPerSample, phiAse, sample_i, sigmaA, sigmaE, nTot);

    // update progressbar
    if((sample_i+1) % 10 == 0) fancyProgressBar(sample_i,hostMesh.numberOfSamples,60,progressStartTime);

  }
  // Copy solution back to host
  CUDA_CHECK_RETURN(hipMemcpy(hostPhiAse, phiAse, hostMesh.numberOfSamples * gridDim.y * sizeof(float), hipMemcpyDeviceToHost));

  // Calculate dndt Ase
  for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
    for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){
      hostPhiAse[sample_i + hostMesh.numberOfSamples * wave_i] = float((double(hostPhiAse[sample_i + hostMesh.numberOfSamples * wave_i]) / (hostRaysPerSample * 4.0f * 3.14159)));
      double gain_local = double(nTot) * (betaCellsVector->at(sample_i)) * double(hostSigmaE->at(wave_i) + hostSigmaA->at(wave_i)) - double(nTot * hostSigmaA->at(wave_i));
      dndtAse->at(sample_i + hostMesh.numberOfSamples * wave_i) = gain_local * hostPhiAse[sample_i + hostMesh.numberOfSamples * wave_i] / crystalFluorescence;

    }
  }
  // Stop time
  runtime = difftime(time(0),starttime);


  // Free Memory
  free(hostPhiAse);
  free(hostImportance);
  free(hostRaysPerPrism);
  free(hostIndicesOfPrisms);
  hipFree(phiAse);
  hipFree(importance);
  hipFree(indicesOfPrisms);
  hipFree(raysPerPrism);
  hipFree(sumPhi);
  hipFree(raysDump);
  hipFree(sigmaA);
  hipFree(sigmaE);
  hipDeviceReset();

  return runtime;

}
