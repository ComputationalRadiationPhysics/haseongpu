#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <calc_sample_phi_ase.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>
#include <mesh.h>
#include <ctime> /* progressBar */
#include <progressbar.h> /*progressBar */

#define SEED 1234

/**
 * @brief Calculates which ray should start in which prism. Thus
 *        every thread in on gpu knows the index of the prism
 *        where its rays starts.
 *
 **/
void calcIndicesOfPrism(unsigned *indicesOfPrisms, unsigned* raysPerPrism, unsigned raysPerSample, Mesh mesh, dim3 gridDim){
  for(unsigned wave_i=0; wave_i < gridDim.y; ++wave_i){
    for(unsigned prism_i=0, absoluteRay = 0; prism_i < mesh.numberOfPrisms; ++prism_i){
      for(unsigned ray_i=0; ray_i < raysPerPrism[prism_i + mesh.numberOfPrisms * wave_i]; ++ray_i){
	indicesOfPrisms[absoluteRay + raysPerSample * wave_i] = prism_i;
	absoluteRay++;
	assert(absoluteRay <= raysPerSample);
      }
      
    }
    
  }
  
}

/**
 * @brief Gives every 200 blocks an index to the sigma_a/_e array or -1
 *        if this wavelength will be ignored.
 **/
void calcIndicesOfWavelengths(int *indicesOfWavelength, dim3 gridDim, std::vector<bool> ignoreWavelength){
  for(unsigned wave_i=0; wave_i < gridDim.y; ++wave_i){
    if(ignoreWavelength[wave_i]){
      indicesOfWavelength[wave_i] = -1;
    }
    else{
      indicesOfWavelength[wave_i] = wave_i;

    }

  }
 
}

double calcExpectation(double phiAse, double phiAseSquare, unsigned raysPerSample){
  double a = phiAseSquare / raysPerSample;
  double b = (phiAse / raysPerSample) * (phiAse / raysPerSample);

  return sqrt(abs((a - b) / raysPerSample));
}

float calcDndtAse (unsigned &threads, 
		   unsigned &blocks,
		   unsigned &hostRaysPerSample,
		   Mesh mesh,
		   Mesh hostMesh,
		   std::vector<double> *hostSigmaA,
		   std::vector<double> *hostSigmaE,
		   std::vector<double> *dndtAse,
		   std::vector<float> *hostPhiAse,
		   std::vector<double> *expectation
		   ){

  // Variable declaration
  // CPU
  double *hostImportance;
  unsigned *hostRaysPerPrism;
  float runtime;
  unsigned *hostIndicesOfPrisms;
  int *hostIndicesOfWavelengths;
  float *hostPhiAseSquare;
  time_t starttime,progressStartTime;
  unsigned hostRaysPerSampleSave;
  float expectationThreshold;
  unsigned maxRaysPerSample;


  // GPU
  float *phiAse;
  float *phiAseSquare;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  double *importance;
  unsigned *indicesOfPrisms;
  int *indicesOfWavelengths;
  unsigned *raysPerPrism;
  unsigned *cumulativeSums;
  double * sigmaA;
  double * sigmaE;

  dim3 blockDim(256);
  dim3 gridDim(200, hostSigmaE->size());
  threads = blockDim.x;
  blocks = gridDim.x * gridDim.y;
    
  starttime = time(0);
  hostRaysPerSampleSave = hostRaysPerSample;
  expectationThreshold = 0.001;
  maxRaysPerSample = 10000000; // 10M


  // Memory allocation on host
  hostPhiAseSquare         = (float*)    malloc (hostMesh.numberOfSamples * gridDim.y * sizeof(float));
  hostImportance           = (double*)   malloc (hostMesh.numberOfPrisms  * gridDim.y * sizeof(double));
  hostRaysPerPrism         = (unsigned*) malloc (hostMesh.numberOfPrisms  * gridDim.y * sizeof(unsigned));
  hostIndicesOfPrisms      = (unsigned*) malloc (maxRaysPerSample         * gridDim.y * sizeof(unsigned));
  hostIndicesOfWavelengths = (int*) malloc (gridDim.y * sizeof(int));

  for(unsigned i=0; i < hostRaysPerSample * gridDim.y; ++i) hostIndicesOfPrisms[i] = 0;
  for(unsigned i=0; i < gridDim.y; ++i) hostIndicesOfWavelengths[i] = 0;
  for(unsigned i=0; i < hostMesh.numberOfSamples * gridDim.y; ++i) hostPhiAseSquare[i] = 0.f;
  for(unsigned i=0; i < hostMesh.numberOfPrisms * gridDim.y; ++i) hostRaysPerPrism[i] = 1;
  for(unsigned i=0; i < hostMesh.numberOfPrisms * gridDim.y; ++i) hostImportance[i] = 1.0;

  // CUDA Mersenne twister for more than 200 blocks (for every wavelength)
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.y * gridDim.x  * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, gridDim.y * sizeof(mtgp32_kernel_params)));
  for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, &(devKernelParams[wave_i])));
    CURAND_CALL(hiprandMakeMTGP32KernelState(&(devMTGPStates[gridDim.x * wave_i]), mtgp32dc_params_fast_11213, &(devKernelParams[wave_i]), gridDim.x, SEED + wave_i));
  }

  // Memory allocation on device
  CUDA_CHECK_RETURN(hipMalloc(&indicesOfPrisms, maxRaysPerSample * gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&indicesOfWavelengths, gridDim.y * sizeof(int)));
  CUDA_CHECK_RETURN(hipMalloc(&phiAse, hostMesh.numberOfSamples * gridDim.y * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&phiAseSquare, hostMesh.numberOfSamples * gridDim.y * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&importance, hostMesh.numberOfPrisms * gridDim.y * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&raysPerPrism, hostMesh.numberOfPrisms * gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&cumulativeSums,  hostMesh.numberOfPrisms * gridDim.y * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&sigmaA, gridDim.y * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&sigmaE, gridDim.y * sizeof(double)));

  // Copy host to device
  CUDA_CHECK_RETURN(hipMemcpy(phiAse, &(hostPhiAse->at(0)), hostMesh.numberOfSamples * gridDim.y * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(phiAseSquare, hostPhiAseSquare, hostMesh.numberOfSamples * gridDim.y * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(sigmaA, &(hostSigmaA->at(0)), hostSigmaA->size() * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(sigmaE, &(hostSigmaE->at(0)), gridDim.y * sizeof(double), hipMemcpyHostToDevice));
  
  // Calculate Phi Ase foreach sample
  fprintf(stderr, "\nC Start Phi Ase calculation\n");
  progressStartTime = time(0);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){
    bool expectationIsMet = false;
    std::vector<bool> ignoreWavelength(gridDim.y, false);
    std::vector<unsigned> raysPerSamplePerWave(gridDim.y, hostRaysPerSampleSave);
    hostRaysPerSample = hostRaysPerSampleSave;

    while(!expectationIsMet){
      expectationIsMet = true;

      hostRaysPerSample = importanceSampling(sample_i, mesh, hostRaysPerSample, sigmaA, sigmaE, importance, raysPerPrism, blockDim, gridDim);
      CUDA_CHECK_RETURN(hipMemcpy(hostRaysPerPrism, raysPerPrism, hostMesh.numberOfPrisms * gridDim.y * sizeof(unsigned),hipMemcpyDeviceToHost));

      // Prism scheduling for gpu threads
      calcIndicesOfPrism(hostIndicesOfPrisms, hostRaysPerPrism, hostRaysPerSample, hostMesh, gridDim);
      CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, hostRaysPerSample * gridDim.y * sizeof(unsigned), hipMemcpyHostToDevice));

      // Filter wavelengths which reached expectations
      calcIndicesOfWavelengths(hostIndicesOfWavelengths, gridDim, ignoreWavelength);
      CUDA_CHECK_RETURN(hipMemcpy(indicesOfWavelengths, hostIndicesOfWavelengths, gridDim.y * sizeof(int), hipMemcpyHostToDevice));

      // Start Kernel
      calcSamplePhiAse<<< gridDim, blockDim >>>(devMTGPStates, mesh, indicesOfPrisms, indicesOfWavelengths, importance, hostRaysPerSample, phiAse, phiAseSquare, sample_i, sigmaA, sigmaE);

      // Calculate expectations
      for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
        int sampleOffset = sample_i + hostMesh.numberOfSamples * wave_i;

        // Copy solution (for this samplepoint) back to host
        CUDA_CHECK_RETURN(hipMemcpy(&(hostPhiAse->at(sampleOffset)), &(phiAse[sampleOffset]), sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipMemcpy(&(hostPhiAseSquare[sampleOffset]), &(phiAseSquare[sampleOffset]), sizeof(float), hipMemcpyDeviceToHost));

        // Check square error
        expectation->at(sampleOffset) =  calcExpectation(hostPhiAse->at(sampleOffset), hostPhiAseSquare[sampleOffset], hostRaysPerSample);
        if(expectation->at(sampleOffset) >= expectationThreshold){
	  expectationIsMet = false;
	  ignoreWavelength[wave_i] = false;
        }
	else{
	  ignoreWavelength[wave_i] = true;
	}

      }

      // Stop calculations on maxRaysPerSample (keep solution)
      if(hostRaysPerSample == maxRaysPerSample) break;

      // If the threshold is still too high, increase the number of rays and reset the previously calculated value
      if(!expectationIsMet){
        hostRaysPerSample *= 10;
        for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
	  if(!ignoreWavelength[wave_i]){
	    int sampleOffset = sample_i + hostMesh.numberOfSamples * wave_i;
	    raysPerSamplePerWave[wave_i] *= 10;
	    hostPhiAse->at(sampleOffset) = 0;
	    hostPhiAseSquare[sampleOffset] = 0;
	    CUDA_CHECK_RETURN( hipMemcpy(&(phiAse[sampleOffset]), &(hostPhiAse->at(sampleOffset)), sizeof(float), hipMemcpyHostToDevice));
	    CUDA_CHECK_RETURN( hipMemcpy(&(phiAseSquare[sampleOffset]), &(hostPhiAseSquare[sampleOffset]), sizeof(float), hipMemcpyHostToDevice));
	  }
	  
        }

      }

    }

    // Update progressbar
    if((sample_i+1) % 10 == 0) fancyProgressBar(sample_i,hostMesh.numberOfSamples,60,progressStartTime);

    // Calculate dndt Ase, after one point is completely sampled
    for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
	int sampleOffset = sample_i + hostMesh.numberOfSamples * wave_i;
	hostPhiAse->at(sampleOffset) = float((double(hostPhiAse->at(sampleOffset)) / (raysPerSamplePerWave[wave_i] * 4.0f * 3.14159)));
	double gain_local = double(hostMesh.nTot) * hostMesh.betaCells[sample_i] * double(hostSigmaE->at(wave_i) + hostSigmaA->at(wave_i)) - double(hostMesh.nTot * hostSigmaA->at(wave_i));
	dndtAse->at(sampleOffset) = gain_local * hostPhiAse->at(sampleOffset) / hostMesh.crystalFluorescence;
    
    }

  }


  // Stop time
  runtime = difftime(time(0),starttime);

  // Free Memory
  // HINT Don't free importance if we return value to main
  free(hostImportance);
  free(hostRaysPerPrism);
  free(hostIndicesOfPrisms);
  hipFree(phiAse);
  hipFree(importance);
  hipFree(indicesOfPrisms);
  hipFree(raysPerPrism);
  hipFree(sigmaA);
  hipFree(sigmaE);
  hipDeviceReset();

  return runtime;
}
