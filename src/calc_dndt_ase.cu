#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <calc_sample_phi_ase.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>

#define SEED 1234
#define MIN_COMPUTE_CAPABILITY 2


/** Queries the devices to find the one with the highest Compute Capability
 * and sets it as our current device. 
 * Will result in a visible error and terminate program execution, 
 * if no suitable device is detected
 */
int getCorrectDevice(int verbose){
  int count = 0, candidate = -1;
  unsigned minCapability = MIN_COMPUTE_CAPABILITY;
  hipDeviceProp_t prop;

  CUDA_CHECK_RETURN( hipGetDeviceCount(&count) );
  
  for(int i=0; i<count; ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, i) );
    if(prop.major >= minCapability){
      minCapability = prop.major;
      candidate = i;
    }
  }

  if(candidate == -1){
    fprintf(stderr,"\nNone of the CUDA-capable devices is sufficient!\n");
    exit(1);
  }else{
    if(verbose > 0){
      CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, candidate) );
      fprintf(stderr,"\nC using CUDA device: %s (Compute Capability %d.%d)\n", prop.name, prop.major, prop.minor); 
    }
    CUDA_CHECK_RETURN( hipSetDevice(candidate) );
  }
  return candidate;
}

void printIntermediateValues(unsigned size, double *importance, float *phi, std::vector<double> *dndtAse, int iteration){
  fprintf(stderr,"###### ITERATION: %d ##########\n",iteration);
//  for(int i=0; i<size; ++i){
//    fprintf(stderr, "Importance[%d]: %f\n",i,importance[i]);
//  }
  for(int i=0; i<size; ++i){
    fprintf(stderr, "hostPhi[%d]: %f\n",i,phi[i]);
  }
//  for(int i=0; i<size; ++i){
//    fprintf(stderr, "dndtAse[%d]: %f\n",i,dndtAse->at(i));
//  }

}

/** GPU Kernel Variables
 * The idea is, that the number of threads is fixed (to maximize GPU occupancy)
 * and the number of blocks as well (200 is the maximum for the standard
 * Mersenne Twister implementaion). Therefore, the number of rays per sample
 * are fixed to be k*200*256.
 * That means, sometimes we have to increase the number of rays a little.
 *
 * \var raysPerThread is used to give every thread k iterations (to simulate k rays)
 *
 * note that every samplepoint receives the exact same number of rays.
 *
 * \var p_in: coordinates of the sample-points of one layer (first all x-coordinates, then all y-coordinates)
 * \var n_*: values of the normal-vectors for the 3 rectangular sides of each prism (described in 2D)
 * \var beta_v: the beta values of the prisms
 * \var phi: the accumulated ASE-Flux for each sample point
 * \var forbidden: the side of the prism through which the ray "entered" the prism
 * \var n_p: the points where the normals (n_x,n_y) start
 * \var neighbors: indices to the adjacent triangles in t_in
 * \var t_in: indices of the points which are considered to be a triangle (A points start from 0, B points from size_t, C points from size_t*2)
 * \var cell_type: determines which cell type we are looking at.
 * other input parameters are put to the GPU by the setupGlobalVariablesKernel
 */
float calcDndtAse(
      std::vector<double> *dndtAse, 
      unsigned &threads, 
      unsigned &blocks, 
      unsigned &hostRaysPerSample,
      std::vector<double> *betaValuesVector,
      std::vector<double> *xOfNormalsVector,
      std::vector<double> *yOfNormalsVector,
      std::vector<unsigned> *triangleIndicesVector,
      std::vector<int> *forbiddenVector,
      std::vector<int> *neighborsVector,
      std::vector<int> *positionsOfNormalVectorsVector,
      std::vector<double> *pointsVector,
      std::vector<double> *betaCellsVector,
      std::vector<float> *surfacesVector,
      std::vector<double> *xOfTriangleCenterVector,
      std::vector<double> *yOfTriangleCenterVector,
      float hostNTot,
      float hostSigmaA,
      float hostSigmaE,
      unsigned hostNumberOfPoints,
      unsigned hostNumberOfTriangles,
      unsigned hostNumberOfLevels,
      float hostThicknessOfPrism,
      float hostCrystalFluorescence)
{
  // Variable declarations
  // CPU
  double* hostImportance;
  unsigned* hostNumberOfImportantRays;
  int* hostIndicesOfPrisms;
  unsigned hostNumberOfPrisms;
  unsigned hostRaysPerThread;
  unsigned hostNumberOfSamples;
  hipEvent_t start, stop;
  float runtimeGpu;
  float *hostPhiASE;
  unsigned kernelcount;
  // GPU
  double  *points, *xOfNormals, *yOfNormals, *betaValues;
  float *phiASE;
  int *forbidden, *positionsOfNormalVectors, *neighbors, *triangleIndices;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  double *importance;
  unsigned *indicesOfPrisms;
  
  // Variables defintions
  threads = 256; //OPTIMIZE: find perfect number of threads - MUST be the same as the size of shared memory in kernel
  blocks = 200;
  hostNumberOfPrisms = (hostNumberOfTriangles * (hostNumberOfLevels-1));
  hostNumberOfSamples = hostNumberOfPoints * hostNumberOfLevels;
  
  hostPhiASE = (float*) malloc(hostNumberOfSamples * sizeof(float));
  hostImportance = (double*) malloc(hostNumberOfPrisms * sizeof(double));
  hostNumberOfImportantRays = (unsigned*) malloc(hostNumberOfPrisms * sizeof(unsigned));
  hostIndicesOfPrisms = (int*) malloc(hostRaysPerSample * sizeof(int));

  runtimeGpu = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  kernelcount = 0;

  for(int i=0; i < hostRaysPerSample; ++i) hostIndicesOfPrisms[i] = 0;
  for(int i=0; i < hostNumberOfSamples; ++i) hostPhiASE[i] = 0.f;
  for(int i=0; i < hostNumberOfPrisms; ++i) hostNumberOfImportantRays[i] = 1;
  for(int i=0; i < hostNumberOfPrisms; ++i)hostImportance[i] = 1.0;

  // check, if we run on the correct machine / select a good device
  getCorrectDevice(1);

  // Init mersenne twister PRNG
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, blocks * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, blocks, SEED));

  // Set global variables on device
  //@OPTIMIZE: initialize the constants as constants...
  //@OPTIMIZE: call by value, not global variable
  setupGlobalVariablesKernel<<<1,1>>>(double(hostSigmaE), 
              double(hostSigmaA),
              double(hostNTot), 
              hostNumberOfTriangles, 
              double(hostThicknessOfPrism),
              hostNumberOfLevels, 
              hostNumberOfPoints); 

  // Memory allocation on device
  CUDA_CHECK_RETURN(hipMalloc(&points, 2 * hostNumberOfPoints * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&xOfNormals, 3 * hostNumberOfTriangles * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&yOfNormals, 3 * hostNumberOfTriangles * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&neighbors, 3 * hostNumberOfTriangles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMalloc(&forbidden, 3 * hostNumberOfTriangles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMalloc(&positionsOfNormalVectors, 3 * hostNumberOfTriangles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMalloc(&triangleIndices, 3 * hostNumberOfTriangles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMalloc(&betaValues, hostNumberOfPrisms * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&phiASE, hostNumberOfSamples * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&importance, hostNumberOfPrisms * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&indicesOfPrisms, hostRaysPerSample * sizeof(unsigned)));

  /// Copy data from host to device
  CUDA_CHECK_RETURN(hipMemcpy(points, (double*) &(pointsVector->at(0)), 2 * hostNumberOfPoints * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(xOfNormals, (double*) &(xOfNormalsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(yOfNormals, (double*) &(yOfNormalsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(neighbors,(int*) &(neighborsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(forbidden, (int*) &(forbiddenVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(positionsOfNormalVectors, (int*) &(positionsOfNormalVectorsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(triangleIndices, (unsigned*) &(triangleIndicesVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(betaValues, (double*) &(betaValuesVector->at(0)), hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(phiASE, hostPhiASE, hostNumberOfSamples * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(importance, hostImportance, hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));


  // Calculate Phi Ase foreach sample
  fprintf(stderr, "\nC Start Phi Ase calculation\n");
  hipEventRecord(start, 0);
  //for(int point_i = 0; point_i < hostNumberOfPoints ; ++point_i){
    //for(int level_i = 0; level_i < hostNumberOfLevels; ++level_i){
    int point_i = 1;
    int level_i = 0;{{
    // Importance for one sample
      unsigned realRaysPerSample = importanceSampling(point_i, level_i, hostImportance, hostNumberOfImportantRays, 
          (double*) &(pointsVector->at(0)), 
          (double*) &(xOfNormalsVector->at(0)), 
          (double*) &(yOfNormalsVector->at(0)),
          (int*) &(positionsOfNormalVectorsVector->at(0)), 
          (int*) &(neighborsVector->at(0)), 
          (int*) &(forbiddenVector->at(0)), 
          (double*) &(betaValuesVector->at(0)), 
          (double*) &(xOfTriangleCenterVector->at(0)),
          (double*) &(yOfTriangleCenterVector->at(0)), 
          (float*) &(surfacesVector->at(0)), 
          hostRaysPerSample,hostNumberOfPoints, hostNumberOfLevels, hostNumberOfTriangles, 
          hostThicknessOfPrism, hostSigmaA, hostSigmaE, hostNTot);

      // Prism scheduling for gpu threads
      int absoluteRay=0;
      for(int prism_i=0; prism_i < hostNumberOfPrisms; ++prism_i){
        for(int ray_i=0; ray_i < hostNumberOfImportantRays[prism_i]; ++ray_i){
          hostIndicesOfPrisms[absoluteRay++] = prism_i;
          assert(absoluteRay <= realRaysPerSample);
        }
      }
      assert(absoluteRay==realRaysPerSample);

      // Copy dynamic sample date to device
      CUDA_CHECK_RETURN(hipMemcpy(importance, hostImportance, hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));
      CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, realRaysPerSample * sizeof(unsigned), hipMemcpyHostToDevice));


      for(int print_i=0;print_i< 36;print_i++){
        //fprintf(stderr,"Prism %d: %d rays hostImportance=%f\n",print_i,hostNumberOfImportantRays[print_i],hostImportance[print_i]);
      }
      // Start Kernel
      calcSamplePhiAse<<< blocks, threads >>> ( devMTGPStates, phiASE, point_i, level_i, hostRaysPerThread, 
          points, xOfNormals, yOfNormals, positionsOfNormalVectors, 
          neighbors, forbidden, triangleIndices, betaValues, importance, 
          indicesOfPrisms,realRaysPerSample );

     // if(kernelcount==0)
     // {
     //   // Print experiment data
     //   testKernel<<<1,1>>>(points, xOfNormals, yOfNormals,
     //       neighbors, forbidden, positionsOfNormalVectors,
     //       triangleIndices, betaValues, phiASE, importance,
     //       indicesOfPrisms, hostNTot, hostSigmaA, hostSigmaE,
     //       hostNumberOfPoints, hostNumberOfTriangles, hostNumberOfLevels,
     //       hostThicknessOfPrism, hostCrystalFluorescence, 5);
     // }

      //if(level_i==0){
  //CUDA_CHECK_RETURN(hipMemcpy(hostPhiASE, phiASE, hostNumberOfPoints * hostNumberOfLevels * sizeof(float), hipMemcpyDeviceToHost));
      //printIntermediateValues(10, hostImportance, hostPhiASE, dndtAse,point_i);
      //}

      if(kernelcount % 200 == 0)
        fprintf(stderr, "C Sampling point %d done\n",kernelcount);
      kernelcount++;
    }
  }

  // Stop time
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&runtimeGpu, start, stop);

  // Calculate dndt Ase
  CUDA_CHECK_RETURN(hipMemcpy(hostPhiASE, phiASE, hostNumberOfPoints * hostNumberOfLevels * sizeof(float), hipMemcpyDeviceToHost));
  for(int sample_i=0; sample_i < hostNumberOfSamples; ++sample_i){
    hostPhiASE[sample_i] = float( (double(hostPhiASE[sample_i]) / (hostRaysPerSample * 4.0f * 3.14159))); //should be divided by realRaysPerSample for each samplepoint!
    double gain_local = double(hostNTot) * (betaCellsVector->at(sample_i)) * double(hostSigmaE + hostSigmaA) - double(hostNTot * hostSigmaA);
    dndtAse->at(sample_i) = gain_local * hostPhiASE[sample_i] / hostCrystalFluorescence;

  }


  // Free Memory
  hipFree(points);
  hipFree(xOfNormals);
  hipFree(yOfNormals);
  hipFree(neighbors);
  hipFree(forbidden);
  hipFree(positionsOfNormalVectors);
  hipFree(betaValues);
  hipFree(importance);


  hipDeviceReset();
  return runtimeGpu;
}

