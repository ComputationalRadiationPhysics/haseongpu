#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <calc_sample_phi_ase.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>

#define TEST_VALUES true
#define SMALL 1E-06
#define VERY_SMALL 0.0


/*********************************************************************************************
 * HOST FUNCTIONS
 *********************************************************************************************/
//----------------------------------------------------
// Host Code
//----------------------------------------------------
/** GPU Kernel Variables
 * The idea is, that the number of threads is fixed (to maximize GPU occupancy)
 * and the number of blocks as well (200 is the maximum for the standard
 * Mersenne Twister implementaion). Therefore, the number of rays per sample
 * are fixed to be k*200*256.
 * That means, sometimes we have to increase the number of rays a little.
 *
 * \var raysPerThread is used to give every thread k iterations (to simulate k rays)
 *
 * note that every samplepoint receives the exact same number of rays.
 */

/** Variables for the device
 * These are on-GPU representations of the input parameters
 * of variable size.
 *
 * \var p_in: coordinates of the sample-points of one layer (first all x-coordinates, then all y-coordinates)
 * \var n_*: values of the normal-vectors for the 3 rectangular sides of each prism (described in 2D)
 * \var beta_v: the beta values of the prisms
 * \var phi: the accumulated ASE-Flux for each sample point
 * \var forbidden: the side of the prism through which the ray "entered" the prism
 * \var n_p: the points where the normals (n_x,n_y) start
 * \var neighbors: indices to the adjacent triangles in t_in
 * \var t_in: indices of the points which are considered to be a triangle (A points start from 0, B points from size_t, C points from size_t*2)
 * \var cell_type: determines which cell type we are looking at.
 * other input parameters are put to the GPU by the setupGlobalVariablesKernel
 */
float calcDndtAse(
		  std::vector<double> *dndtAse, 
		  unsigned &threads, 
		  unsigned &blocks, 
		  unsigned &hostRaysPerSample,
		  std::vector<double> *betaValuesVector,
		  std::vector<double> *xOfNormalsVector,
		  std::vector<double> *yOfNormalsVector,
		  std::vector<unsigned> *cellTypesVector,
		  std::vector<unsigned> *triangleIndicesVector,
		  std::vector<int> *forbiddenVector,
		  std::vector<int> *neighborsVector,
		  std::vector<int> *positionsOfNormalVectorsVector,
		  std::vector<double> *pointsVector,
		  std::vector<double> *betaCellsVector,
		  std::vector<float> *surfacesVector,
		  std::vector<double> *xOfTriangleCenterVector,
		  std::vector<double> *yOfTriangleCenterVector,
		  float hostCladAbsorption,
		  unsigned hostCladNumber,
		  float hostNTot,
		  float hostSigmaA,
		  float hostSigmaE,
		  unsigned hostNumberOfPoints,
		  unsigned hostNumberOfTriangles,
		  unsigned hostNumberOfLevels,
		  float hostThicknessOfPrism,
		  float hostCrystalFluorescence)
{
  // Variable declarations
  // CPU
  double* hostImportance;
  unsigned* hostNumberOfImportantRays;
  int* hostIndicesOfPrisms;
  unsigned hostNumberOfPrisms;
  unsigned hostRaysPerThread;
  unsigned hostNumberOfSamples;
  hipEvent_t start, stop;
  float runtimeGpu;
  float *hostPhiASE;
  unsigned kernelcount;
  // GPU
  double  *points, *xOfNormals, *yOfNormals, *betaValues;
  float *phiASE;
  int *forbidden, *positionsOfNormalVectors, *neighbors, *triangleIndices, *cellTypes, *surfacesNormalized;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  double *importance;
  unsigned *indicesOfPrisms;
	
  // Variables defintions
  threads = 50; //OPTIMIZE: find perfect number of threads
  blocks = 200;
  hostNumberOfPrisms = (hostNumberOfTriangles * (hostNumberOfLevels-1));
  hostNumberOfSamples = hostNumberOfPoints * hostNumberOfLevels;

  
  hostPhiASE = (float*) malloc(hostNumberOfSamples * sizeof(float));
  hostImportance = (double*) malloc(hostNumberOfPrisms * sizeof(double));
  hostNumberOfImportantRays = (unsigned*) malloc(hostNumberOfPrisms * sizeof(unsigned));
  hostIndicesOfPrisms = (int*) malloc(hostRaysPerSample * sizeof(int));

  runtimeGpu = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  kernelcount = 0;

  for(int i=0; i < hostRaysPerSample; ++i){
    hostIndicesOfPrisms[i] = 0;

  }

  for(int i=0; i < hostNumberOfSamples; ++i){
    hostPhiASE[i] = 0.f;

  }

  for(int i=0; i < hostNumberOfPrisms; ++i){
    hostNumberOfImportantRays[i] = 1;
    hostImportance[i] = 1.0;
  }


  // Init mersenne twister PRNG
  {
    /**Allocate space for PRNG states on device */
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, blocks * sizeof(hiprandStateMtgp32)));

    /** Allocate space for MTGP kernel parameters */
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));

    /**Reformat from predefined parameter sets to kernel format,
     * and copy kernel parameters to device memory */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));

    /** Initialize one state per thread block */
    /** \TODO initialize with time */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, blocks, 1234));
  }

  // Allocation of memory on the GPU and setting of global GPU-variables
  {
    //fprintf(stderr, "\nC Filling the device Variables\n");
    //Create constant values on GPU
    setupGlobalVariablesKernel<<<1,1>>>(double(hostSigmaE), 
					double(hostSigmaA),
					hostCladNumber,
					double(hostCladAbsorption),
					double(hostNTot), 
					hostNumberOfTriangles, 
					double(hostThicknessOfPrism),
					hostNumberOfLevels, 
					hostNumberOfPoints); //@OPTIMIZE: initialize the constants as constants...

    hipDeviceSynchronize();

    // Memory allocation on device
    CUDA_CHECK_RETURN(hipMalloc(&points, 2 * hostNumberOfPoints * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&xOfNormals, 3 * hostNumberOfTriangles * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&yOfNormals, 3 * hostNumberOfTriangles * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&neighbors, 3 * hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&forbidden, 3 * hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&positionsOfNormalVectors, 3 * hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&triangleIndices, 3 * hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&cellTypes, hostNumberOfTriangles * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&betaValues, hostNumberOfPrisms * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&phiASE, hostNumberOfSamples * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc(&importance, hostNumberOfPrisms * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc(&indicesOfPrisms, hostRaysPerSample * sizeof(unsigned)));

    /// Copy data from host to device
    CUDA_CHECK_RETURN(hipMemcpy(points, (double*) &(pointsVector->at(0)), 2 * hostNumberOfPoints * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(xOfNormals, (double*) &(xOfNormalsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(yOfNormals, (double*) &(yOfNormalsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(neighbors,(int*) &(neighborsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(forbidden, (int*) &(forbiddenVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(positionsOfNormalVectors, (int*) &(positionsOfNormalVectorsVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(triangleIndices, (unsigned*) &(triangleIndicesVector->at(0)), 3 * hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(cellTypes, (unsigned*) &(cellTypesVector->at(0)), hostNumberOfTriangles * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(betaValues, (double*) &(betaValuesVector->at(0)), hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(phiASE, hostPhiASE, hostNumberOfSamples * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(importance, hostImportance, hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));
  }

  //fprintf(stderr, "C hostCrystalFluorescence: %e\n",hostCrystalFluorescence);
  // testKernel<<<1,1>>>(points, xOfNormals, yOfNormals, positionsOfNormalVectors, neighbors, forbidden, triangleIndices, cellTypes, betaValues,surface);

  // Start Kernels
  {
    fprintf(stderr, "C Start Kernel\n");
    hipEventRecord(start, 0);
		
    // Every Kernel calculates one sample point
    for(int point_i = 0; point_i < hostNumberOfPoints ; ++point_i){
      for(int level_i = 0; level_i < hostNumberOfLevels; ++level_i){
	hipDeviceSynchronize();

	// Reinit (TODO is really needed ?)
	for(int i=0; i < hostNumberOfPrisms; ++i){
	  hostNumberOfImportantRays[i] = 1;
	  hostImportance[i] = 1.0;
	}
	for(int i=0; i < hostRaysPerSample; ++i){
	  hostIndicesOfPrisms[i] = 0;

	}

	// Importance for one sample
	importanceSampling(point_i, level_i, hostImportance, hostNumberOfImportantRays, 
			   (double*) &(pointsVector->at(0)), 
			   (double*) &(xOfNormalsVector->at(0)), 
			   (double*) &(yOfNormalsVector->at(0)),
			   (int*) &(positionsOfNormalVectorsVector->at(0)), 
			   (int*) &(neighborsVector->at(0)), 
			   (int*) &(forbiddenVector->at(0)), 
			   (unsigned*) &(cellTypesVector->at(0)), 
			   (double*) &(betaValuesVector->at(0)), 
			   (double*) &(xOfTriangleCenterVector->at(0)),
			   (double*) &(yOfTriangleCenterVector->at(0)), 
			   (float*) &(surfacesVector->at(0)), 
			   hostRaysPerSample,hostNumberOfPoints, hostNumberOfLevels, hostNumberOfTriangles, 
			   hostThicknessOfPrism, hostSigmaA, hostSigmaE, hostCladNumber, hostCladAbsorption,hostNTot);

	CUDA_CHECK_RETURN(hipMemcpy(importance, hostImportance, hostNumberOfPrisms * sizeof(double), hipMemcpyHostToDevice));

	// Prism scheduling for gpu threads
	int index=0;
	for(int prism_i=0; prism_i < hostNumberOfPrisms; ++prism_i){
		for(int ray_i=0 ; ray_i < hostNumberOfImportantRays[prism_i]; ++ray_i){
			hostIndicesOfPrisms[index] = prism_i;
			index++;
			assert(index <= hostRaysPerSample);
		}
	}


	CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, hostRaysPerSample * sizeof(unsigned), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	calcSamplePhiAse<<< blocks, threads >>> ( devMTGPStates, phiASE, point_i, level_i, hostRaysPerThread, 
					      points, xOfNormals, yOfNormals, positionsOfNormalVectors, 
					      neighbors, forbidden, triangleIndices, cellTypes, betaValues, importance, 
					      surfacesNormalized,indicesOfPrisms,hostRaysPerSample );


	if(kernelcount % 200 == 0)
	   fprintf(stderr, "C Sampling point %d done\n",kernelcount);
	kernelcount++;
      }
    }

    hipDeviceSynchronize();
  }


  // Final calculations
  {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runtimeGpu, start, stop);
    CUDA_CHECK_RETURN(hipMemcpy(hostPhiASE, phiASE, hostNumberOfPoints * hostNumberOfLevels * sizeof(float), hipMemcpyDeviceToHost));

    for(int sample_i=0; sample_i < hostNumberOfSamples; ++sample_i){
      hostPhiASE[sample_i] = float( (double(hostPhiASE[sample_i]) / (hostRaysPerSample * 4.0f * 3.14159)));
      double gain_local = double(hostNTot) * (betaCellsVector->at(sample_i)) * double(hostSigmaE + hostSigmaA) - double(hostNTot * hostSigmaA);
      dndtAse->at(sample_i) = gain_local * hostPhiASE[sample_i] / hostCrystalFluorescence;
      	
      }
    
  }


  // Free Memory
  {
    hipFree(points);
    hipFree(xOfNormals);
    hipFree(yOfNormals);
    hipFree(neighbors);
    hipFree(forbidden);
    hipFree(positionsOfNormalVectors);
    hipFree(betaValues);
    hipFree(importance);
  }

  hipDeviceReset();
  return runtimeGpu;
}

