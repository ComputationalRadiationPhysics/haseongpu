#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <calc_sample_phi_ase.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>
#include <mesh.h>
#include <ctime> /* progressBar */
#include <progressbar.h> /*progressBar */

#define SEED 4321

float calcDndtAse (unsigned &threads, 
		   unsigned &blocks,
		   unsigned &hostRaysPerSample,
		   Mesh mesh,
		   Mesh hostMesh,
		   std::vector<double> *betaCellsVector,
		   float nTot,
		   std::vector<double> *hostSigmaA,
		   std::vector<double> *hostSigmaE,
		   float crystalFluorescence,
		   std::vector<double> *dndtAse
		   ){

  // Variable declaration
  // CPU
  double *hostImportance;
  unsigned *hostRaysPerPrism;
  float runtime;
  unsigned *hostIndicesOfPrisms;
  float *hostPhiAse;
  time_t starttime,progressStartTime;

  // GPU
  float *phiAse;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  double *importance;
  unsigned *indicesOfPrisms;
  float *sumPhi;
  unsigned *raysDump;
  unsigned *raysPerPrism;
  unsigned *cumulativeSums;
  double * sigmaA;
  double * sigmaE;

  //OPTIMIZE: find perfect number of threads - MUST be the same as the size of shared memory in kernel
  dim3 blockDim(256);
  dim3 gridDim(200, hostSigmaE->size());
  threads = blockDim.x;
  blocks = gridDim.x * gridDim.y;
    
  starttime = time(0);

  hostPhiAse          = (float*)    malloc (hostMesh.numberOfSamples * hostSigmaE->size() * sizeof(float));
  hostImportance      = (double*)   malloc (hostMesh.numberOfPrisms  * hostSigmaE->size() * sizeof(double));
  hostRaysPerPrism    = (unsigned*) malloc (hostMesh.numberOfPrisms  * hostSigmaE->size() * sizeof(unsigned));
  hostIndicesOfPrisms = (unsigned*) malloc (hostRaysPerSample        * hostSigmaE->size() * sizeof(unsigned));

  for(unsigned i=0; i < hostRaysPerSample * hostSigmaE->size(); ++i) hostIndicesOfPrisms[i] = 0;
  for(unsigned i=0; i < hostMesh.numberOfSamples * hostSigmaE->size(); ++i) hostPhiAse[i] = 0.f;
  for(unsigned i=0; i < hostMesh.numberOfPrisms * hostSigmaE->size(); ++i) hostRaysPerPrism[i] = 1;
  for(unsigned i=0; i < hostMesh.numberOfPrisms * hostSigmaE->size(); ++i) hostImportance[i] = 1.0;

  // CUDA Mersenne twister
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.y * gridDim.x  * sizeof(hiprandStateMtgp32)));

  // TODO maybe change seed for different mersenne twister
  CUDA_CALL(hipMalloc((void**)&devKernelParams, gridDim.y * sizeof(mtgp32_kernel_params)));
  for(unsigned mersenne_i = 0; mersenne_i < gridDim.y; ++mersenne_i){
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, &(devKernelParams[mersenne_i])));
    CURAND_CALL(hiprandMakeMTGP32KernelState(&(devMTGPStates[gridDim.x * mersenne_i]), mtgp32dc_params_fast_11213, &(devKernelParams[mersenne_i]), gridDim.x, SEED + mersenne_i));
  }

  // Memory allocation on device
  CUDA_CHECK_RETURN(hipMalloc(&phiAse, hostMesh.numberOfSamples * hostSigmaE->size() * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&importance, hostMesh.numberOfPrisms * hostSigmaE->size() * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&indicesOfPrisms, hostRaysPerSample * hostSigmaE->size() * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&raysPerPrism, hostMesh.numberOfPrisms * hostSigmaE->size() * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&sumPhi, hostSigmaE->size() * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&raysDump, hostSigmaE->size() * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&cumulativeSums,  hostMesh.numberOfPrisms * hostSigmaE->size() * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&sigmaA, hostSigmaE->size() * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&sigmaE, hostSigmaE->size() * sizeof(double)));

  // Copy host to device
  CUDA_CHECK_RETURN(hipMemcpy(phiAse, hostPhiAse, hostMesh.numberOfSamples * hostSigmaE->size() * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(sigmaA, &(hostSigmaA->at(0)), hostSigmaA->size() * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(sigmaE, &(hostSigmaE->at(0)), hostSigmaE->size() * sizeof(double), hipMemcpyHostToDevice));
  

  // Calculate Phi Ase foreach sample
  fprintf(stderr, "\nC Start Phi Ase calculation\n");
  progressStartTime = time(0);
  for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){

    hostRaysPerSample = importanceSampling(sample_i, mesh, hostRaysPerSample, sigmaA, sigmaE, nTot, importance, sumPhi, raysPerPrism, indicesOfPrisms, raysDump, cumulativeSums, blockDim, gridDim);

    CUDA_CHECK_RETURN(hipMemcpy(hostRaysPerPrism, raysPerPrism, hostMesh.numberOfPrisms * hostSigmaE->size() * sizeof(unsigned),hipMemcpyDeviceToHost));

    // Prism scheduling for gpu threads
    for(unsigned wave_i=0; wave_i < hostSigmaE->size(); ++wave_i){
      for(unsigned prism_i=0, absoluteRay = 0; prism_i < hostMesh.numberOfPrisms; ++prism_i){
    	for(unsigned ray_i=0; ray_i < hostRaysPerPrism[prism_i + hostMesh.numberOfPrisms * wave_i]; ++ray_i){
    	  hostIndicesOfPrisms[absoluteRay + hostRaysPerSample * wave_i] = prism_i;
    	  absoluteRay++;
    	  assert(absoluteRay <= hostRaysPerSample);
    	}
      }
    }

    // Copy dynamic sample data to device
    CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, hostRaysPerSample * hostSigmaE->size() * sizeof(unsigned), hipMemcpyHostToDevice));

    // Start Kernel
    calcSamplePhiAse<<< gridDim, blockDim >>>(devMTGPStates, mesh, indicesOfPrisms, importance, hostRaysPerSample, phiAse, sample_i, sigmaA, sigmaE, nTot);

    // update progressbar
    if((sample_i+1) % 10 == 0) fancyProgressBar(sample_i,hostMesh.numberOfSamples,60,progressStartTime);

  }
  // Copy solution back to host
  CUDA_CHECK_RETURN(hipMemcpy(hostPhiAse, phiAse, hostMesh.numberOfSamples * hostSigmaE->size() * sizeof(float), hipMemcpyDeviceToHost));

  // Calculate dndt Ase
  for(unsigned wave_i = 0; wave_i < hostSigmaE->size(); ++wave_i){
    for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){
      hostPhiAse[sample_i + hostMesh.numberOfSamples * wave_i] = float((double(hostPhiAse[sample_i + hostMesh.numberOfSamples * wave_i]) / (hostRaysPerSample * 4.0f * 3.14159)));
      double gain_local = double(nTot) * (betaCellsVector->at(sample_i)) * double(hostSigmaE->at(wave_i) + hostSigmaA->at(wave_i)) - double(nTot * hostSigmaA->at(wave_i));
      dndtAse->at(sample_i + hostMesh.numberOfSamples * wave_i) = gain_local * hostPhiAse[sample_i + hostMesh.numberOfSamples * wave_i] / crystalFluorescence;

    }
  }
  // Stop time
  runtime = difftime(time(0),starttime);


  // Free Memory
  free(hostPhiAse);
  free(hostImportance);
  free(hostRaysPerPrism);
  free(hostIndicesOfPrisms);
  hipFree(phiAse);
  hipFree(importance);
  hipFree(indicesOfPrisms);
  hipFree(raysPerPrism);
  hipFree(sumPhi);
  hipFree(raysDump);
  hipFree(sigmaA);
  hipFree(sigmaE);
  hipDeviceReset();

  return runtime;

}
