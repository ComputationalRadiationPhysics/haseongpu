#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <calc_sample_phi_ase.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>
#include <hip/hip_runtime_api.h>
#include <mesh.h>

#define SEED 1234

float calcDndtAse (unsigned &threads, 
		   unsigned &blocks, 
		   unsigned &hostRaysPerSample,
		   Mesh mesh,
		   Mesh hostMesh,
		   std::vector<double> *betaCellsVector,
		   float nTot,
		   float sigmaA,
		   float sigmaE,
		   float crystalFluorescence,
		   std::vector<double> *dndtAse){

  // Variable declaration
  // CPU
  double *hostImportance;
  unsigned *hostRaysPerPrism;
  hipEvent_t start, stop;
  float runtimeGpu;
  unsigned *hostIndicesOfPrisms;
  float *hostPhiAse;

  // GPU
  float *phiAse;
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;
  double *importance;
  unsigned *indicesOfPrisms;
  float *sumPhi;
  unsigned *raysDump;
  unsigned *raysPerPrism;

  //OPTIMIZE: find perfect number of threads - MUST be the same as the size of shared memory in kernel
  threads = 256; 
  blocks = 200;

  hostPhiAse          = (float*)    malloc (hostMesh.numberOfSamples * sizeof(float));
  hostImportance      = (double*)   malloc (hostMesh.numberOfPrisms  * sizeof(double));
  hostRaysPerPrism    = (unsigned*) malloc (hostMesh.numberOfPrisms  * sizeof(unsigned));
  hostIndicesOfPrisms = (unsigned*) malloc (hostRaysPerSample        * sizeof(unsigned));
  runtimeGpu = 0.0;

  for(unsigned i=0; i < hostRaysPerSample; ++i) hostIndicesOfPrisms[i] = 0;
  for(unsigned i=0; i < hostMesh.numberOfSamples; ++i) hostPhiAse[i] = 0.f;
  for(unsigned i=0; i < hostMesh.numberOfPrisms; ++i) hostRaysPerPrism[i] = 1;
  for(unsigned i=0; i < hostMesh.numberOfPrisms; ++i) hostImportance[i] = 1.0;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Init mersenne twister PRNG
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, blocks * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
  CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
  CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, mtgp32dc_params_fast_11213, devKernelParams, blocks, SEED));

  // Memory allocation on device
  CUDA_CHECK_RETURN(hipMalloc(&phiAse, hostMesh.numberOfSamples * sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&importance, hostMesh.numberOfPrisms * sizeof(double)));
  CUDA_CHECK_RETURN(hipMalloc(&indicesOfPrisms, hostRaysPerSample * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&raysPerPrism, hostMesh.numberOfPrisms * sizeof(unsigned)));
  CUDA_CHECK_RETURN(hipMalloc(&sumPhi, sizeof(float)));
  CUDA_CHECK_RETURN(hipMalloc(&raysDump, sizeof(unsigned)));

  // Copy host to device
  CUDA_CHECK_RETURN(hipMemcpy(phiAse, hostPhiAse, hostMesh.numberOfSamples * sizeof(float), hipMemcpyHostToDevice));

  // Calculate Phi Ase foreach sample
  fprintf(stderr, "\nC Start Phi Ase calculation\n");
  hipEventRecord(start, 0);
  for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){
    if(sample_i % 200 == 0) fprintf(stderr, "C Sampling point %d/%d done\n", sample_i, hostMesh.numberOfSamples);

    importanceSampling(sample_i, mesh, hostRaysPerSample, sigmaA, sigmaE, nTot, importance, sumPhi, raysPerPrism, indicesOfPrisms, raysDump, threads, blocks);
    CUDA_CHECK_RETURN(hipMemcpy(hostRaysPerPrism,raysPerPrism, hostMesh.numberOfPrisms*sizeof(unsigned),hipMemcpyDeviceToHost));

    // Prism scheduling for gpu threads
    for(unsigned prism_i=0, absoluteRay = 0; prism_i < hostMesh.numberOfPrisms; ++prism_i){
      for(unsigned ray_i=0; ray_i < hostRaysPerPrism[prism_i]; ++ray_i){
        hostIndicesOfPrisms[absoluteRay++] = prism_i;
        assert(absoluteRay <= hostRaysPerSample);
      }
    }

    // Copy dynamic sample data to device
    CUDA_CHECK_RETURN(hipMemcpy(indicesOfPrisms, hostIndicesOfPrisms, hostRaysPerSample * sizeof(unsigned), hipMemcpyHostToDevice));

    // Start Kernel
    calcSamplePhiAse<<< blocks, threads >>>(devMTGPStates, mesh, indicesOfPrisms, importance, hostRaysPerSample, phiAse, sample_i, sigmaA, sigmaE, nTot);

  }
  // Copy solution back to host
  CUDA_CHECK_RETURN(hipMemcpy(hostPhiAse, phiAse, hostMesh.numberOfSamples * sizeof(float), hipMemcpyDeviceToHost));

  // Stop time
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&runtimeGpu, start, stop);

  // Calculate dndt Ase
  for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){
    hostPhiAse[sample_i] = float( (double(hostPhiAse[sample_i]) / (hostRaysPerSample * 4.0f * 3.14159)));
    double gain_local = double(nTot) * (betaCellsVector->at(sample_i)) * double(sigmaE + sigmaA) - double(nTot * sigmaA);
    dndtAse->at(sample_i) = gain_local * hostPhiAse[sample_i] / crystalFluorescence;
        
  }

  // Free Memory
  free(hostPhiAse);
  free(hostImportance);
  free(hostRaysPerPrism);
  free(hostIndicesOfPrisms);
  hipFree(phiAse);
  hipFree(importance);
  hipFree(indicesOfPrisms);
  hipFree(raysPerPrism);
  hipFree(sumPhi);
  hipFree(raysDump);
  hipDeviceReset();

  return runtimeGpu;

}
