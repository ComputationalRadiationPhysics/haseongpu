#include "hip/hip_runtime.h"
#include "calc_dndt_ase.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <assert.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <cudachecks.h>
#include <importance_sampling.h>
#include <test_functions.h>
#include <cuda_utils.h> /* copyToDevice, copyFromDevice */
#include "calc_sample_phi_ase.h"
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>

/* include MTGP pre-computed parameter sets */
/* include <rocrand/rocrand_mtgp32_11213.h> */

#include <hip/hip_runtime_api.h>
#include <mesh.h>
#include <ctime> /* progressBar */
#include <progressbar.h> /*progressBar */


#define SEED 1234

/**
 * @brief Calculates which ray should start in which prism. Thus
 *        every thread in on gpu knows the index of the prism
 *        where its rays starts.
 *
 **/
void calcIndicesOfPrism(std::vector<unsigned> &indicesOfPrisms, std::vector<unsigned> &numberOfReflections, std::vector<unsigned> raysPerPrism, unsigned reflectionSlices, unsigned raysPerSample, Mesh mesh){
  // Init vectors with zero (slow and not needed anymore)
  // for(unsigned i=0;  i < indicesOfPrisms.size() ; ++i) indicesOfPrisms[i] = 0;
  // for(unsigned i=0;  i < numberOfReflections.size() ; ++i) numberOfReflections[i] = 0;

  // Calc new values
  unsigned absoluteRay = 0;
  for(unsigned reflection_i =0; reflection_i < reflectionSlices; ++reflection_i){
    for(unsigned prism_i=0; prism_i < mesh.numberOfPrisms; ++prism_i){
      unsigned reflectionOffset = reflection_i * mesh.numberOfPrisms;
      for(unsigned ray_i=0; ray_i < raysPerPrism[prism_i + reflectionOffset]; ++ray_i){
        indicesOfPrisms[absoluteRay] = prism_i;
        numberOfReflections[absoluteRay] = reflection_i;
        absoluteRay++;
        assert(absoluteRay <= raysPerSample);

      }

    }

  }

}

double calcExpectation(double phiAse, double phiAseSquare, unsigned raysPerSample){
  double a = phiAseSquare / raysPerSample;
  double b = (phiAse / raysPerSample) * (phiAse / raysPerSample);

  return sqrt(abs((a - b) / raysPerSample));
}

float calcDndtAse (unsigned &threads, 
		   unsigned &blocks,
		   unsigned &hostRaysPerSample,
		   unsigned maxRaysPerSample,
		   Mesh mesh,
		   Mesh hostMesh,
		   std::vector<double> hostSigmaA,
		   std::vector<double> hostSigmaE,
		   float expectationThreshold,
		   bool useReflections,
		   std::vector<double> &dndtAse,
		   std::vector<float> &hostPhiAse,
		   std::vector<double> &expectation
		   ){

  // Variable declaration
  // CPU
  float runtime;
  time_t starttime,progressStartTime;
  unsigned hostRaysPerSampleSave;
  unsigned maxReflections;
  unsigned reflectionSlices;
  bool distributeRandomly;

  std::cout << hostRaysPerSample << std::endl;
  std::cout << maxRaysPerSample << std::endl;

  // GPU
  hiprandStateMtgp32 *devMTGPStates;
  mtgp32_kernel_params *devKernelParams;

  // Variable Definitions
  dim3 blockDim(256);
  dim3 gridDim(200, hostSigmaE.size());
  threads = blockDim.x;
  blocks = gridDim.x;

  starttime = time(0);
  hostRaysPerSampleSave = hostRaysPerSample;

  if(useReflections){
    maxReflections = hostMesh.getMaxReflections(); 
  }
  else {
    maxReflections = 0;
  }

  reflectionSlices = 1 + 2 * maxReflections;
  distributeRandomly = true;

  // Memory allocation on host
  std::vector<unsigned> hostIndicesOfPrisms(maxRaysPerSample, 0);
  std::vector<unsigned> hostNumberOfReflections(maxRaysPerSample, 0);
  std::vector<double>   hostImportance(hostMesh.numberOfPrisms * reflectionSlices, 0);
  std::vector<unsigned> hostRaysPerPrism(hostMesh.numberOfPrisms * reflectionSlices, 1);
  std::vector<float>    hostPhiAseSquare(hostMesh.numberOfSamples * gridDim.y, 0);

  // Memory allocation/init and copy for device
  unsigned *indicesOfPrisms     = copyToDevice(hostIndicesOfPrisms);
  unsigned *numberOfReflections = copyToDevice(hostNumberOfReflections);
  unsigned *raysPerPrism        = copyToDevice(hostRaysPerPrism);
  double   *importance          = copyToDevice(hostImportance);
  float    *phiAseSquare        = copyToDevice(hostPhiAseSquare);
  float    *phiAse              = copyToDevice(hostPhiAse);
  
  // CUDA Mersenne twister for more than 200 blocks (for every wavelength)
  CUDA_CALL(hipMalloc((void **)&devMTGPStates, gridDim.x  * sizeof(hiprandStateMtgp32)));
  CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));

  // TODO remove unused states (if using only 1 wavelength at a time...)
  for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, &(devKernelParams[wave_i])));
    CURAND_CALL(hiprandMakeMTGP32KernelState(&(devMTGPStates[gridDim.x * wave_i]), mtgp32dc_params_fast_11213, &(devKernelParams[wave_i]), gridDim.x, SEED + wave_i));
  }

  // Calculate Phi Ase foreach sample
  fprintf(stderr, "\nC Start Phi Ase calculation\n");
  progressStartTime = time(0);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  std::vector<unsigned> centerSample(expectation.size(), 0);

  for(unsigned wave_i = 0; wave_i < gridDim.y; ++wave_i){
    for(unsigned sample_i = 0; sample_i < hostMesh.numberOfSamples; ++sample_i){
      int sampleOffset = sample_i + hostMesh.numberOfSamples * wave_i;
      hostRaysPerSample = hostRaysPerSampleSave;

      while(true){
        importanceSampling(sample_i, reflectionSlices, mesh, hostRaysPerSample, hostSigmaA[wave_i], hostSigmaE[wave_i], importance, raysPerPrism, distributeRandomly, blockDim, gridDim);
	copyFromDevice(hostRaysPerPrism, raysPerPrism);

        // Prism scheduling for gpu threads
        calcIndicesOfPrism(hostIndicesOfPrisms, hostNumberOfReflections, hostRaysPerPrism, reflectionSlices, hostRaysPerSample, hostMesh);
	copyToDevice(hostIndicesOfPrisms, indicesOfPrisms);
	copyToDevice(hostNumberOfReflections, numberOfReflections);

	// TESTING OUTPUT
	 if(sample_i == 1386)
	   centerSample.assign(hostRaysPerPrism.begin(), hostRaysPerPrism.end());

        // Start Kernel
        calcSamplePhiAse<<< 200, blockDim >>>(devMTGPStates, mesh, indicesOfPrisms, wave_i, numberOfReflections, importance, hostRaysPerSample, phiAse, phiAseSquare, sample_i, hostSigmaA[wave_i], hostSigmaE[wave_i]);

        // Copy solution (for this samplepoint) back to host
	hostPhiAse[sampleOffset]       = copyFromDevice(&(phiAse[sampleOffset]));
	hostPhiAseSquare[sampleOffset] = copyFromDevice(&(phiAseSquare[sampleOffset]));

        // Check square error
        expectation.at(sampleOffset) = calcExpectation(hostPhiAse.at(sampleOffset), hostPhiAseSquare[sampleOffset], hostRaysPerSample);

        if(expectation.at(sampleOffset) < expectationThreshold) break;
        if((hostRaysPerSample * 10) > maxRaysPerSample)         break;

        // fprintf(stderr,"increasing from %d to %d\n",hostRaysPerSample, hostRaysPerSample*10);
        // If the threshold is still too high, increase the number of rays and reset the previously calculated value
        hostRaysPerSample *= 10;
        hostPhiAse.at(sampleOffset) = 0;
        hostPhiAseSquare[sampleOffset] = 0;
	copyToDevice(hostPhiAse[sampleOffset], &(phiAse[sampleOffset]));
	copyToDevice(hostPhiAseSquare[sampleOffset], &(phiAseSquare[sampleOffset]));

      }
      // Update progressbar
      if((sample_i+1) % 10 == 0) fancyProgressBar(sample_i,hostMesh.numberOfSamples,60,progressStartTime);

      // Calculate dndt Ase, after one point is completely sampled
      hostPhiAse.at(sampleOffset) = float((double(hostPhiAse.at(sampleOffset)) / (hostRaysPerSample * 4.0f * 3.14159)));
      double gain_local = double(hostMesh.nTot) * hostMesh.betaCells[sample_i] * double(hostSigmaE[wave_i] + hostSigmaA[wave_i]) - double(hostMesh.nTot * hostSigmaA[wave_i]);
      dndtAse.at(sampleOffset) = gain_local * hostPhiAse.at(sampleOffset) / hostMesh.crystalFluorescence;


    }
  }

  // Stop time
  runtime = difftime(time(0),starttime);

  // TESTING OUTPUT
   expectation.assign(centerSample.begin(), centerSample.end());

  // Free Memory
  hipFree(phiAse);
  hipFree(importance);
  hipFree(indicesOfPrisms);
  hipFree(raysPerPrism);
  hipFree(numberOfReflections);
  hipFree(phiAseSquare);
  hipDeviceReset();

  return runtime;
}

