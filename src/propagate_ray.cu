#include "hip/hip_runtime.h"
#include <mesh.h>
#include <stdio.h>
#include <geometry.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h> /* printf */

__host__ __device__ double checkSurface(const int currentLevel, const double zPos, const double zVec, const double length, const double thickness){
  double denominator = zVec;
  if (denominator != 0.0){
    double nominator = currentLevel * thickness - zPos;
    double lengthTmp = nominator/denominator;
    if (lengthTmp <= length && lengthTmp > 0.0){
      return lengthTmp;
    }

  }
  return 0;
}

__host__ __device__ double checkEdge(const Triangle triangle, const int edge, const Ray ray, const double length){
  double denominator = triangle.edges[edge].normal.dir.x * ray.dir.x + triangle.edges[edge].normal.dir.y * ray.dir.y;
  if (denominator != 0.0)
    {
      double nominator =	  
	triangle.edges[edge].normal.dir.x * triangle.edges[edge].normal.p.x
	+ triangle.edges[edge].normal.dir.y * triangle.edges[edge].normal.p.y
	- triangle.edges[edge].normal.dir.x * ray.p.x 
	- triangle.edges[edge].normal.dir.y * ray.p.y; 

      double lengthTmp = nominator/denominator;
      if(lengthTmp <= length && lengthTmp > 0.0){
	return lengthTmp;
      }

    }
  
  return 0;
}

__host__ __device__ int calcTriangleRayIntersection(double *length, const Triangle triangle,  const Ray ray, const unsigned level, const int forbiddenEdge, const double thickness){
  int edge = -1;
  // Check 3 edges of triangle
  for(int edge_i = 0; edge_i < 3; ++edge_i){
    if(edge_i != forbiddenEdge){
      double lengthTmp = checkEdge(triangle, edge_i, ray, *length);
      if(lengthTmp){
	*length = lengthTmp;
	edge = edge_i;
      }
    }
  }
  
  // check the upper surface
  if (forbiddenEdge != 3){
    double lengthTmp = checkSurface(level + 1, ray.p.z, ray.dir.z, *length, thickness);
    if(lengthTmp){
      *length = lengthTmp;
      edge = 3;
    }
  }

  // check the lower surface
  if (forbiddenEdge != 4){
    double lengthTmp = checkSurface(level, ray.p.z, ray.dir.z, *length, thickness);
    if (lengthTmp){
      *length = lengthTmp;
      edge = 4;
    }
  }
  return edge;
}


__host__ __device__ Ray calcNextRay(Ray ray, const double length){
  ray.p.x = ray.p.x + length * ray.dir.x;
  ray.p.y = ray.p.y + length * ray.dir.y;
  ray.p.z = ray.p.z + length * ray.dir.z;

  return ray;

}

__host__ __device__ double calcPrismGain(const Triangle triangle, const unsigned level, const double length, const double sigmaA, const double sigmaE, const double nTot){
  return (double) exp(nTot * (triangle.betaValues[level] * ( sigmaE + sigmaA ) - sigmaA ) * length);
 
}

__host__ __device__ void updateFromEdge(Triangle *triangle, int *forbiddenEdge, unsigned *level, const int edge){
   switch(edge){
   case 0:
   case 1:
   case 2:
     // One of three edges
     *forbiddenEdge = triangle->edges[edge].forbidden;
     *triangle = *(triangle->edges[edge].neighbor);
     break;

   case 3:
     // Upper surface
     *forbiddenEdge = 4;
     (*level)++;
     break;

   case 4:
     // Lower surface
     *forbiddenEdge = 3;
     (*level--);
     break;

  }

}

__host__ __device__ double propagateRay(Ray nextRay, unsigned nextLevel, Triangle nextTriangle, const double sigmaA, const double sigmaE, const double nTot, const double thickness){
  double distanceTotal     = nextRay.length;
  double distanceRemaining = nextRay.length;
  double length  = 0;
  double gain    = 1;
  int nextForbiddenEdge = -1;
  int nextEdge          = -1;

  nextRay = normalizeRay(nextRay);
  while(fabs(distanceRemaining) > SMALL){
    // Calc gain for triangle intersection
    length             = distanceRemaining;
    nextEdge           = calcTriangleRayIntersection(&length, nextTriangle, nextRay, nextLevel, nextForbiddenEdge, thickness);
    nextRay            = calcNextRay(nextRay, length);
    gain              *= calcPrismGain(nextTriangle, nextLevel, length, sigmaA, sigmaE, nTot);
    distanceRemaining -= length;

    // Calc nextTriangle, nextForbiddenEdge and nextLevel
    if(nextEdge != -1){
      updateFromEdge(&nextTriangle, &nextForbiddenEdge, &nextLevel, nextEdge);
    }

  }

  return gain /= (distanceTotal * distanceTotal);
}
