#include "hip/hip_runtime.h"
#include "propagate_ray.h"
#include <mesh.h>
#include <stdio.h>
#include <geometry.h>
#include <reflection.h> /* calcNextReflection */
#include <hip/hip_runtime_api.h>
#include <stdio.h> /* printf */
#include <assert.h> /* assert */

/**
 * @brief Checks a level-plane(currentLevel * thickness) for intersection with an ray (zPos, zVec).
 *        If the intersection-length is greater then length. 
 *        Than the intersection-length will be returned. 
 *        Otherwise 0 will be returned.
 *
 * @return intersection-length if intersection-length <= length
 * @return 0 if intersection-length > length
 *
 **/
__device__ double checkSurface(const int currentLevel, const double zPos, const double zVec, const double length, const double thickness){
  double denominator = zVec;
  if (denominator != 0.0){
    double nominator = currentLevel * thickness - zPos;
    double lengthTmp = nominator/denominator;
    if (lengthTmp <= length && lengthTmp > 0.0){
      return lengthTmp;
    }

  }
  return 0;
}

/**
 * @brief Checks an edges of the given triangle/prism for an intersection
 *        with ray and calculates the intersection-length. If the intersection-length
 *        is greater then length. Than the intersection-length will be
 *        returned. Otherwise 0 will be returned.
 *
 * @return intersection-length if intersection-length <= length
 * @return 0 if intersection-length > length
 **/
__device__ double checkEdge(const unsigned triangle, const int edge, const Ray ray, const Mesh &mesh, const double length){
  NormalRay normal = mesh.getNormal(triangle, edge);
  double denominator = normal.dir.x * ray.dir.x + normal.dir.y * ray.dir.y;

  if (denominator != 0.0)
    {
      double nominator =	  
	normal.dir.x * normal.p.x
	+ normal.dir.y * normal.p.y
	- normal.dir.x * ray.p.x 
	- normal.dir.y * ray.p.y; 

      double lengthTmp = nominator/denominator;
      if(lengthTmp <= length && lengthTmp > 0.0){
	return lengthTmp;
      }

    }
  
  return 0;
}

/**
 * @brief Calculates the intersection-length for the propagated ray and
 *        the current triangle.
 *
 * @return edge number of the intesected edge (-1 for no intersection)
 *
 **/
__device__ int calcTriangleRayIntersection(double *length, const unsigned triangle,  const Ray ray, const unsigned level, const int forbiddenEdge, const Mesh &mesh){
  int edge = -1;
  // Check 3 edges of triangle
  for(int edge_i = 0; edge_i < 3; ++edge_i){
    if(edge_i != forbiddenEdge){
      double lengthTmp = checkEdge(triangle, edge_i, ray, mesh, *length);
      if(lengthTmp){
	*length = lengthTmp;
	edge = edge_i;
      }
    }
  }
  
  // check the upper surface
  if (forbiddenEdge != 3){
    double lengthTmp = checkSurface(level + 1, ray.p.z, ray.dir.z, *length, mesh.thickness);
    if(lengthTmp){
      *length = lengthTmp;
      edge = 3;
    }
  }

  // check the lower surface
  if (forbiddenEdge != 4){
    double lengthTmp = checkSurface(level, ray.p.z, ray.dir.z, *length, mesh.thickness);
    if (lengthTmp){
      *length = lengthTmp;
      edge = 4;
    }
  }
  return edge;
}

/**
 * @brief This is simple vector calculation. The startpoint
 *        of ray will be moved by length.
 * 
 * @return ray is the ray with moved startpoint
 *
 **/
__device__ Ray calcNextRay(Ray ray, const double length){
  ray.p.x = ray.p.x + length * ray.dir.x;
  ray.p.y = ray.p.y + length * ray.dir.y;
  ray.p.z = ray.p.z + length * ray.dir.z;

  return ray;

}

/**
 * @brief Calculates the gain for the given prism(triangle and level) and 
 *        the intersection-length of the ray.
 *
 * @return gain
 *
 **/
__device__ double calcPrismGain(const unsigned triangle, const unsigned level, const double length, const Mesh &mesh, const double sigmaA, const double sigmaE){
  if (mesh.getCellType(triangle) == mesh.cladNumber){
    return exp(-(mesh.cladAbsorption) * length);
  }
  else {
     return (double) exp(mesh.nTot * (mesh.getBetaValue(triangle, level) * ( sigmaE + sigmaA ) - sigmaA ) * length);
   }
 
}

/**
 * @brief Sets the next triangle, next forbiddenEdge 
 *        and next level depending on the cutted edge of 
 *        the current triangle and the propagated ray.
 *
 **/
__device__ void updateFromEdge(unsigned *triangle, int *forbiddenEdge, unsigned *level, const Mesh &mesh, const int edge){
   switch(edge){
   case 0:
   case 1:
   case 2:
     // One of three edges
     *forbiddenEdge = mesh.getForbiddenEdge(*triangle, edge);
     *triangle = mesh.getNeighbor(*triangle, edge);
     break;

   case 3:
     // Upper surface
     *forbiddenEdge = 4;
     if(*level != mesh.numberOfLevels) (*level)++;
     break;

   case 4:
     // Lower surface
     *forbiddenEdge = 3;
     if(*level != 0) (*level)--;
     break;

  }

}

__device__ double propagateRay(Ray nextRay, unsigned *nextLevel, unsigned *nextTriangle, const Mesh &mesh, 
			       const double sigmaA, const double sigmaE){
  double distanceTotal     = nextRay.length;
  double distanceRemaining = nextRay.length;
  double length  = 0;
  double gain    = 1;
  int nextForbiddenEdge = -1;
  int nextEdge          = -1;

  // Length to small, could be same points
  if(distanceTotal < SMALL)
     return 1;

  nextRay = normalizeRay(nextRay);
  while(fabs(distanceRemaining) > SMALL){
    assert(*nextLevel <= mesh.numberOfLevels);
    // Calc gain for triangle intersection
    length             = distanceRemaining;
    nextEdge           = calcTriangleRayIntersection(&length, *nextTriangle, nextRay, *nextLevel, nextForbiddenEdge, mesh);
    nextRay            = calcNextRay(nextRay, length);
    double gainTmp     = calcPrismGain(*nextTriangle, *nextLevel, length, mesh, sigmaA, sigmaE);
    gain              *= gainTmp;
    assert(length >= 0);

    distanceRemaining -= length;

    // Calc nextTriangle, nextForbiddenEdge and nextLevel
    if(nextEdge != -1){
      updateFromEdge(nextTriangle, &nextForbiddenEdge, nextLevel, mesh, nextEdge);
    }

  }

  return gain;
}


__device__ double propagateRayWithReflection(Point startPoint, 
					     const Point endPoint, 
					     const unsigned reflections, 
					     ReflectionPlane reflectionPlane, 
					     unsigned startLevel, 
					     unsigned startTriangle, 
					     const Mesh &mesh, 
					     const double sigmaA, 
					     const double sigmaE){

  double distanceTotal = 0;
  double gain = 1.0;

  for(unsigned reflection = 0; reflection < reflections; ++reflection){
    float reflectivity = mesh.getReflectivity(reflectionPlane, startTriangle);;
    float totalReflectionAngle = mesh.getReflectionAngle(reflectionPlane);
    Point reflectionPoint = {0,0,0};
    double reflectionAngle = 0;

    // Calc reflectionPoint and reflectionAngle
    calcNextReflection(startPoint, endPoint, (reflections - reflection), reflectionPlane, &reflectionPoint, &reflectionAngle, mesh);
    Ray reflectionRay   = generateRay(startPoint, reflectionPoint);
    distanceTotal += reflectionRay.length;
    gain  *= propagateRay(reflectionRay, &startLevel, &startTriangle, mesh, sigmaA, sigmaE);

    assert(reflectionAngle <= 90);
    assert(reflectionAngle >= 0 );

    if(reflectionAngle <= totalReflectionAngle) 
      gain             *= reflectivity;

    startPoint          = reflectionPoint;
    reflectionPlane     = reflectionPlane == TOP_REFLECTION ? BOTTOM_REFLECTION : TOP_REFLECTION;
    
    }

  Ray ray = generateRay(startPoint, endPoint);
  gain  *= propagateRay(ray, &startLevel, &startTriangle, mesh, sigmaA, sigmaE);
  distanceTotal += ray.length;
  
  
  return gain / (distanceTotal * distanceTotal);

}
