#include "hip/hip_runtime.h"
// Libraries
#include <stdio.h> /* fprintf, memcpy, strstr, strcmp */
#include <assert.h> /* assert */
#include <string> /* string */
#include <vector> /* vector */
#include <stdlib.h> /* atoi */
#include <pthread.h> /* pthread_t, pthread_join */
#include <algorithm> /* max_element */
#include <numeric> /* accumulate*/

// User header files
#include <calc_phi_ase.h>
#include <calc_phi_ase_threaded.h>
#include <calc_phi_ase_mpi.h>
#include <parser.h>
#include <write_to_vtk.h>
#include <write_matlab_output.h>
#include <for_loops_clad.h>
#include <cudachecks.h>
#include <mesh.h>
#include <test_environment.h>

#include <logging.h>
#include <ray_histogram.h>

#define MIN_COMPUTE_CAPABILITY_MAJOR 2
#define MIN_COMPUTE_CAPABILITY_MINOR 0
unsigned verbosity = V_ERROR | V_INFO | V_WARNING; // extern through logging.h


/** 
 * @brief Queries for devices on the running mashine and collects
 *        them on the devices array. Set the first device in this 
 *        array as computaion-device. On Errors the programm will
 *        be stoped by exit(). Otherwise you can set the device by command
 *        line parameter --device=
 * 
 * @param verbose > 0 prints debug output
 * 
 * @return vector of possible devices
 */
std::vector<unsigned> getCorrectDevice(unsigned maxGpus){
  hipDeviceProp_t prop;
  int minMajor = MIN_COMPUTE_CAPABILITY_MAJOR;
  int minMinor = MIN_COMPUTE_CAPABILITY_MINOR;
  int count;
  std::vector<unsigned> devices;

  // Get number of devices
  CUDA_CHECK_RETURN( hipGetDeviceCount(&count));

  // Check devices for compute capability and if device is busy
  unsigned devicesAllocated = 0;
  for(int i=0; i < count; ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, i) );
    if( (prop.major > minMajor) || (prop.major == minMajor && prop.minor >= minMinor) ){
      hipSetDevice(i);
      int* occupy; //TODO: occupy gets allocated, but never hipFree'd -> small memory leak!
      if(hipMalloc((void**) &occupy, sizeof(int)) == hipSuccess){
        devices.push_back(i);
        devicesAllocated++;
        if(devicesAllocated == maxGpus)
          break;

      }

    }

  }

  if(devices.size() == 0){
    dout(V_ERROR) << "None of the free CUDA-capable devices is sufficient!" << std::endl;
    exit(1);
  }

  hipSetDevice(devices.at(0));

  dout(V_INFO) << "Found " << int(devices.size()) << " available CUDA devices with Compute Capability >= " << minMajor << "." << minMinor << "):" << std::endl;
  for(unsigned i=0; i<devices.size(); ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, devices[i]) );
    dout(V_INFO) << "[" << devices[i] << "] " << prop.name << " (Compute Capability " << prop.major << "." << prop.minor << ")" << std::endl;
  }

  return devices;

}

double calcDndtAse(const Mesh& mesh, const double sigmaA, const double sigmaE, const float phiAse, const unsigned sample_i){
  double gain_local = mesh.nTot * mesh.betaCells[sample_i] * (sigmaE + sigmaA) - double(mesh.nTot * sigmaA);
  return gain_local * phiAse / mesh.crystalFluorescence;
}


int main(int argc, char **argv){
  unsigned raysPerSample = 0;
  unsigned maxRaysPerSample = 0;
  unsigned maxRepetitions = 4;
  float maxMSE = 0;
  float  avgMSE = 0;
  unsigned highMSE = 0;
  std::string runmode("");
  std::string compareLocation("");
  float runtime = 0.0;
  bool writeVtk = false;
  bool useReflections = false;
  std::vector<unsigned> devices; // will be assigned in getCOrrectDevice();
  unsigned maxGpus = 0;
  RunMode mode = NONE;
  int minSampleRange = 0;
  int maxSampleRange = 0;
  time_t starttime   = time(0);
  unsigned usedGpus  = 0;

  std::string inputPath;
  std::string outputPath;
  verbosity = 31; //ALL //TODO: remove in final code

  // Wavelength data
  std::vector<double> sigmaA;
  std::vector<double> sigmaE;
  std::vector<float> mseThreshold;

  // Parse Commandline
  parseCommandLine(argc, argv, &raysPerSample, &maxRaysPerSample, &inputPath,
		   &writeVtk, &compareLocation, &mode, &useReflections, &maxGpus, &minSampleRange, &maxSampleRange, &maxRepetitions, &outputPath);

  // Set/Test device to run experiment with
  //TODO: this call takes a LOT of time (2-5s). Can this be avoided?
  //TODO: maybe move this to a place where GPUs are actually needed (for_loops_clad doesn't even need GPUs!)
  devices = getCorrectDevice(maxGpus);


  // sanity checks
  if(checkParameterValidity(argc, raysPerSample, &maxRaysPerSample, inputPath, devices.size(), mode, &maxGpus, minSampleRange, maxSampleRange, maxRepetitions, outputPath)) return 1;

  // Parse wavelengths from files
  if(fileToVector(inputPath + "sigma_a.txt", &sigmaA)) return 1;
  if(fileToVector(inputPath + "sigma_e.txt", &sigmaE)) return 1;
  if(fileToVector(inputPath + "mse_threshold.txt", &mseThreshold)) return 1;
  assert(sigmaA.size() == sigmaE.size());
  assert(mseThreshold.size() == sigmaE.size());


  // Parse experientdata and fill mesh
  Mesh hMesh;
  std::vector<Mesh> dMesh(maxGpus);

  // TODO: split into hMesh and dMesh parsing 
  // -> parse dMesh only where needed
  if(Mesh::parseMultiGPU(hMesh, dMesh, inputPath, devices, maxGpus)) return 1;

  // Solution vector
  std::vector<double> dndtAse(hMesh.numberOfSamples * sigmaE.size(), 0);
  std::vector<float>  phiAse(hMesh.numberOfSamples * sigmaE.size(), 0);
  std::vector<double> mse(hMesh.numberOfSamples * sigmaE.size(), 1000);
  std::vector<unsigned> totalRays(hMesh.numberOfSamples * sigmaE.size(), 0);

  // for(unsigned i = 0; i < hMesh.numberOfPrisms; ++i){
  //   dout(V_DEBUG) << i << " " << hMesh.betaValues[i] << std::endl;
  // }

  // Run Experiment
  std::vector<pthread_t> threadIds(maxGpus, 0);
  std::vector<float> runtimes(maxGpus, 0);
  switch(mode){
    case RAY_PROPAGATION_GPU:
      for(unsigned gpu_i = 0; gpu_i < maxGpus; ++gpu_i){
        const unsigned samplesPerNode = maxSampleRange-minSampleRange+1;
        const float samplePerGpu = samplesPerNode / (float) maxGpus;
        unsigned minSample_i = gpu_i * samplePerGpu;
        unsigned maxSample_i = min((float)samplesPerNode, (gpu_i + 1) * samplePerGpu);

        minSample_i += minSampleRange;
        maxSample_i += minSampleRange; 

        threadIds[gpu_i] = calcPhiAseThreaded( raysPerSample,
            maxRaysPerSample,
            maxRepetitions,
            dMesh.at(gpu_i),
            hMesh,
            sigmaA,
            sigmaE,
            mseThreshold,
            useReflections,
            phiAse, 
            mse, 
            totalRays,
            devices.at(gpu_i),
            minSample_i,
            maxSample_i,
            runtimes.at(gpu_i)
            );
      }
      joinAll(threadIds);
      usedGpus = maxGpus;
      for(std::vector<float>::iterator it = runtimes.begin(); it != runtimes.end(); ++it){
        runtime = max(*it, runtime);
      }
      hipDeviceReset();      
      runmode="Ray Propagation GPU";
      break;

    case RAY_PROPAGATION_MPI:
      usedGpus = calcPhiAseMPI( raysPerSample,
          maxRaysPerSample,
          maxRepetitions,
          dMesh.at(0),
          hMesh,
          sigmaA,
          sigmaE,
          mseThreshold,
          useReflections,
          phiAse,
          mse,
          totalRays,
          devices.at(0),
          maxSampleRange
          );
      runmode = "RAY PROPAGATION MPI";
      break;

    case FOR_LOOPS: //Possibly deprecated!
      // TODO: make available for MPI?
      runtime = forLoopsClad( &dndtAse,
          raysPerSample,
          &hMesh,
          hMesh.betaCells,
          hMesh.nTot,
          sigmaA.at(0),
          sigmaE.at(0),
          hMesh.numberOfPoints,
          hMesh.numberOfTriangles,
          hMesh.numberOfLevels,
          hMesh.thickness,
          hMesh.crystalFluorescence);
      runmode = "For Loops";
      break;

    case TEST:
      testEnvironment(raysPerSample,
          maxRaysPerSample,
          dMesh.at(0),
          hMesh,
          sigmaA,
          sigmaE,
          mseThreshold.at(0),
          useReflections,
          dndtAse,
          phiAse,
          mse
          );
      hipDeviceReset();
      runmode="Test Environment";
      break;
    default:
      exit(0);
  }


  if(verbosity & V_DEBUG){
    // Print Solutions
    for(unsigned wave_i = 0; wave_i < sigmaE.size(); ++wave_i){
      dout(V_DEBUG) << "\n\nSolutions " <<  wave_i << std::endl;
      for(unsigned sample_i = 0; sample_i < hMesh.numberOfSamples; ++sample_i){
        int sampleOffset = sample_i + hMesh.numberOfSamples * wave_i;
        dndtAse.at(sampleOffset) = calcDndtAse(hMesh, sigmaA.at(wave_i), sigmaE.at(wave_i), phiAse.at(sampleOffset), sample_i);
        if(sample_i <=10)
          dout(V_DEBUG) << "Dndt ASE[" << sample_i << "]: " << dndtAse.at(sampleOffset) << " " << mse.at(sampleOffset) << std::endl;
      }
      for(unsigned sample_i = 0; sample_i < hMesh.numberOfSamples; ++sample_i){
        int sampleOffset = sample_i + hMesh.numberOfSamples * wave_i;
        dout(V_DEBUG) << "PHI ASE[" << sample_i << "]: " << phiAse.at(sampleOffset) << " " << mse.at(sampleOffset) <<std::endl;
        if(sample_i >= 10) break;
      }
    }
  }

  // Compare with vtk
  // if(compareLocation!="") {
  //   std::vector<double> compareAse = compareVtk(dndtAse, compareLocation, hMesh.numberOfSamples);

  // }


  // Write experiment data
  // output folder has to be the same as TMP_FOLDER in the calling MatLab script
  writeMatlabOutput(
      outputPath,
      phiAse,
      totalRays,
      mse,
      sigmaE.size(),
      hMesh.numberOfSamples,
      hMesh.numberOfLevels
      );


  // FOR OUTPUT
  if(writeVtk){
    std::vector<double> tmpPhiAse(phiAse.begin(), phiAse.end());
    std::vector<double> tmpTotalRays(totalRays.begin(), totalRays.end());

    writeToVtk(hMesh, dndtAse, outputPath + "vtk/dndt", raysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, runtime);
    writeToVtk(hMesh, tmpPhiAse, outputPath + "vtk/phiase", raysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, runtime);
    writeToVtk(hMesh, mse, outputPath + "vtk/mse", raysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, runtime);
    writeToVtk(hMesh, tmpTotalRays, outputPath + "vtk/total_rays", raysPerSample, maxRaysPerSample, mseThreshold.at(0), useReflections, runtime);
  }

  if(verbosity & V_STAT){
    // Filter maxMSE
    for(std::vector<double>::iterator it = mse.begin(); it != mse.end(); ++it){
      maxMSE = max(maxMSE, *it);
      avgMSE += *it;
      if(*it > mseThreshold.at(0))
        highMSE++;
    }
    avgMSE /= mse.size();

    //Print statistics
    std::cout.imbue(std::locale(""));
    dout(V_STAT | V_NOLABEL) << std::endl;
    dout(V_STAT) << "=== Statistics ===" << std::endl;
    dout(V_STAT) << "Runmode           : " << runmode.c_str() << std::endl;
    dout(V_STAT) << "Prisms            : " << (int) hMesh.numberOfPrisms << std::endl;
    dout(V_STAT) << "Samples           : " << (int) dndtAse.size() << std::endl;
    dout(V_STAT) << "Wavelength        : " << (int) sigmaE.size() << std::endl;
    dout(V_STAT) << "RaysPerSample     : " << raysPerSample;
    if(maxRaysPerSample > raysPerSample) { dout(V_STAT | V_NOLABEL) << " - " << maxRaysPerSample << " (adaptive)"; }
    dout(V_STAT | V_NOLABEL) << std::endl;
    dout(V_STAT) << "sum(totalRays)    : " << std::accumulate(totalRays.begin(), totalRays.end(), 0.) << std::endl;
    dout(V_STAT) << "MSE threshold     : " << *(std::max_element(mseThreshold.begin(),mseThreshold.end())) << std::endl;
    dout(V_STAT) << "max. MSE          : " << maxMSE << std::endl;
    dout(V_STAT) << "avg. MSE          : " << avgMSE << std::endl;
    dout(V_STAT) << "too high MSE      : " << highMSE << std::endl;
    dout(V_STAT) << "Nr of GPUs        : " << usedGpus << std::endl;
    dout(V_STAT) << "Runtime           : " << difftime(time(0),starttime) << "s" << std::endl;
    dout(V_STAT) << std::endl;
    if(maxRaysPerSample > raysPerSample){
      dout(V_STAT) << "=== Sampling resolution as Histogram ===" << std::endl;
      ray_histogram(totalRays,raysPerSample,maxRaysPerSample,highMSE);
    }
    dout(V_STAT) << std::endl;

  }
  return 0;

}
