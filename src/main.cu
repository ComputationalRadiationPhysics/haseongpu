// Libraries
#include <stdio.h> /* fprintf, memcpy, strstr, strcmp */
#include <assert.h> /* assert */
#include <string> /* string */
#include <vector> /* vector */

// User header files
#include <calc_dndt_ase.h>
#include <parser.h>
#include <write_to_vtk.h>
#include <write_dndt_ase.h>
#include <for_loops_clad.h>
#include <cudachecks.h>
#include <mesh.h>

#define MIN_COMPUTE_CAPABILITY_MAJOR 2
#define MIN_COMPUTE_CAPABILITY_MINOR 0

/** 
 * @brief Queries for devices on the running mashine and collects
 *        them on the devices array. Set the first device in this 
 *        array as computaion-device. On Errors the programm will
 *        be stoped by exit(). Otherwise you can set the device by command
 *        line parameter --device=
 * 
 * @param verbose > 0 prints debug output
 *        devices Array of possible devices to use
 *        device  number of device you want to set
 * 
 * @return Number of devices in devices array
 */
unsigned getCorrectDevice(int verbose,unsigned **devices, int device){
  int count = 0, candidate = 0;
  unsigned correctDevices = 0;
  hipDeviceProp_t prop;
  int minMajor = MIN_COMPUTE_CAPABILITY_MAJOR;
  int minMinor = MIN_COMPUTE_CAPABILITY_MINOR;

  CUDA_CHECK_RETURN( hipGetDeviceCount(&count));
  
  for(int i=0; i<count; ++i){
	  CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, i) );
	  if( (prop.major > minMajor) || (prop.major == minMajor && prop.minor >= minMinor) ){
		  correctDevices++;
	  }
  }

  if(correctDevices == 0){
    fprintf(stderr,"\nNone of the CUDA-capable devices is sufficient!\n");
    exit(1);
  }

  (*devices) = (unsigned*) malloc(sizeof(unsigned) * correctDevices);

  if(verbose > 0){
	  fprintf(stderr,"\nFound %d CUDA devices with Compute Capability >= %d.%d):\n", correctDevices, minMajor,minMinor); 
  }

  candidate = 0;
  for(int i=0; i<count; ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, i) );
    if( (prop.major > minMajor) || (prop.major == minMajor && prop.minor >= minMinor) ){
		if(verbose > 0){
			fprintf(stderr,"[%d] %s (Compute Capability %d.%d)\n", candidate, prop.name, prop.major, prop.minor); 
		}
		(*devices)[candidate]=i;
		candidate++;
    }
  }

  if(device == -1){
    CUDA_CHECK_RETURN( hipSetDevice((*devices)[0]) );
  }
  else{
    CUDA_CHECK_RETURN( hipSetDevice((*devices)[device]));
  }
  return correctDevices;
}

int main(int argc, char **argv){
  unsigned raysPerSample = 0;
  char runmode[100];
  char experimentLocation[256] = "";
  char compareLocation[256] = "";
  float runtime = 0.0;
  unsigned blocks;
  unsigned threads;
  bool silent = false;
  unsigned *devices; // will be assigned in getCOrrectDevice();
  unsigned numberOfDevices=0;
  int device = -1;
  
  // Constant data
  float nTot = 0;
  float crystalFluorescence = 0;
  std::vector<double> * betaCells = new std::vector<double>;
  std::vector<double> *sigmaA = new std::vector<double>;
  std::vector<double> *sigmaE = new std::vector<double>;

  // Parse Commandline
  if(argc <= 1){
    fprintf(stderr, "C No commandline arguments found\n");
    fprintf(stderr, "C Usage    : ./octrace --mode=[runmode] --rays=[number of rays] --experiment=[location to experiment-data]\n");
    fprintf(stderr, "C Runmodes : for_loops\n");
    fprintf(stderr, "             ray_propagation_gpu\n");
    return 1;
  }
  
  // Parse number of rays
  for(int i=1; i < argc; ++i){
    if(strncmp(argv[i], "--rays=", 6) == 0){
      const char* pos = strrchr(argv[i],'=');
      raysPerSample = atoi(pos+1);
      if(raysPerSample == 0){
	fprintf(stderr, "C Please specify the number of rays per sample Point with --rays=\n");
      }
    }
  }

  // Parse location of experiements
  for(int i=1; i < argc; ++i){
    if(strncmp(argv[i], "--experiment=", 12) == 0){
      memcpy (experimentLocation, argv[i]+13, strlen(argv[i])-13 );
    } 
  }

  // Parse which cuda device to choose
  for(int i=1; i < argc; ++i){
    if(strncmp(argv[i], "--device=", 8) == 0){
      const char* pos = strrchr(argv[i],'=');
      device = atoi(pos+1);
    } 
  }

  // Parse what vtk file to compare with
  for(int i=1; i < argc; ++i){
    if(strncmp(argv[i], "--compare=", 9) == 0){
      memcpy (compareLocation, argv[i]+10, strlen(argv[i])-10 );
    } 
  }


  // Check if we want no output
  for(int i=1; i < argc; ++i){
    if(strncmp(argv[i], "--silent", 7) == 0){
		silent=true;
    } 
  }
  
  std::string root(experimentLocation);

  // Add slash at the end, if missing
  if(root[root.size()-1] == 'w')
    root.erase(root.size()-1, 1);
  else if(root[root.size()-1] != '/')
    root.append("/");

  // Parse constant from files
  if(fileToValue(root + "n_tot.txt", nTot)) return 1;
  if(fileToValue(root + "tfluo.txt", crystalFluorescence)) return 1;
  if(fileToVector(root + "beta_cell.txt", betaCells)) return 1;
  if(fileToVector(root + "sigma_a.txt", sigmaA)) return 1;
  if(fileToVector(root + "sigma_e.txt", sigmaE)) return 1;
  assert(sigmaA->size() == sigmaE->size());
  
  // Set/Test device to run experiment
  numberOfDevices = getCorrectDevice(1,&devices, device);

  // Parse experiemntdata and fill mesh 
  Mesh hMesh;
  Mesh *dMesh = new Mesh[numberOfDevices];
  if(Mesh::parseMultiGPU(&hMesh, &dMesh, root, numberOfDevices, devices)) return 1;

  // Debug
  // fprintf(stderr, "C nTot: %e\n", nTot);
  // fprintf(stderr, "C sigmaA: %e\n", sigmaA);
  // fprintf(stderr, "C sigmaE: %e\n", sigmaE);
  // fprintf(stderr, "C numberOfTriangles: %d\n", hMesh.numberOfTriangles);
  // fprintf(stderr, "C numberOfLevels: %d\n", hMesh.numberOfLevels); 
  // fprintf(stderr, "C numberOfPrisms: %d\n", hMesh.numberOfPrisms);
  // fprintf(stderr, "C numberOfPoints: %d\n", hMesh.numberOfPoints); 
  // fprintf(stderr, "C numberOfSamples: %d\n\n", hMesh.numberOfSamples);

  // Solution vector
  std::vector<double> *ase = new std::vector<double>(hMesh.numberOfSamples * sigmaE->size(), 0);

  // Run Experiment
  for(int i=1; i < argc; ++i){
    if(strncmp(argv[i], "--mode=", 6) == 0){
      if(strstr(argv[i], "ray_propagation_gpu") != 0 ){
	// threads and blocks will be set in the following function (by reference)
	CUDA_CHECK_RETURN(hipSetDevice(devices[0]));
	runtime = calcDndtAse(threads, 
			      blocks, 
			      raysPerSample,
			      dMesh[0],
			      hMesh,
			      betaCells,
			      nTot,
			      sigmaA,
			      sigmaE,
			      crystalFluorescence,
			      ase
			      );
	strcpy(runmode, "Ray Propagation New GPU");
	break;
      }
      else if(strstr(argv[i], "for_loops") != 0){
	// threads and blocks will be set in the following function (by reference)
	runtime = forLoopsClad(
			ase,
			raysPerSample,
			&hMesh,
			betaCells,
			nTot,
			sigmaA->at(0),
			sigmaE->at(0),
			hMesh.numberOfPoints,
			hMesh.numberOfTriangles,
			hMesh.numberOfLevels,
			hMesh.thickness,
			crystalFluorescence);
	strcpy(runmode, "For Loops");
	break;
      }
      else{
	fprintf(stderr, "C Please specify the runmode with --mode=\n");
	return 1;
      }
    
    }
    else{
      fprintf(stderr, "C Please specify the runmode with --mode=\n");
      return 1;
    }


  }

  // Print Solution
  for(unsigned wave_i = 0; wave_i < sigmaE->size(); ++wave_i){
    fprintf(stderr, "\n\nC Solutions %d\n", wave_i);
    for(unsigned sample_i = 0; sample_i < ase->size(); ++sample_i){
      fprintf(stderr, "C ASE PHI of sample %d: %.80f\n", sample_i, ase->at(sample_i + hMesh.numberOfSamples * wave_i));
      if(silent){
	if(sample_i >= 10) break;
      }
    }
  }

  // Print statistics
  fprintf(stderr, "\n");
  fprintf(stderr, "C Statistics\n");
  fprintf(stderr, "C Prism             : %d\n", (int) hMesh.numberOfPrisms);
  fprintf(stderr, "C Samples           : %d\n", (int) ase->size());
  fprintf(stderr, "C Rays/Sample       : %d\n", raysPerSample);
  fprintf(stderr, "C Rays Total        : %zu\n", raysPerSample * ase->size());
  fprintf(stderr, "C GPU Blocks        : %d\n", blocks);
  fprintf(stderr, "C GPU Threads/Block : %d\n", threads);
  fprintf(stderr, "C GPU Threads Total : %d\n", threads * blocks);
  fprintf(stderr, "C Runmode           : %s \n", runmode);
  fprintf(stderr, "C Runtime           : %f s\n", runtime);
  fprintf(stderr, "\n");

  // Write experiment data
  writeToVtk(&hMesh, ase, "octrace.vtk");
  compareVtk(ase, compareLocation, hMesh.numberOfSamples);
  writeToVtk(&hMesh, ase, "octrace_compare.vtk");
  writeDndtAse(ase);

  // Free memory
  delete betaCells;
  delete sigmaE;
  delete sigmaA;

  return 0;
}


