#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_vector_types.h"
#include "assert.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"

#define SMALL 1E-06
#define CUDA_CHECK_RETURN(value) {				\
	hipError_t _mCudaStat = value;				\
	if (_mCudaStat != hipSuccess) {				\
		fprintf(stderr, "Error %s at line %d in file %s\n",	\
				hipGetErrorString(_mCudaStat), __LINE__, __FILE__);	\
		exit(1);							\
	}								\
}

//----------------------------------------------------
// Structures
//----------------------------------------------------
typedef struct point {
	float x;
	float y;
	float z;
} POINT;

typedef struct vector {
	float x;
	float y;
	float z;
} VECTOR;

typedef struct ray {
	point start;
	vector direction;
} RAY;

typedef struct triangle {
	point a;
	point b;
	point c;
} TRIANGLE;

typedef struct plane {
	point start;
	vector normal;

} PLANE;

//------------------------------------------
typedef float4 pointCu;
typedef float4 vectorCu;

typedef struct triangleCu{
	pointCu A;
	pointCu B;
	pointCu C;
} TRIANGLE_CU;

typedef struct prismCu{
	triangleCu t1;
	float height; //OPTIMIZE: The height could be stored as 4th parameter of one of the Triangle-coordinates?
} PRISM_CU;

typedef struct planeCu {
	pointCu P;
	vectorCu normal;
} PLANE_CU;

// Describes one vertex of the input-Mesh
typedef struct vertexCu {
	pointCu P;		// the Position
	float4 G;		// The ASE-Gain in this Point (values from the rays are added)

	// OPTIMIZE: distribute Writes of G over more than 1 position in this
	// variable (e.g. through modulo thread-ID)
	// -> could result in less concurrent write-operations
	// Alternatively, save G in 4th coordinate of P
} VERTEX_CU;

typedef struct rayCu {
	pointCu P;			// the random starting point
	vectorCu direction;  // the position of the vertexCu, where the ray is going to
	float phiAse;		// the accumulated ASE-Flux for this ray
	// OPTIMIZE: ASE-Flux might be stored as 4th parameter of P or direction
} RAY_CU;

//----------------------------------------------------
// Auxillary function declaration
//----------------------------------------------------

float distance(point a, point b);
void  printPoint(point p);

// New functions
bool  collide(triangleCu t, pointCu p);
bool  collide(triangleCu t, rayCu r);
bool  collide(prismCu pr, rayCu r);
float4 toBarycentric(triangleCu t, pointCu p);
pointCu intersection(planeCu p, rayCu r);
std::vector<triangleCu> generateTriangles(int height, int width, float level);
std::vector<prismCu> generatePrisms(int height, int width, float level);
std::vector<rayCu> generateRays(int height, int width, int level, unsigned maxRays);
rayCu   generateRay(int height, int weight, int level);

//----------------------------------------------------
// Device Code
//----------------------------------------------------

/**
  @brief Calculates A-B for 2 float4-based inputs
 **/
__device__ pointCu subtractPoints(pointCu A, pointCu B){
	pointCu C;
	C.x = A.x - B.x;
	C.y = A.y - B.y;
	C.z = A.z - B.z;
	C.w = A.w - B.w;
	return C;
}

__device__ rayCu generateRayGpu(pointCu vertexPoint, prismCu startPrism, hiprandState randomstate){
	float u = hiprand_uniform(&randomstate);
	float v = hiprand_uniform(&randomstate);
	if((u+v) > 1){ //OPTIMIZE: remove if
		u = 1-u;
		v = 1-v;
	}
	const float w = 1-(u+v);

	pointCu A = startPrism.t1.A;
	pointCu B = startPrism.t1.B;
	pointCu C = startPrism.t1.C;

	// Get x and y coordinates from the random barycentric values
	const float xRand = u*A.x + v*B.x + w*C.x ;
	const float yRand = u*A.y + v*B.y + w*C.y ;

	// Take one of the given z-coordinates and add a random part of the prism height
	const float zRand = A.z + hiprand_uniform(&randomstate) * startPrism.height;

	float ase=0.f;

	// Take the values to assemble a ray
	rayCu r = {
		{xRand, yRand, zRand, 1},
		vertexPoint,
		ase};
	return r;
}

__device__ prismCu selectPrism(int id, prismCu prisms[]){
	//TODO
	return prisms[0];
}

__device__ float propagate(rayCu ray, prismCu prisms[], prismCu startprism){
	float gain = 1.f;
	float vecX = ray.direction.x - ray.P.x;
	float vecY = ray.direction.y - ray.P.y;
	float vecZ = ray.direction.z - ray.P.z;

	const float distanceTotal = sqrt(vecX*vecX+vecY*vecY+vecZ*vecZ);
	float distance = distanceTotal;
	float length = distanceTotal;
	vecX /= distanceTotal;
	vecY /= distanceTotal;
	vecZ /= distanceTotal;

	prismCu current = startprism;


	for(;;){
		length = distance;
		//generate the triangle surfaces of the prism
		const triangleCu t1 = current.t1;
		const triangleCu t2 = { 
			{t1.A.x, t1.A.y, t1.A.z + t1.A.w, 1},
			{t1.B.x, t1.B.y, t1.B.z + t1.B.w, 1},
			{t1.C.x, t1.C.y, t1.C.z + t1.C.w, 1}
		};

		// OPTIMIZE: make use of the rectangles!
		const triangleCu surfaces[8] = {
			t1,
			t2,
			{t1.A, t1.B, t2.A},
			{t1.B, t2.B, t2.A},
			{t1.B, t1.C, t2.C},
			{t1.B, t2.B, t2.C},
			{t1.A, t1.C, t2.C},
			{t1.A, t2.A, t2.C}
		};

		int i=0;
		float lengthHelp = 0.f;
		for(i=0; i<8 ; ++i){ //OPTIMIZE: unroll, so that every surface can be optimized differently
			// get the generating vectors for the plane
			vectorCu AB = subtractPoints(surfaces[i].B, surfaces[i].A);
			vectorCu AC = subtractPoints(surfaces[i].C, surfaces[i].A);

			planeCu pl;
			pl.P = surfaces[i].A;
			// cross product of the vectors
			pl.normal.x = AB.y*AC.z - AB.z*AC.y;
			pl.normal.y = AB.z*AC.x - AB.x*AC.z;
			pl.normal.z = AB.x*AC.y - AB.y*AC.x;

			// direction * pl.normal
			float denominator = (ray.direction.x * pl.normal.x) + (ray.direction.y * pl.normal.y) + (ray.direction.z * pl.normal.z);
			float d = 0.f;
			float nominator = 0.f;
			if(denominator != 0.f) //OPTIMIZE: check if we have a lot of branch diversion, or if all threads behave the same
			{
				// A * pl.normal
				d = (surfaces[i].A.x * pl.normal.x) + (surfaces[i].A.y * pl.normal.y) + (surfaces[i].A.z * pl.normal.z);
				// d - (P * pl.normal)
				nominator = d - ((ray.P.x * pl.normal.x) + (ray.P.y * pl.normal.y) + (ray.P.z * pl.normal.y)); 
				lengthHelp = nominator/denominator;
				if(lengthHelp < length && lengthHelp > 0.f) //OPTIMIZE: most threads should do the same?
				{
					length = lengthHelp;
				}
			}
		}


		//with the new length, get the gain and add it
		// TODO
		gain *= exp(length);

		// calculate values for next iteration
		distance -= length;
		if(abs(distance) < SMALL)
		{
			break;
		}

		ray.P.x += length*vecX;
		ray.P.y += length*vecY;
		ray.P.z += length*vecZ;

		//TODO:
		// calculate the next PRISM (maybe with help of some neighbor-datastructure?

	}


	return gain;
}

__global__ void setupKernel ( hiprandState * state, unsigned long seed ){
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init ( seed, id, 0, &state[id] );
	// OPTIMIZE: Use MersenneTwister or even a better PRNG
} 

// does the raytracing for a single ray (randomly generated) and a single (given) Vertex
__global__ void raytraceStep( hiprandState* globalState, vertexCu vertex, prismCu prisms[]) {
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	hiprandState localState = globalState[id];

	//OPTIMIZE: the Octree should/could produce a subset of the prism-array!


	// this should give the same prism multiple times (so that every thread uses the same prism, which yields
	// big benefits for the memory access (and caching!)
	const prismCu startprism = selectPrism(id, prisms);	

	rayCu ray = generateRayGpu(vertex.P,startprism, localState); //TODO:verify

	float gain = propagate(ray,prisms,startprism);

	//atomicAdd(&(vertex.G.x),gain);

	globalState[id] = localState;
}


//----------------------------------------------------
// Host Code
//----------------------------------------------------
int main(){

	//Variable definitions
	const unsigned maxRays = 1000000;
	const unsigned maxTriangles = 10000;
	const unsigned maxVertices = 5;
	const unsigned length = ceil(sqrt(maxTriangles / 2));
	const unsigned depth  = 10;
	const unsigned maxPrisms = length * length * depth * 2;
	unsigned ray_i, prism_i, vertex_i;
	float runtimeGpu = 0.0;
	float runtimeCpu = 0.0;
	hipEvent_t start, stop;
	bool useCpu = false;
	bool useGpu = true;
	hiprandState* devStates;

	// Generate testdata
	std::vector<vertexCu> vertices;
	std::vector<prismCu> prisms = generatePrisms(length, length, depth);
	std::vector<rayCu> rays = generateRays(length, length, depth, maxRays);
	std::vector<float> collisions(maxPrisms, 0);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// CPU Raytracing
	{
		hipEventRecord(start, 0);
		if(useCpu){
			for(ray_i = 0; ray_i < rays.size(); ++ray_i){
				for(prism_i = 0; prism_i < prisms.size(); ++prism_i){
					if(collide(prisms[prism_i], rays[ray_i])){
						fprintf(stdout, "CPU: Ray %d hits on prism %d\n", ray_i, prism_i);
						collisions[prism_i]++;
					}

				}
			}

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&runtimeCpu, start, stop);
		}
	}

	// GPU Raytracing
	rayCu* hRays, *dRays;
	prismCu* hPrisms, *dPrisms;
	float4* hCollisions, *dCollisions;
	int threads = 256;
	int blocks = ceil(maxPrisms / threads);
	if(useGpu){

		//initialize memory
		{
			// Memory allocation on host
			CUDA_CHECK_RETURN(hipHostAlloc( (void**)&hPrisms, maxPrisms * sizeof(prismCu), hipHostMallocDefault));
			CUDA_CHECK_RETURN(hipHostAlloc( (void**)&hRays, maxRays * sizeof(rayCu), hipHostMallocDefault));
			CUDA_CHECK_RETURN(hipHostAlloc( (void**)&hCollisions, maxPrisms * sizeof(float4), hipHostMallocDefault));

			// Memory initialisation on host
			for(ray_i = 0; ray_i < maxRays; ++ray_i){
				hRays[ray_i] = rays[ray_i];
			}
			for(prism_i = 0; prism_i < maxPrisms; ++prism_i){
				hPrisms[prism_i] = prisms[prism_i];
			}


			// Memory allocation on device
			CUDA_CHECK_RETURN(hipMalloc(&dRays, maxRays * sizeof(rayCu)));
			CUDA_CHECK_RETURN(hipMalloc(&dPrisms, maxPrisms * sizeof(prismCu)));
			CUDA_CHECK_RETURN(hipMalloc(&dCollisions, maxPrisms * sizeof(float4)));

			// Copy data from host to device
			hipEventRecord(start, 0);
			CUDA_CHECK_RETURN(hipMemcpy(dRays, hRays, maxRays * sizeof(rayCu), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(dPrisms, hPrisms, maxPrisms * sizeof(prismCu), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(dCollisions, hCollisions, maxPrisms * sizeof(float4), hipMemcpyHostToDevice));

		}


		// Generating Random Numbers
		CUDA_CHECK_RETURN(hipMalloc(&devStates, threads*blocks*sizeof( hiprandState )));
		setupKernel<<< threads, blocks >>> ( devStates, time(NULL) );

		// start the Kernels
		for(vertex_i = 0; vertex_i < maxVertices; ++vertex_i){
			raytraceStep<<< threads, blocks >>> ( devStates , vertices[vertex_i] , dPrisms);
		}

		// Copy data from device to host
		CUDA_CHECK_RETURN(hipMemcpy(hCollisions, dCollisions, maxPrisms * sizeof(float4), hipMemcpyDeviceToHost));

		// Free memory on device
		hipFree(devStates);

		// Evaluate device data
		{
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&runtimeGpu, start, stop);
		
		
		for(prism_i = 0; prism_i < maxPrisms; ++prism_i){
			if(hCollisions[prism_i].x > 0)
				fprintf(stderr, "GPU: (%f, %f, %f, %f) collission on prism %d\n", hCollisions[prism_i].x, hCollisions[prism_i].y, hCollisions[prism_i].z, hCollisions[prism_i].w, prism_i);

		}
		for(prism_i = 0; prism_i < maxPrisms; ++prism_i){
			if((hCollisions[prism_i].x != collisions[prism_i]) && useCpu && useGpu){
				fprintf(stderr, "\033[31;1m[Error]\033[m CPU(%.0f) != GPU(%.0f) on prism %d\n",collisions[prism_i], hCollisions[prism_i].x, prism_i);
			}
		}
		}
	}

	// print statistics
	{
	fprintf(stderr, "\n");
	fprintf(stderr, "Prism       : %d\n", maxPrisms);
	fprintf(stderr, "Triangles   : %d\n", maxPrisms * 8);
	fprintf(stderr, "Rays        : %d\n", maxRays);
	fprintf(stderr, "GPU Blocks  : %d\n", blocks);
	fprintf(stderr, "GPU Threads : %d\n", threads);
	fprintf(stderr, "Runtime_GPU : %f s\n", runtimeGpu / 1000.0);
	fprintf(stderr, "Runtime_CPU : %f s\n", runtimeCpu / 1000.0);
	fprintf(stderr, "\n");
	}
	// Cleanup
	hipHostFree(hRays);
	hipHostFree(hPrisms);
	hipHostFree(hCollisions);


	return 0;
}

//----------------------------------------------------
// Auxillary function definition
//----------------------------------------------------

float4 toBarycentric(triangleCu t, pointCu p){
	float x1,x2,x3, y1,y2,y3, x,y;
	float4 b;

	x1 = t.A.x;
	x2 = t.B.x;
	x3 = t.C.x;

	y1 = t.A.y;
	y2 = t.B.y;
	y3 = t.C.y;

	x = p.x;
	y = p.y;

	b.x = ((y2-y3)*(x-x3)+(x3-x2)*(y-y3)) / ((y2-y3)*(x1-x3)+(x3-x2)*(y1-y3));
	b.y = ((y3-y1)*(x-x3)+(x1-x3)*(y-y3)) / ((y2-y3)*(x1-x3)+(x3-x2)*(y1-y3));
	b.z = 1 - b.x - b.y;
	b.w = 0;

	// In case of division by 0 --> nan
	if((fabs((b.x + b.y + b.z) - 1)) != (fabs((b.x + b.y + b.z) - 1)))
		b.z = 2;
	return b;
}

/**
  @brief Detects collisions of triangle and point with
  precondition, that the point is on the same 
  plane as the point.
 **/
bool collide(triangleCu t, pointCu p){
	float4 b = toBarycentric(t, p);
	return (b.x > 0) && (b.x < 1) && (b.y > 0) && (b.y < 1) && (b.z > 0) && (b.z < 1) && (b.z == b.z);
}


/**
  @brief Detects collisions of a triangle and a ray without
  a precondition.
 **/
bool collide(triangleCu t, rayCu r){
	planeCu pl;
	float b1, b2, b3, c1, c2, c3;

	b1 = t.B.x;
	b2 = t.B.y;
	b3 = t.B.z;

	c1 = t.C.x;
	c2 = t.C.y;
	c3 = t.C.z;

	pl.P = t.A;
	pl.normal.x = (b2*c3 - b3*c2);
	pl.normal.y = (b3*c1 - b1*c3);
	pl.normal.z = (b1*c2 - b2*c1);

	return collide(t, intersection(pl, r));
}

bool collide(prismCu pr, rayCu r){
	bool hasCollide;
	pointCu A1 = pr.t1.A;
	pointCu B1 = pr.t1.B;
	pointCu C1 = pr.t1.C;
	pointCu A2 = {pr.t1.A.x, pr.t1.A.y, pr.t1.A.w, 1};
	pointCu B2 = {pr.t1.B.x, pr.t1.B.y, pr.t1.B.w, 1};
	pointCu C2 = {pr.t1.C.x, pr.t1.C.y, pr.t1.C.w, 1};

	triangleCu triangles[8] = {
		pr.t1,
		{A2, B2, C2},
		{A1, B1, A2},
		{B1, B2, A2},
		{B1, C1, C2},
		{B1, B2, C2},
		{A1, C1, C2},
		{A1, A2, C2}};

	hasCollide = 
		collide(triangles[0], r)
		|| collide(triangles[1], r)
		|| collide(triangles[2], r) 
		|| collide(triangles[3], r)
		|| collide(triangles[4], r) 
		|| collide(triangles[5], r) 
		|| collide(triangles[6], r) 
		|| collide(triangles[7], r);

	return hasCollide;
}

/**
  @brief Intersection calculates the intersection between a plane p
  and a ray r. There is no detection for rays in the plane
  or for parallel plane. 

  It uses the normal of the plane to derive the coordinate form 
  of the plane. With the help of a coordinate form it is very
  easy to get the intersection point between a ray and a plane.

  ray   g: y~ = x~ + t*p~
  plane E: y~ = a~ + r*b~ + s*c~
  d  = n1*(x1+t*p1) + n2*(x2+t*p2) + n3*(x3+t*p3)
  d  = n~ * a~
 **/
pointCu intersection(planeCu pl, rayCu r){
	pointCu intersectionPoint = {0.0,0.0,0.0};

	float t, d;

	// vector coordinates
	float n1, n2, n3, x1, x2, x3, p1, p2, p3, a1, a2, a3;

	// just get the coordinates from the structs
	n1 = pl.normal.x;
	n2 = pl.normal.y;
	n3 = pl.normal.z;

	a1 = pl.P.x;
	a2 = pl.P.y;
	a3 = pl.P.z;

	x1 = r.P.x;
	x2 = r.P.y;
	x3 = r.P.z;

	p1 = r.direction.x;
	p2 = r.direction.y;
	p3 = r.direction.z;

	// calculation of intersection
	d = n1*a1 + n2*a2 + n3*a3;
	t = (d - n1*x1 - n2*x2 - n3*x3) / (n1*p1 + n2*p2 + n3*p3);

	intersectionPoint.x = x1 + t * p1;
	intersectionPoint.y = x2 + t * p2;
	intersectionPoint.z = x3 + t * p3;

	return intersectionPoint;

}

float distance(point a, point b){
	float d = sqrt(pow((b.x - a.x), 2) + pow((b.y - a.y),2) + pow((b.z - a.z),2));
	return fabs(d);
}

std::vector<triangleCu> generateTriangles(int height, int weight, float level){
	int h,w;
	std::vector<triangleCu> triangles;
	for(h = 0; h < height; ++h){
		for(w = 0; w < weight; ++w){
			triangleCu t1 = {
				{float(h), float(w), level, 1},
				{float(h), float(w+1), level, 1},
				{float(h+1), float(w), level, 1}};
			triangleCu t2 = {
				{float(h), float(w+1), level, 1},
				{float(h+1), float(w+1), level, 1},
				{float(h+1), float(w), level, 1}};
			triangles.push_back(t1);
			triangles.push_back(t2);

		}

	}

	return triangles;
}

std::vector<prismCu> generatePrisms(int height, int weight, float level){
	int h,w,l;
	std::vector<prismCu> prisms;
	for(l = 0; l < level; ++l){
		for(h = 0; h < height; ++h){
			for(w = 0; w < weight; ++w){
				triangleCu a1 = {
					{float(h), float(w), l, l+1},
					{float(h), float(w+1), l, l+1},
					{float(h+1), float(w), l, l+1}};
				triangleCu b1 = {
					{float(h), float(w+1), l, 1+1},
					{float(h+1), float(w+1), l, 1+1},
					{float(h+1), float(w), l, 1+1}};

				prismCu pr1 = {a1};
				prismCu pr2 = {b1};

				prisms.push_back(pr1);
				prisms.push_back(pr2);

			}

		}

	}

	return prisms;
}

rayCu generateRay(const int heigth, const int width, const int level){
	float randHeigth = float(rand() % heigth) + (rand() / (float) RAND_MAX);
	float randWidth  = float(rand() % width ) + (rand() / (float) RAND_MAX);
	float rand_level  = float(rand() % level ) + (rand() / (float) RAND_MAX);

	float dirX = (rand() / (float) RAND_MAX);
	float dirY = (rand() / (float) RAND_MAX);
	float dirZ = (rand() / (float) RAND_MAX);

	rayCu r = {
		{randHeigth, randWidth, rand_level, 1},
		{dirX, dirY, dirZ, 0}};
	return r;
}


std::vector<rayCu> generateRays(const int height, const int width, const int level, const unsigned maxRays){
	std::vector<rayCu> rays;
	unsigned ray_i;
	for(ray_i = 0; ray_i < maxRays; ++ray_i){
		rayCu ray = generateRay(height, width, level);
		rays.push_back(ray);
	}
	return rays;
}

void printPoint(point p){
	fprintf(stdout, "Point\n");
	fprintf(stdout, "x: %f\n", p.x);
	fprintf(stdout, "y: %f\n", p.y);
	fprintf(stdout, "z: %f\n", p.z);

}
