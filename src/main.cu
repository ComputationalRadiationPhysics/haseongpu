#include "hip/hip_runtime.h"
// Libraries
#include <stdio.h> /* fprintf, memcpy, strstr, strcmp */
#include <assert.h> /* assert */
#include <string> /* string */
#include <vector> /* vector */
#include <stdlib.h> /* atoi */
#include <pthread.h> /* pthread_t, pthread_join */

// User header files
#include <calc_phi_ase.h>
#include <parser.h>
#include <write_to_vtk.h>
#include <write_matlab_output.h>
#include <for_loops_clad.h>
#include <cudachecks.h>
#include <mesh.h>
#include <test_environment.h>
#include <thread.h>

#define MIN_COMPUTE_CAPABILITY_MAJOR 2
#define MIN_COMPUTE_CAPABILITY_MINOR 0

/** 
 * @brief Queries for devices on the running mashine and collects
 *        them on the devices array. Set the first device in this 
 *        array as computaion-device. On Errors the programm will
 *        be stoped by exit(). Otherwise you can set the device by command
 *        line parameter --device=
 * 
 * @param verbose > 0 prints debug output
 * 
 * @return vector of possible devices
 */
std::vector<unsigned> getCorrectDevice(int verbose){
  hipDeviceProp_t prop;
  int minMajor = MIN_COMPUTE_CAPABILITY_MAJOR;
  int minMinor = MIN_COMPUTE_CAPABILITY_MINOR;
  int count;
  std::vector<unsigned> devices;

  // Get number of devices
  CUDA_CHECK_RETURN( hipGetDeviceCount(&count));

  // Check devices for compute capability and if device is busy
  for(int i=0; i<count; ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, i) );
    if( (prop.major > minMajor) || (prop.major == minMajor && prop.minor >= minMinor) ){
      hipSetDevice(i);
      int* test;
      if(hipMalloc((void**) &test, sizeof(int)) == hipSuccess){
        devices.push_back(i);
        hipFree(test);
        hipDeviceReset();
      }
    }
  }

  if(devices.size() == 0){
    fprintf(stderr,"\nNone of the free CUDA-capable devices is sufficient!\n");
    exit(1);
  }

  hipSetDevice(devices.at(0));

  if(verbose > 0){
    fprintf(stderr,"\nFound %d available CUDA devices with Compute Capability >= %d.%d):\n", int(devices.size()), minMajor,minMinor);
    for(unsigned i=0; i<devices.size(); ++i){
      CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, devices[i]) );
      fprintf(stderr,"[%d] %s (Compute Capability %d.%d)\n", devices[i], prop.name, prop.major, prop.minor);
    }
  }

  return devices;

}

double calcDndtAse(const Mesh& mesh, const double sigmaA, const double sigmaE, const float phiAse, const unsigned sample_i){
  double gain_local = mesh.nTot * mesh.betaCells[sample_i] * (sigmaE + sigmaA) - double(mesh.nTot * sigmaA);
  return gain_local * phiAse / mesh.crystalFluorescence;
}

int main(int argc, char **argv){
  unsigned raysPerSample = 0;
  unsigned maxRaysPerSample = 0;
  float maxExpectation = 0;
  std::string runmode("");
  std::string compareLocation("");
  float runtime = 0.0;
  bool silent = false;
  bool writeVtk = false;
  bool useReflections = false;
  float expectationThreshold = 0;
  std::vector<unsigned> devices; // will be assigned in getCOrrectDevice();
  unsigned maxGpus = 0;
  RunMode mode = NONE;

  std::string experimentPath;

  // Wavelength data
  std::vector<double> sigmaA;
  std::vector<double> sigmaE;

  // Set/Test device to run experiment with
  devices = getCorrectDevice(1);

  // Parse Commandline
  parseCommandLine(argc, argv, &raysPerSample, &maxRaysPerSample, &experimentPath, &silent,
		   &writeVtk, &compareLocation, &mode, &useReflections, &expectationThreshold, &maxGpus);

  // sanity checks
  if(checkParameterValidity(argc, raysPerSample, &maxRaysPerSample, experimentPath, devices.size(), mode, &expectationThreshold, &maxGpus)) return 1;

  // Parse wavelengths from files
  if(fileToVector(experimentPath + "sigma_a.txt", &sigmaA)) return 1;
  if(fileToVector(experimentPath + "sigma_e.txt", &sigmaE)) return 1;
  assert(sigmaA.size() == sigmaE.size());

  // Parse experientdata and fill mesh
  Mesh hMesh;
  std::vector<Mesh> dMesh(maxGpus);
  
  if(Mesh::parseMultiGPU(hMesh, dMesh, experimentPath, devices, maxGpus)) return 1;

  // Solution vector
  std::vector<double> dndtAse(hMesh.numberOfSamples * sigmaE.size(), 0);
  std::vector<float>  phiAse(hMesh.numberOfSamples * sigmaE.size(), 0);
  std::vector<double> expectation(hMesh.numberOfSamples * sigmaE.size(), 1000);
  std::vector<unsigned> totalRays(hMesh.numberOfSamples * sigmaE.size(), 0);

  fprintf(stderr, "reflectionAngle: %f\n",hMesh.getReflectionAngle(-1));
  fprintf(stderr, "reflectionAngle: %f\n",hMesh.getReflectionAngle(1));
  fprintf(stderr, "maxreflections: %d\n",hMesh.getMaxReflections());
  
  // Run Experiment
  std::vector<float> runtimes(maxGpus, 0);
  std::vector<pthread_t> threadIds(maxGpus, 0);
  float samplePerGpu = hMesh.numberOfSamples / (float) maxGpus;
  switch(mode){
    case RAY_PROPAGATION_GPU:
      for(unsigned gpu_i = 0; gpu_i < maxGpus; ++gpu_i){
	unsigned minSample_i = gpu_i * samplePerGpu;
	unsigned maxSample_i = min((float)hMesh.numberOfSamples, (gpu_i + 1) * samplePerGpu);
	threadIds[gpu_i] = calcPhiAseThreaded( raysPerSample,
					       maxRaysPerSample,
					       dMesh.at(devices.at(gpu_i)),
					       hMesh,
					       sigmaA,
					       sigmaE,
					       expectationThreshold,
					       useReflections,
					       phiAse,
					       expectation,
						   totalRays,
					       devices.at(gpu_i),
					       minSample_i,
					       maxSample_i,
					       runtimes.at(gpu_i)
					       );

      }
      joinAll(threadIds);
      for(std::vector<float>::iterator it = runtimes.begin(); it != runtimes.end(); ++it){
	runtime = max(*it, runtime);
      }
      hipDeviceReset();      
      runmode="Ray Propagation New GPU";
      break;

    case FOR_LOOPS:
      runtime = forLoopsClad( &dndtAse,
			      raysPerSample,
			      &hMesh,
			      hMesh.betaCells,
			      hMesh.nTot,
			      sigmaA.at(0),
			      sigmaE.at(0),
			      hMesh.numberOfPoints,
			      hMesh.numberOfTriangles,
			      hMesh.numberOfLevels,
			      hMesh.thickness,
			      hMesh.crystalFluorescence);
      runmode = "For Loops";
      break;

  case TEST:
    testEnvironment(raysPerSample,
		    maxRaysPerSample,
		    dMesh.at(0),
		    hMesh,
		    sigmaA,
		    sigmaE,
		    expectationThreshold,
		    useReflections,
		    dndtAse,
		    phiAse,
		    expectation
		    );
    hipDeviceReset();
    runmode="Test Environment";
    break;
  default:
    exit(0);
  }

  // Filter maxExpectation
  for(std::vector<double>::iterator it = expectation.begin(); it != expectation.end(); ++it){
    maxExpectation = max(maxExpectation, *it);
  }


  // Print Solutions
  for(unsigned wave_i = 0; wave_i < sigmaE.size(); ++wave_i){
    fprintf(stderr, "\n\nC Solutions %d\n", wave_i);
    for(unsigned sample_i = 0; sample_i < dndtAse.size(); ++sample_i){
      int sampleOffset = sample_i + hMesh.numberOfSamples * wave_i;
      dndtAse.at(sampleOffset) = calcDndtAse(hMesh, sigmaA.at(wave_i), sigmaE.at(wave_i), phiAse.at(sampleOffset), sample_i);
      if(silent && sample_i <=10)
	fprintf(stderr, "C Dndt ASE[%d]: %.80f %.10f\n", sample_i, dndtAse.at(sampleOffset), expectation.at(sampleOffset));
    }
  }

  // Compare with vtk
  if(compareLocation!="") {
    std::vector<double> compareAse = compareVtk(dndtAse, compareLocation, hMesh.numberOfSamples);
    //if(writeVtk) writeToVtk(hMesh, compareAse, "octrace_compare", raysPerSample, maxRaysPerSample, expectationThreshold, useReflections, runtime);
  }

  // Print statistics
  fprintf(stderr, "\n");
  fprintf(stderr, "C Statistics\n");
  fprintf(stderr, "C Prism             : %d\n", (int) hMesh.numberOfPrisms);
  fprintf(stderr, "C Samples           : %d\n", (int) dndtAse.size());
  fprintf(stderr, "C MSE threshold     : %f\n", expectationThreshold);
  fprintf(stderr, "C max. MSE          : %f\n", maxExpectation);
  fprintf(stderr, "C Runmode           : %s \n", runmode.c_str());
  fprintf(stderr, "C Runtime           : %f s\n", runtime);
  fprintf(stderr, "\n");

  // Write experiment data
  writeMatlabOutput(
		  phiAse,
		  totalRays,
		  expectation,
		  sigmaE.size(),
		  hMesh.numberOfSamples);

  if(writeVtk) writeToVtk(hMesh, dndtAse, "octrace_dndt", raysPerSample, maxRaysPerSample, expectationThreshold, useReflections, runtime);
  if(writeVtk) writeToVtk(hMesh, expectation, "octrace_expectation", raysPerSample, maxRaysPerSample, expectationThreshold, useReflections, runtime);

  return 0;
}


