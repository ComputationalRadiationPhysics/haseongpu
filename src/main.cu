#include "hip/hip_runtime.h"
// Libraries
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_vector_types.h"
#include "assert.h"
#include "string.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"


// User header files
#include "datatypes.h"
#include "geometry.h"
#include "datatypes.h"
#include "generate_testdata.h"
#include "print.h"
#include "geometry_gpu.h"
#include "ase_bruteforce_kernel.h"
#include "ase_bruteforce_cpu.h"
#include "testdata_transposed.h"
#include "naive_ray_propagation.h"
#include "buildgrid.h"

int main(int argc, char **argv){
  const unsigned rays_per_sample = pow(2,18);
  const int threads = 256;
  char runmode[20];
  float runtime = 0.0;
  
  // Parse Commandline
  if(argc <= 1){
    fprintf(stderr, "C No commandline arguments found\n");
    fprintf(stderr, "C Usage    : ./octrace --mode=[runmode]\n");
    fprintf(stderr, "C Runmodes : bruteforce_gpu\n");
    fprintf(stderr, "             naive_ray_propagation\n");
    return 0;
  }
  
  // Generate testdata
  fprintf(stderr, "C Generate Testdata\n");
  std::vector<PrismCu>  *prisms = generatePrismsFromTestdata(host_mesh_z, host_p_in, host_size_p, host_t_in, host_size_t, host_mesh_z);
  std::vector<PointCu> *samples = generateSamplesFromTestdata(host_mesh_z, host_p_in, host_size_p);
  std::vector<double>    *betas = generateBetasFromTestdata(host_beta_v, host_mesh_z * host_size_t);
  std::vector<double>      *ase = new std::vector<double>(samples->size(), 0);
  const unsigned rays_total = rays_per_sample * samples->size();

  // Run 
  unsigned i;
  for(i=1; i < argc; ++i){
    if(strncmp(argv[i], "--mode=", 6) == 0){
      if(strstr(argv[i], "bruteforce_gpu") != 0){
  	runtime = runAseBruteforceGpu(samples, prisms, rays_per_sample, betas, ase, threads);
	strcpy(runmode, "Bruteforce GPU");

      }
      else if(strstr(argv[i], "naive_ray_propagation") != 0){
	runtime = runNaiveRayPropagation(ase);
	strcpy(runmode, "Naive Ray Propagation GPU");
	  }
	  else{
	fprintf(stderr, "C Runmode is not known\n");
	return 0;

      }

    }

  }

  // Print Solution
  unsigned sample_i;
  fprintf(stderr, "C Solutions\n");
  for(sample_i = 0; sample_i < ase->size(); ++sample_i){
    fprintf(stderr, "C ASE PHI of sample %d: %.80f\n", sample_i, ase->at(sample_i));

  }

  // Print statistics
  unsigned blocksPerSample = rays_per_sample / threads;
  unsigned blocks = blocksPerSample * samples->size();
  fprintf(stderr, "\n");
  fprintf(stderr, "C Statistics\n");
  fprintf(stderr, "C Prism             : %d\n", (int) prisms->size());
  fprintf(stderr, "C Triangles         : %d\n", (int) prisms->size() * 8);
  fprintf(stderr, "C Samples           : %d\n", (int) samples->size());
  fprintf(stderr, "C Rays/Sample       : %d\n", rays_per_sample);
  fprintf(stderr, "C GPU Blocks        : %d\n", blocks);
  fprintf(stderr, "C GPU Threads       : %d\n", threads);
  fprintf(stderr, "C GPU Blocks/Sample : %d\n", blocksPerSample);
  fprintf(stderr, "C Runmode           : %s \n", runmode);
  fprintf(stderr, "C Runtime           : %f s\n", runtime / 1000.0);
  fprintf(stderr, "\n");

  return 0;
}


