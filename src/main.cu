#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_vector_types.h"
#include "assert.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"
#include "datatypes.h"

#define SMALL 1E-06
#define CUDA_CHECK_RETURN(value) {				\
	hipError_t _mCudaStat = value;				\
	if (_mCudaStat != hipSuccess) {				\
		fprintf(stderr, "Error %s at line %d in file %s\n",	\
				hipGetErrorString(_mCudaStat), __LINE__, __FILE__);	\
		exit(1);							\
	}								\
}


//----------------------------------------------------
// Structures
//----------------------------------------------------
typedef struct point {
	float x;
	float y;
	float z;
} POINT;

typedef struct vector {
	float x;
	float y;
	float z;
} VECTOR;

typedef struct ray {
	point start;
	vector direction;
} RAY;

typedef struct triangle {
	point a;
	point b;
	point c;
} TRIANGLE;

typedef struct plane {
	point start;
	vector normal;

} PLANE;

//------------------------------------------

//----------------------------------------------------
// Auxillary function declaration
//----------------------------------------------------

float distance(point a, point b);
void  printPoint(point p);

// New functions
bool  collide(TriangleCu t, PointCu p);
bool  collide(TriangleCu t, RayCu r);
bool  collide(PrismCu pr, RayCu r);
float4 toBarycentric(TriangleCu t, PointCu p);
PointCu intersection(PlaneCu p, RayCu r);
std::vector<TriangleCu> generateTriangles(int height, int width, float level);
std::vector<PrismCu> generatePrisms(int height, int width, float level);
std::vector<RayCu> generateRays(int height, int width, int level, unsigned maxRays);
RayCu   generateRay(int height, int weight, int level);
std::vector<VertexCu> generateSamples(int height, int width, int level);

//----------------------------------------------------
// Device Code
//----------------------------------------------------

/**
  @brief Calculates A-B for 2 float4-based inputs
 **/
__device__ PointCu subtractPoints(PointCu A, PointCu B){
	PointCu C;
	C.x = A.x - B.x;
	C.y = A.y - B.y;
	C.z = A.z - B.z;
	C.w = A.w - B.w;
	return C;
}

__device__ RayCu generateRayGpu(PointCu vertexPoint, PrismCu startPrism, hiprandState randomstate){
	float u = hiprand_uniform(&randomstate);
	float v = hiprand_uniform(&randomstate);
	if((u+v) > 1){ //OPTIMIZE: remove if
		u = 1-u;
		v = 1-v;
	}
	const float w = 1-(u+v);

	PointCu A = startPrism.t1.A;
	PointCu B = startPrism.t1.B;
	PointCu C = startPrism.t1.C;

	// Get x and y coordinates from the random barycentric values
	const float xRand = u*A.x + v*B.x + w*C.x ;
	const float yRand = u*A.y + v*B.y + w*C.y ;

	// Take one of the given z-coordinates and add a random part of the prism height
	const float zRand = A.z + hiprand_uniform(&randomstate) * startPrism.t1.A.w;

	float ase=0.f;

	// Take the values to assemble a ray
	RayCu r = {
		{xRand, yRand, zRand, ase},
		vertexPoint};
	return r;
}

__device__ float distance(PointCu a, PointCu b){
  float d = sqrt(pow((b.x - a.x), 2) + pow((b.y - a.y),2) + pow((b.z - a.z),2));
  return fabs(d);
}

__device__ PrismCu selectPrism(int id, PrismCu prisms[], int totalNumberOfPrisms){
	int totalNumberOfThreads = blockDim.x * gridDim.x;
	int threadsPerPrism = ceil( float(totalNumberOfThreads) / float(totalNumberOfPrisms) );
	int prism = id / threadsPerPrism;
	return prisms[prism];
}

__device__ float propagate(RayCu ray, PrismCu prisms[], PrismCu startprism){
	float gain = 1.f;
	float vecX = ray.direction.x - ray.P.x;
	float vecY = ray.direction.y - ray.P.y;
	float vecZ = ray.direction.z - ray.P.z;

	float distanceTotal = sqrt(vecX*vecX+vecY*vecY+vecZ*vecZ);
	float distance = distanceTotal;
	float length = distanceTotal;
	vecX /= distanceTotal;
	vecY /= distanceTotal;
	vecZ /= distanceTotal;

	PrismCu current = startprism;


	for(;;){
		length = distance;
		//generate the triangle surfaces of the prism
		const TriangleCu t1 = current.t1;
		const TriangleCu t2 = { 
			{t1.A.x, t1.A.y, t1.A.z + t1.A.w, 1},
			{t1.B.x, t1.B.y, t1.B.z + t1.B.w, 1},
			{t1.C.x, t1.C.y, t1.C.z + t1.C.w, 1}
		};

		// OPTIMIZE: make use of the rectangles!
		const TriangleCu surfaces[8] = {
			t1,
			t2,
			{t1.A, t1.B, t2.A},
			{t1.B, t2.B, t2.A},
			{t1.B, t1.C, t2.C},
			{t1.B, t2.B, t2.C},
			{t1.A, t1.C, t2.C},
			{t1.A, t2.A, t2.C}
		};

		int i=0;
		float lengthHelp = 0.f;
		for(i=0; i<8 ; ++i){ //OPTIMIZE: unroll, so that every surface can be optimized differently
			// get the generating vectors for the plane
			VectorCu AB = subtractPoints(surfaces[i].B, surfaces[i].A);
			VectorCu AC = subtractPoints(surfaces[i].C, surfaces[i].A);

			PlaneCu pl;
			pl.P = surfaces[i].A;
			// cross product of the vectors
			pl.normal.x = AB.y*AC.z - AB.z*AC.y;
			pl.normal.y = AB.z*AC.x - AB.x*AC.z;
			pl.normal.z = AB.x*AC.y - AB.y*AC.x;

			// direction * pl.normal
			float denominator = (ray.direction.x * pl.normal.x) + (ray.direction.y * pl.normal.y) + (ray.direction.z * pl.normal.z);
			float d = 0.f;
			float nominator = 0.f;
			if(denominator != 0.f) //OPTIMIZE: check if we have a lot of branch diversion, or if all threads behave the same
			{
				// A * pl.normal
				d = (surfaces[i].A.x * pl.normal.x) + (surfaces[i].A.y * pl.normal.y) + (surfaces[i].A.z * pl.normal.z);
				// d - (P * pl.normal)
				nominator = d - ((ray.P.x * pl.normal.x) + (ray.P.y * pl.normal.y) + (ray.P.z * pl.normal.y)); 
				lengthHelp = nominator/denominator;
				if(lengthHelp < length && lengthHelp > 0.f) //OPTIMIZE: most threads should do the same?
				{
					length = lengthHelp;
				}
			}
		}


		//with the new length, get the gain and add it
		// @TODO
		gain += length;

		// calculate values for next iteration
		distance -= length;
		if(abs(distance) < SMALL)
		{
			break;
		}

		ray.P.x += length*vecX;
		ray.P.y += length*vecY;
		ray.P.z += length*vecZ;

		//@TODO:
		// calculate the next PRISM (maybe with help of some neighbor-datastructure?

	}


	return gain;
}

__global__ void setupKernel ( hiprandState * state, unsigned long seed ){
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	hiprand_init ( seed, id, 0, &state[id] );
	// OPTIMIZE: Use MersenneTwister or even a better PRNG
} 

// does the raytracing for a single ray (randomly generated) and a single (given) Vertex
//__global__ void raytraceStep( hiprandState* globalState, VertexCu vertex, PrismCu prisms[], int prismCount) {
__global__ void raytraceStep( VertexCu vertex, PrismCu prisms[], int prismCount) {
	//int id = threadIdx.x + blockDim.x*blockIdx.x;
	//hiprandState localState = globalState[id];

	//OPTIMIZE: the Octree should/could produce a subset of the prism-array!


	// this should give the same prism multiple times (so that every thread uses the same prism, which yields
	// big benefits for the memory access (and caching!)
	//PrismCu startprism = selectPrism(id, prisms, prismCount);	

	//RayCu ray = generateRayGpu(vertex.P,startprism, localState); //@TODO:verify
	//float initial_distance = distance(ray.P, ray.direction);

	//float gain = propagate(ray,prisms,startprism);

	//assert(fabs(gain-initial_distance) > 0.001);
		
	//atomicAdd(&(vertex.P.w),gain);
	//@TODO: find out, why atomic add won't compile
	//globalState[id] = localState;
}


//----------------------------------------------------
// Host Code
//----------------------------------------------------
int main(){

	//Variable definitions
	const unsigned maxRays = 100;
	const unsigned maxTriangles = 100;
	const unsigned maxVertices = 5;
	const unsigned length = ceil(sqrt(maxTriangles / 2));
	const unsigned depth  = 10;
	const unsigned maxPrisms = length * length * depth * 2;
	unsigned prism_i, vertex_i;
	float runtimeGpu = 0.0;
	float runtimeCpu = 0.0;
	hipEvent_t start, stop;
	bool useGpu = true;
	hiprandState* devStates;

	// Generate testdata
	std::vector<VertexCu> vertices = generateSamples(length, length, depth);
	std::vector<PrismCu> prisms = generatePrisms(length, length, depth);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU Raytracing
	PrismCu* hPrisms, *dPrisms;
	VertexCu* hVertices, *dVertices;
	int threads = 256;
	int blocks = ceil(maxPrisms / threads);
	if(useGpu){

		//initialize memory
		{
			// Memory allocation on host
			CUDA_CHECK_RETURN(hipHostAlloc( (void**)&hPrisms, prisms.size() * sizeof(PrismCu), hipHostMallocDefault));
			CUDA_CHECK_RETURN(hipHostAlloc( (void**)&hVertices, vertices.size() * sizeof(VertexCu), hipHostMallocDefault));

			// Memory initialisation on host
			for(prism_i = 0; prism_i < maxPrisms; ++prism_i){
				hPrisms[prism_i] = prisms[prism_i];
			}
			for(prism_i = 0; prism_i < vertices.size() ; ++prism_i){
				hVertices[prism_i] = vertices[prism_i];
			}


			// Memory allocation on device
			CUDA_CHECK_RETURN(hipMalloc(&dPrisms, prisms.size() * sizeof(PrismCu)));
			CUDA_CHECK_RETURN(hipMalloc(&dVertices, vertices.size() * sizeof(PrismCu)));
			CUDA_CHECK_RETURN(hipMalloc(&devStates, threads*blocks*sizeof( hiprandState )));

			// Copy data from host to device
			CUDA_CHECK_RETURN(hipMemcpy(dPrisms, hPrisms, maxPrisms * sizeof(PrismCu), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(dVertices, hVertices, vertices.size() * sizeof(VertexCu), hipMemcpyHostToDevice));
		}


		// Generating Random Numbers
	//	setupKernel<<< threads, blocks >>> ( devStates, time(NULL) );


		hipEventRecord(start, 0);
		// start the Kernels
//		for(vertex_i = 0; vertex_i < maxVertices; ++vertex_i){
//			raytraceStep<<< threads, blocks >>> ( devStates , vertices[vertex_i] , dPrisms, maxPrisms);
//		}
		fprintf(stderr, "\nbetween the kernel");
		raytraceStep<<< threads, blocks >>> ( dVertices[0] , dPrisms, prisms.size() );

		fprintf(stderr, "\nafter the kernel");
		// Free memory on device
		hipFree(devStates);

		// Evaluate device data
		{
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&runtimeGpu, start, stop);
		
		
		}
	}

	// print statistics
	{
	fprintf(stderr, "\n");
	fprintf(stderr, "Prism       : %d\n", maxPrisms);
	fprintf(stderr, "Triangles   : %d\n", maxPrisms * 8);
	fprintf(stderr, "Rays        : %d\n", maxRays);
	fprintf(stderr, "GPU Blocks  : %d\n", blocks);
	fprintf(stderr, "GPU Threads : %d\n", threads);
	fprintf(stderr, "Runtime_GPU : %f s\n", runtimeGpu / 1000.0);
	fprintf(stderr, "Runtime_CPU : %f s\n", runtimeCpu / 1000.0);
	fprintf(stderr, "\n");
	}
	// Cleanup
	hipHostFree(hPrisms);


	return 0;
}

//----------------------------------------------------
// Auxillary function definition
//----------------------------------------------------

float4 toBarycentric(TriangleCu t, PointCu p){
	float x1,x2,x3, y1,y2,y3, x,y;
	float4 b;

	x1 = t.A.x;
	x2 = t.B.x;
	x3 = t.C.x;

	y1 = t.A.y;
	y2 = t.B.y;
	y3 = t.C.y;

	x = p.x;
	y = p.y;

	b.x = ((y2-y3)*(x-x3)+(x3-x2)*(y-y3)) / ((y2-y3)*(x1-x3)+(x3-x2)*(y1-y3));
	b.y = ((y3-y1)*(x-x3)+(x1-x3)*(y-y3)) / ((y2-y3)*(x1-x3)+(x3-x2)*(y1-y3));
	b.z = 1 - b.x - b.y;
	b.w = 0;

	// In case of division by 0 --> nan
	if((fabs((b.x + b.y + b.z) - 1)) != (fabs((b.x + b.y + b.z) - 1)))
		b.z = 2;
	return b;
}

/**
  @brief Detects collisions of triangle and point with
  precondition, that the point is on the same 
  plane as the point.
 **/
bool collide(TriangleCu t, PointCu p){
	float4 b = toBarycentric(t, p);
	return (b.x > 0) && (b.x < 1) && (b.y > 0) && (b.y < 1) && (b.z > 0) && (b.z < 1) && (b.z == b.z);
}


/**
  @brief Detects collisions of a triangle and a ray without
  a precondition.
 **/
bool collide(TriangleCu t, RayCu r){
	PlaneCu pl;
	float b1, b2, b3, c1, c2, c3;

	b1 = t.B.x;
	b2 = t.B.y;
	b3 = t.B.z;

	c1 = t.C.x;
	c2 = t.C.y;
	c3 = t.C.z;

	pl.P = t.A;
	pl.normal.x = (b2*c3 - b3*c2);
	pl.normal.y = (b3*c1 - b1*c3);
	pl.normal.z = (b1*c2 - b2*c1);

	return collide(t, intersection(pl, r));
}

bool collide(PrismCu pr, RayCu r){
	bool hasCollide;
	PointCu A1 = pr.t1.A;
	PointCu B1 = pr.t1.B;
	PointCu C1 = pr.t1.C;
	PointCu A2 = {pr.t1.A.x, pr.t1.A.y, pr.t1.A.w, 1};
	PointCu B2 = {pr.t1.B.x, pr.t1.B.y, pr.t1.B.w, 1};
	PointCu C2 = {pr.t1.C.x, pr.t1.C.y, pr.t1.C.w, 1};

	TriangleCu triangles[8] = {
		pr.t1,
		{A2, B2, C2},
		{A1, B1, A2},
		{B1, B2, A2},
		{B1, C1, C2},
		{B1, B2, C2},
		{A1, C1, C2},
		{A1, A2, C2}};

	hasCollide = 
		collide(triangles[0], r)
		|| collide(triangles[1], r)
		|| collide(triangles[2], r) 
		|| collide(triangles[3], r)
		|| collide(triangles[4], r) 
		|| collide(triangles[5], r) 
		|| collide(triangles[6], r) 
		|| collide(triangles[7], r);

	return hasCollide;
}

/**
  @brief Intersection calculates the intersection between a plane p
  and a ray r. There is no detection for rays in the plane
  or for parallel plane. 

  It uses the normal of the plane to derive the coordinate form 
  of the plane. With the help of a coordinate form it is very
  easy to get the intersection point between a ray and a plane.

  ray   g: y~ = x~ + t*p~
  plane E: y~ = a~ + r*b~ + s*c~
  d  = n1*(x1+t*p1) + n2*(x2+t*p2) + n3*(x3+t*p3)
  d  = n~ * a~
 **/
PointCu intersection(PlaneCu pl, RayCu r){
	PointCu intersectionPoint = {0.0,0.0,0.0};

	float t, d;

	// vector coordinates
	float n1, n2, n3, x1, x2, x3, p1, p2, p3, a1, a2, a3;

	// just get the coordinates from the structs
	n1 = pl.normal.x;
	n2 = pl.normal.y;
	n3 = pl.normal.z;

	a1 = pl.P.x;
	a2 = pl.P.y;
	a3 = pl.P.z;

	x1 = r.P.x;
	x2 = r.P.y;
	x3 = r.P.z;

	p1 = r.direction.x;
	p2 = r.direction.y;
	p3 = r.direction.z;

	// calculation of intersection
	d = n1*a1 + n2*a2 + n3*a3;
	t = (d - n1*x1 - n2*x2 - n3*x3) / (n1*p1 + n2*p2 + n3*p3);

	intersectionPoint.x = x1 + t * p1;
	intersectionPoint.y = x2 + t * p2;
	intersectionPoint.z = x3 + t * p3;

	return intersectionPoint;

}

float distance(point a, point b){
	float d = sqrt(pow((b.x - a.x), 2) + pow((b.y - a.y),2) + pow((b.z - a.z),2));
	return fabs(d);
}

std::vector<TriangleCu> generateTriangles(int height, int weight, float level){
	int h,w;
	std::vector<TriangleCu> triangles;
	for(h = 0; h < height; ++h){
		for(w = 0; w < weight; ++w){
			TriangleCu t1 = {
				{float(h), float(w), level, 1},
				{float(h), float(w+1), level, 1},
				{float(h+1), float(w), level, 1}};
			TriangleCu t2 = {
				{float(h), float(w+1), level, 1},
				{float(h+1), float(w+1), level, 1},
				{float(h+1), float(w), level, 1}};
			triangles.push_back(t1);
			triangles.push_back(t2);

		}

	}

	return triangles;
}

std::vector<PrismCu> generatePrisms(int height, int weight, float level){
	int h,w,l;
	std::vector<PrismCu> prisms;
	for(l = 0; l < level; ++l){
		for(h = 0; h < height; ++h){
			for(w = 0; w < weight; ++w){
				TriangleCu a1 = {
					{float(h), float(w), l, l+1},
					{float(h), float(w+1), l, l+1},
					{float(h+1), float(w), l, l+1}};
				TriangleCu b1 = {
					{float(h), float(w+1), l, 1+1},
					{float(h+1), float(w+1), l, 1+1},
					{float(h+1), float(w), l, 1+1}};

				PrismCu pr1 = {a1};
				PrismCu pr2 = {b1};

				prisms.push_back(pr1);
				prisms.push_back(pr2);

			}

		}

	}

	return prisms;
}

RayCu generateRay(const int heigth, const int width, const int level){
	float randHeigth = float(rand() % heigth) + (rand() / (float) RAND_MAX);
	float randWidth  = float(rand() % width ) + (rand() / (float) RAND_MAX);
	float rand_level  = float(rand() % level ) + (rand() / (float) RAND_MAX);

	float dirX = (rand() / (float) RAND_MAX);
	float dirY = (rand() / (float) RAND_MAX);
	float dirZ = (rand() / (float) RAND_MAX);

	RayCu r = {
		{randHeigth, randWidth, rand_level, 1},
		{dirX, dirY, dirZ, 0}};
	return r;
}


std::vector<RayCu> generateRays(const int height, const int width, const int level, const unsigned maxRays){
	std::vector<RayCu> rays;
	unsigned ray_i;
	for(ray_i = 0; ray_i < maxRays; ++ray_i){
		RayCu ray = generateRay(height, width, level);
		rays.push_back(ray);
	}
	return rays;
}

void printPoint(point p){
	fprintf(stdout, "Point\n");
	fprintf(stdout, "x: %f\n", p.x);
	fprintf(stdout, "y: %f\n", p.y);
	fprintf(stdout, "z: %f\n", p.z);

}
std::vector<VertexCu> generateSamples(int height, int width, int level){
  std::vector<VertexCu> samplePoints;
  int h,w,l;
  for(l = 0; l <= level; ++l){
    for(h = 0; h <= height; ++h){
      for(w = 0; w <= width; ++w){
	
		  VertexCu p = {{float(h), float(w), float(l)}};

		  samplePoints.push_back(p);
      }
    }
  }
  return samplePoints;
}
