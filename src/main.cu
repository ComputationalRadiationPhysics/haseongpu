#include "hip/hip_runtime.h"
// Libraries
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_vector_types.h"
#include "assert.h"
#include "string.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"
#include "datatypes.h"

// User header files
#include "geometry.h"
#include "datatypes.h"
#include "generate_testdata.h"
#include "print.h"
#include "geometry_gpu.h"
#include "ase_bruteforce_kernel.h"
#include "ase_bruteforce_cpu.h"

int main(int argc, char **argv){
  const unsigned max_rays = 10000;
  const unsigned max_triangles = 2;
  const unsigned depth  = 2;
  const unsigned length = ceil(sqrt(max_triangles / 2));
  const int threads = 256;
  char runmode[20];
  float runtime = 0.0;
  
  // Parse Commandline
  if(argc <= 1){
    fprintf(stderr, "C No commandline arguments found\n");
    fprintf(stderr, "C Usage    : ./octrace --mode=[runmode]\n");
    fprintf(stderr, "C Runmodes : bruteforce_cpu\n");
    fprintf(stderr, "             bruteforce_gpu\n");
    return 0;
  }
  
  // Generate testdata
  fprintf(stderr, "C Generate Testdata\n");
  std::vector<PrismCu> *prisms  = generate_prisms(length, length, depth);
  std::vector<PointCu> *samples = generate_samples(length, length, depth);
  std::vector<RayCu> *rays      = generate_sample_rays(length, length, depth, max_rays, samples);
  std::vector<float> *ase       = new std::vector<float>(samples->size(), 0);

  // Run 
  unsigned i;
  for(i=1; i < argc; ++i){
    if(strncmp(argv[i], "--mode=", 6) == 0){
      if(strstr(argv[i], "bruteforce_cpu") != 0){
  	runtime = runAseBruteforceCpu(samples, prisms, rays, ase);
	strcpy(runmode, "Bruteforce CPU");

      }
      else if(strstr(argv[i], "bruteforce_gpu") != 0){
  	runtime = runAseBruteforceGpu(samples, prisms, rays, ase, threads);
	strcpy(runmode, "Bruteforce GPU");

      }
      else{
	fprintf(stderr, "C Runmode is not know\n");
	return 0;

      }

    }
       
  }

  // Print Solution
  unsigned sample_i;
  fprintf(stderr, "C Solutions\n");
  for(sample_i = 0; sample_i < samples->size(); ++sample_i){
    fprintf(stderr, "C ASE PHI of sample %d: %f\n", sample_i, ase->at(sample_i));

  }

  // Print statistics
  unsigned blocks_per_sample = ceil(rays->size() / (threads * samples->size()));
  unsigned blocks = blocks_per_sample * samples->size();
  fprintf(stderr, "\n");
  fprintf(stderr, "C Statistics\n");
  fprintf(stderr, "C Prism             : %d\n", (int) prisms->size());
  fprintf(stderr, "C Triangles         : %d\n", (int) prisms->size() * 8);
  fprintf(stderr, "C Samples           : %d\n", (int) samples->size());
  fprintf(stderr, "C Rays/Sample       : %d\n", max_rays);
  fprintf(stderr, "C GPU Blocks        : %d\n", blocks);
  fprintf(stderr, "C GPU Threads       : %d\n", threads);
  fprintf(stderr, "C GPU Blocks/Sample : %d\n", blocks_per_sample);
  fprintf(stderr, "C Runmode           : %s \n", runmode);
  fprintf(stderr, "C Runtime           : %f s\n", runtime / 1000.0);
  fprintf(stderr, "\n");

  return 0;
}


