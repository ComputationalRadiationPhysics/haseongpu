#include "hip/hip_runtime.h"
// Libraries
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_vector_types.h"
#include "assert.h"
#include "string.h"
#include <vector>
#include "hiprand/hiprand_kernel.h"
#include "datatypes.h"

// User header files
#include "geometry.h"
#include "datatypes.h"
#include "generate_testdata.h"
#include "print.h"
#include "geometry_gpu.h"
#include "ase_bruteforce_kernel.h"
#include "ase_bruteforce_cpu.h"
#include "testdata.h"

int main(int argc, char **argv){
  const unsigned rays_per_sample = 100000;
  const unsigned max_triangles = 2;
  const unsigned depth  = 2;
  const unsigned length = ceil(sqrt(max_triangles / 2));
  const int threads = 256;
  char runmode[20];
  float runtime = 0.0;
  
  // Parse Commandline
  if(argc <= 1){
    fprintf(stderr, "C No commandline arguments found\n");
    fprintf(stderr, "C Usage    : ./octrace --mode=[runmode]\n");
    fprintf(stderr, "C Runmodes : bruteforce_cpu\n");
    fprintf(stderr, "             bruteforce_gpu\n");
    return 0;
  }
  
  // Generate testdata
  fprintf(stderr, "C Generate Testdata\n");
  //std::vector<PrismCu> *prisms  = generate_prisms(length, length, depth);
  std::vector<PrismCu> *prisms  = generatePrismsFromTestdata(host_z_mesh, host_p_in, host_t_in, host_number_of_triangles, host_mesh_z);
  //std::vector<PointCu> *samples = generate_samples(length, length, depth);
  std::vector<PointCu> *samples = generateSampesFromTestdata(host_z_mesh, host_p_in, host_number_of_points);
  std::vector<RayCu> *rays      = generate_sample_rays(length, length, depth, rays_per_sample, samples);
  std::vector<float> *ase       = new std::vector<float>(samples->size(), 0);

  // Run 
  unsigned i;
  for(i=1; i < argc; ++i){
    if(strncmp(argv[i], "--mode=", 6) == 0){
      if(strstr(argv[i], "bruteforce_cpu") != 0){
  	runtime = runAseBruteforceCpu(samples, prisms, rays, ase);
	strcpy(runmode, "Bruteforce CPU");

      }
      else if(strstr(argv[i], "bruteforce_gpu") != 0){
  	runtime = runAseBruteforceGpu(samples, prisms, rays, ase, threads);
	strcpy(runmode, "Bruteforce GPU");

      }
      else{
	fprintf(stderr, "C Runmode is not known\n");
	return 0;

      }

    }
       
  }

  // Print Solution
  unsigned sample_i;
  fprintf(stderr, "C Solutions\n");
  for(sample_i = 0; sample_i < samples->size(); ++sample_i){
    fprintf(stderr, "C ASE PHI of sample %d: %f\n", sample_i, ase->at(sample_i));

  }

  // Print statistics
  unsigned blocksPerSample = ceil(rays->size() / (threads * samples->size()));
  unsigned blocks = blocksPerSample * samples->size();
  fprintf(stderr, "\n");
  fprintf(stderr, "C Statistics\n");
  fprintf(stderr, "C Prism             : %d\n", (int) prisms->size());
  fprintf(stderr, "C Triangles         : %d\n", (int) prisms->size() * 8);
  fprintf(stderr, "C Samples           : %d\n", (int) samples->size());
  fprintf(stderr, "C Rays/Sample       : %d\n", rays_per_sample);
  fprintf(stderr, "C GPU Blocks        : %d\n", blocks);
  fprintf(stderr, "C GPU Threads       : %d\n", threads);
  fprintf(stderr, "C GPU Blocks/Sample : %d\n", blocksPerSample);
  fprintf(stderr, "C Runmode           : %s \n", runmode);
  fprintf(stderr, "C Runtime           : %f s\n", runtime / 1000.0);
  fprintf(stderr, "\n");

  return 0;
}


