#include "hip/hip_runtime.h"
#include <cuda_utils.h>

#include <logging.h>
#include <vector>


/** 
 * @brief Queries for devices on the running mashine and collects
 *        them on the devices array. Set the first device in this 
 *        array as computation-device. On Errors the programm will
 *        be stoped by exit(). 
 * 
 * @param maxGpus max. devices which should be allocated
 * @return vector of possible devices
 */
std::vector<unsigned> getFreeDevices(unsigned maxGpus){
  hipDeviceProp_t prop;
  int minMajor = MIN_COMPUTE_CAPABILITY_MAJOR;
  int minMinor = MIN_COMPUTE_CAPABILITY_MINOR;
  int count;
  std::vector<unsigned> devices;

  // Get number of devices
  CUDA_CHECK_RETURN( hipGetDeviceCount(&count));

  // Check devices for compute capability and if device is busy
  unsigned devicesAllocated = 0;
  for(int i=0; i < count; ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, i) );
    if( (prop.major > minMajor) || (prop.major == minMajor && prop.minor >= minMinor) ){
      hipSetDevice(i);
      int* occupy; //TODO: occupy gets allocated, but never hipFree'd -> small memory leak!
      if(hipMalloc((void**) &occupy, sizeof(int)) == hipSuccess){
        devices.push_back(i);
        devicesAllocated++;
        if(devicesAllocated == maxGpus)
          break;

      }

    }

  }
  // Exit if no device was found
  if(devices.size() == 0){
    dout(V_ERROR) << "None of the free CUDA-capable devices is sufficient!" << std::endl;
    exit(1);
  }

  // Print device information
  hipSetDevice(devices.at(0));
  dout(V_INFO) << "Found " << int(devices.size()) << " available CUDA devices with Compute Capability >= " << minMajor << "." << minMinor << "):" << std::endl;
  for(unsigned i=0; i<devices.size(); ++i){
    CUDA_CHECK_RETURN( hipGetDeviceProperties(&prop, devices[i]) );
    dout(V_INFO) << "[" << devices[i] << "] " << prop.name << " (Compute Capability " << prop.major << "." << prop.minor << ")" << std::endl;
  }

  return devices;

}
