#include "hip/hip_runtime.h"
#include "progressbar.h"
#include <unistd.h>
#include <ctime>
#include <logging.h>
#include <cmath>
#include <iomanip>
#include <fstream>

void printWave(std::ostream &stream,unsigned part,int progress,int length){
  for(int i=0;i<progress ;++i){
    switch((part-i) % 12){
      case 0: stream << "ø"; break;
      case 1: stream << "¤"; break;
      case 2: stream << "º"; break;
      case 3: stream << "°"; break;
      case 4: stream << "`"; break;
      case 5: stream << "°"; break;
      case 6: stream << "º"; break;
      case 7: stream << "¤"; break;
      case 8: stream << "ø"; break;
      case 9: stream << ","; break;
      case 10: stream << "¸"; break;
      case 11: stream << ","; break;
    }
  }
  for(int i=0; i < length-progress ; ++i){
    stream << " ";
  }
}

void simpleProgressBar(unsigned part, unsigned full){
	unsigned length = 80;

	float percentage = (float(part)+1) / float(full);

	dout(V_INFO | V_NOLABEL) << "\r";
	dout(V_INFO) << "Progress: [";
	for(int i=0 ; i < (percentage*length) ; i++){
		dout(V_INFO | V_NOLABEL) << "#";
	}
	for(int i=0;i< length-(percentage*length) ;i++){
		dout(V_INFO | V_NOLABEL) << " ";
	}
	dout(V_INFO | V_NOLABEL) << "] " << int(percentage*100) << "% (" << part+1 << "/" << full << std::flush;
}


void fancyProgressBar(unsigned part, unsigned full, unsigned length, time_t starttime){

	float percentage = (float(part)+1) / float(full);

	dout(V_INFO | V_NOLABEL) << "\r";
	dout(V_INFO) << "Progress: [";
  
  printWave(dout(V_INFO | V_NOLABEL), part, int(percentage*length), length);

	double timeSpent = difftime(time(0),starttime);
	int timeTotal = timeSpent/percentage;
	int timeRemaining = timeTotal-timeSpent;
	dout(V_INFO | V_NOLABEL) << "] " << std::setfill(' ') << std::setw(3) << int(percentage*100) << "% (" << part+1 << "/" << full << ") after " << int(timeSpent) << "s (" << timeTotal << "s total, " << timeRemaining << "s remaining)" << std::flush;
}


void fileProgressBar(unsigned nTotal, std::string path){
	int length = 50;
	static unsigned part = 0;
	static const time_t starttime = time(0);
	static const unsigned fillwidthPart = unsigned(1+log10(nTotal));
	std::ofstream filestream;

	++part;
	const float percentage = float(part) / float(nTotal);

		if(!filestream.is_open()){
			filestream.open(path.c_str(),std::ofstream::trunc);
		}
  
		filestream << "Progress: [";
    printWave(filestream,part,int(percentage*length),length);
		filestream << "] ";

		// write progress in percent
		filestream << std::setfill(' ') << std::setw(3) << unsigned(percentage*100) << "% (" << std::setfill(' ') << std::setw(fillwidthPart) << part << "/" << nTotal << ")" << std::endl;

		// go to next line and write time
		const double timeSpent = difftime(time(0),starttime);
		const int timeTotal = timeSpent/percentage;
		const int timeRemaining = timeTotal-timeSpent;
		filestream << "Runtime " << int(timeSpent) << "s (" << timeTotal << "s total, " << timeRemaining << "s remaining)" << std::flush;

		filestream.close();
}

