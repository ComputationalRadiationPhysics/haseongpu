#include "hip/hip_runtime.h"
#include "progressbar.h"
#include <unistd.h>
#include <ctime>
#include <logging.h>
#include <cmath>
#include <iomanip>
#include <fstream>

void simpleProgressBar(unsigned part, unsigned full){
	unsigned length = 80;

	float percentage = (float(part)+1) / float(full);

	dout(V_INFO | V_NOLABEL) << "\r";
	dout(V_INFO) << "Progress: [";
	for(int i=0 ; i < (percentage*length) ; i++){
		dout(V_INFO | V_NOLABEL) << "#";
	}
	for(int i=0;i< length-(percentage*length) ;i++){
		dout(V_INFO | V_NOLABEL) << " ";
	}
	dout(V_INFO | V_NOLABEL) << "] " << int(percentage*100) << "%% (" << part+1 << "/" << full << std::flush;
}

void fancyProgressBar(unsigned part, unsigned full, unsigned length, time_t starttime){

	float percentage = (float(part)+1) / float(full);

	dout(V_INFO | V_NOLABEL) << "\r";
	dout(V_INFO) << "Progress: [";
	for(int i=0 ; i < (percentage*length) ; i++){
		dout(V_INFO | V_NOLABEL) << "#";
	}
	for(int i=0;i< (length-(percentage*length)-1) ;i++){
		dout(V_INFO | V_NOLABEL) << " ";
	}
	time_t now = time(0);
	double timeSpent = difftime(now,starttime);
	int timeTotal = timeSpent/percentage;
	int timeRemaining = timeTotal-timeSpent;
	dout(V_INFO | V_NOLABEL) << "] " << int(percentage*100) << "% (" << part+1 << "/" << full << ") after " << int(timeSpent) << "s (" << timeTotal << "s total, " << timeRemaining << "remaining)" << std::flush; 
}


void fileProgressBar(unsigned nTotal, std::string path){
	unsigned length = 50;
	static unsigned part = 0;
	static unsigned progress = 0;
	static const time_t starttime = time(0);
	static const unsigned fillwidthPart = unsigned(1+log10(nTotal));
	static std::ofstream filestream;
	static std::streampos currentPositionOfProgress,endOfProgressBar,maximalEndOfFile = 0;

	++part;
	const float percentage = float(part) / float(nTotal);

	if(!filestream.is_open()){
		filestream.open(path.c_str());
	}

	// set up the empty progress bar
	if(part == 1){
		filestream << "Progress: [";
		currentPositionOfProgress = filestream.tellp();
		for(unsigned i=0;i<length;++i){
			filestream << " ";
		}
		filestream << "] ";
		endOfProgressBar = filestream.tellp();
		maximalEndOfFile = endOfProgressBar;
	}

	// write progressbar
	while(unsigned(percentage*length) > progress){
		++progress;
		filestream.seekp(currentPositionOfProgress);
		filestream << "#";
		currentPositionOfProgress = filestream.tellp();
	}

	// write progress in percent
	filestream.seekp(endOfProgressBar);
	filestream << std::setfill(' ') << std::setw(3) << unsigned(percentage*100) << "% (" << std::setfill(' ') << std::setw(fillwidthPart) << part << "/" << nTotal << ")" << std::endl;

	// go to next line and write time
	const double timeSpent = difftime(time(0),starttime);
	const int timeTotal = timeSpent/percentage;
	const int timeRemaining = timeTotal-timeSpent;
	filestream << "Runtime " << int(timeSpent) << "s (" << timeTotal << "s total, " << timeRemaining << "s remaining)" << std::flush;

	// append empty characters to overwrite possible dangling characters
	while(filestream.tellp() < maximalEndOfFile){
		filestream << " ";
	}
	maximalEndOfFile = filestream.tellp();

	// close the file if the progress bar is complete
	if(part == nTotal){
		filestream.close();
	}
}

